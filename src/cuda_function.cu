#include "hip/hip_runtime.h"
#include <itpp/base/array.h>
#include <itpp/base/converters.h>
#include <itpp/base/itassert.h>
#include <itpp/base/mat.h>
#include <itpp/base/matfunc.h>
#include <itpp/base/random.h>
#include <itpp/base/vec.h>
#include <itpp/base/math/elem_math.h>
#include <boost/math/special_functions/gamma.hpp>
#include <sys/time.h>
#include <cmath>
#include <list>
#include <iomanip>
#include <algorithm>
#include <vector>

#include "macros.h"
#include "common.h"
#include "lte_lib.h"
#include "constants.h"
#include "dsp.h"
#include "itpp_ext.h"

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace itpp;

__constant__ hipfftComplex pss_td[3][256];

#define THREAD_DIM_X   (256)

extern "C" void copy_pss_to_device()
{
    int i, t, len;
    hipfftComplex pss[3][256];
  
    for (t = 0; t < 3; t++) {
        len = ROM_TABLES.pss_td[t].length();
        for (i = 0; i < len; i++) {
            pss[t][i].x = ROM_TABLES.pss_td[t][i].real();
            pss[t][i].y = ROM_TABLES.pss_td[t][i].imag();
        }
        for (; i < 256; i++) {
            pss[t][i].x = 0.0f;
            pss[t][i].y = 0.0f;
        }
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pss_td), &pss, sizeof(pss)));
}

#define COMPLEX_MUL_REAL(a, b)  ((a).x * (b).x - (a).y * (b).y)
#define COMPLEX_MUL_IMAG(a, b)  ((a).x * (b).y + (a).y * (b).x)

__global__ void xc_correlate_kernel(hipfftComplex *d_capbuf, float *d_xc_sqr, 
                                    float *d_xc_incoherent_single, float *d_xc_incoherent,
                                    uint8 t, unsigned int n_cap, uint8 ds_comb_arm, 
                                  double f, double fs)
{
    __shared__ hipfftComplex s_fshift_pss[THREAD_DIM_X], s_capbuf[THREAD_DIM_X + 137];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    double k = HIP_PI * f * 2 / fs;
    double shift = k * tid;
    double x1 = cos(shift), y1 = sin(shift);
    double x2 = pss_td[t][tid].x, y2 = pss_td[t][tid].y;
    unsigned int max_m = (n_cap - 100 - 136) / 9600;
    unsigned int i, m;
 
    s_fshift_pss[tid].x = x1*x2 - y1*y2;
    s_fshift_pss[tid].y = -x1*y2 - x2*y1;

    s_capbuf[tid] = d_capbuf[THREAD_DIM_X * bid + tid];

    if (tid < 137) {
        if (THREAD_DIM_X * bid + THREAD_DIM_X + tid < n_cap) {
            s_capbuf[THREAD_DIM_X + tid] = d_capbuf[THREAD_DIM_X * bid + THREAD_DIM_X + tid];
        } else {
            s_capbuf[THREAD_DIM_X + tid] = d_capbuf[tid];
        }
    }
  
    __syncthreads();

    float real, imag;

    real = COMPLEX_MUL_REAL(s_fshift_pss[0], s_capbuf[tid]);
    imag = COMPLEX_MUL_IMAG(s_fshift_pss[0], s_capbuf[tid]);
    for (i = 1; i < 137; i++) {
        real += COMPLEX_MUL_REAL(s_fshift_pss[i], s_capbuf[tid + i]);
        imag += COMPLEX_MUL_IMAG(s_fshift_pss[i], s_capbuf[tid + i]);
    }
    d_xc_sqr[THREAD_DIM_X * bid + tid] = (real * real + imag * imag) / (137.0*137.0);

    __syncthreads();

    if (tid < 16) {
        unsigned int index = 16 * bid + tid;
        float xc_incoherent_single_val = d_xc_sqr[index];
        for (m = 1; m < max_m; m++) {
            unsigned int span = m * 0.005 * fs;
            xc_incoherent_single_val += d_xc_sqr[index + span];
        }
        float xc_incoherent_value = d_xc_incoherent_single[index] = xc_incoherent_single_val / max_m;

        __syncthreads();

        for (i = 1; i <= ds_comb_arm; i++) {
            if (index + i < 9600) {
                xc_incoherent_value += d_xc_incoherent_single[index + i];
            } else {
                xc_incoherent_value += d_xc_incoherent_single[index + i - 9600];
            }
            if (index > i) {
                xc_incoherent_value += d_xc_incoherent_single[index - i];
            } else {
                xc_incoherent_value += d_xc_incoherent_single[index - i + 9600];
            }
        }
        d_xc_incoherent[index] = xc_incoherent_value / (ds_comb_arm * 2 + 1);
    }

    __syncthreads();
}

void xc_correlate_step(const cvec & capbuf, vec &xc_sqr, vf3d & xc_incoherent_single, vf3d & xc_incoherent, uint8 t, uint16 foi, double f, double fs, uint8 ds_comb_arm)
{
    hipfftComplex *h_capbuf;
    float *h_xc_sqr, *h_xc_incoherent_single, *h_xc_incoherent;

    unsigned int n_cap = capbuf.length();
    hipfftComplex *d_capbuf = (hipfftComplex *)NULL;
    float *d_xc_sqr = (float *)NULL;
    float *d_xc_incoherent_single = (float *)NULL;
    float *d_xc_incoherent = (float *)NULL;
 
    h_capbuf = (hipfftComplex *)malloc(n_cap * sizeof(hipfftComplex));
    h_xc_sqr = (float *)malloc(n_cap * sizeof(float));
    h_xc_incoherent_single = (float *)malloc(9600 * sizeof(float));
    h_xc_incoherent = (float *)malloc(9600 * sizeof(float));
 
    checkCudaErrors(hipMalloc(&d_capbuf, n_cap * sizeof(hipfftComplex)));
    checkCudaErrors(hipMalloc(&d_xc_sqr, n_cap * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_single, 9600 * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent, 9600 * sizeof(float)));

    for (unsigned int i = 0; i < n_cap; i++) {
        h_capbuf[i].x = capbuf[i].real();
        h_capbuf[i].y = capbuf[i].imag();
    }
    checkCudaErrors(hipMemcpy(d_capbuf, h_capbuf, n_cap * sizeof(hipfftComplex), hipMemcpyHostToDevice));

    xc_correlate_kernel<<<600, 256>>>(d_capbuf, d_xc_sqr, d_xc_incoherent_single, d_xc_incoherent, t, n_cap, ds_comb_arm, f, fs);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_xc_sqr, d_xc_sqr, n_cap * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_single, d_xc_incoherent_single, 9600 * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent, d_xc_incoherent, 9600 * sizeof(float), hipMemcpyDeviceToHost));

    for (unsigned int i = 0; i < n_cap - 136; i++) {
        xc_sqr[i] = h_xc_sqr[i];
    }

    for (unsigned int i = 0; i < 9600; i++) {
        xc_incoherent_single[t][i][foi] = h_xc_incoherent_single[i];
    }
    for (unsigned int i = 0; i < 9600; i++) {
        xc_incoherent[t][i][foi] = h_xc_incoherent[i];
    }

    free(h_capbuf);
    free(h_xc_sqr);
    free(h_xc_incoherent_single);
    free(h_xc_incoherent);
    checkCudaErrors(hipFree(d_capbuf));
    checkCudaErrors(hipFree(d_xc_sqr));
    checkCudaErrors(hipFree(d_xc_incoherent_single));
    checkCudaErrors(hipFree(d_xc_incoherent));
}


void xcorr_pss2(
  // Inputs
  const cvec & capbuf,
  const vec & f_search_set,
  const uint8 & ds_comb_arm,
  const double & fc_requested,
  const double & fc_programmed,
  const double & fs_programmed,
  // Outputs
  mat & xc_incoherent_collapsed_pow,
  imat & xc_incoherent_collapsed_frq,
  // Following used only for debugging...
  vf3d & xc_incoherent_single,
  vf3d & xc_incoherent,
  vec & sp_incoherent,
  vcf3d & xc,
  vec & sp,
  uint16 & n_comb_xc,
  uint16 & n_comb_sp
) 
{
  // Perform correlations
  const uint32 n_cap=length(capbuf);
  const uint16 n_f=length(f_search_set);

  // Set aside space for the vector and initialize with NAN's.
#ifndef NDEBUG
  xc = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f,NAN)));
  vcf3d xc2 = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f,NAN)));
  xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f,NAN)));
  xc_incoherent = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f,NAN)));
#else
  xc = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f)));
  vcf3d xc2 = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f)));
  xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));
  xc_incoherent = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));
#endif
  vec xc_sqr = vec(n_cap-136);

  // Local variables declared outside of the loop.
  double f_off;
  cvec temp;
  complex <double> acc;
  uint16 foi;
  uint8 t;
  uint32 k;
  uint8 m;

  struct timeval tv1, tv2;
  gettimeofday(&tv1, NULL);

  // Loop and perform correlations.
  // Incoherently combine correlations
  n_comb_xc = floor_i((xc[0].size()-100)/9600);
  const uint8 ds_com_arm_weight = (2*ds_comb_arm+1);
  printf("n_cap=%d\n", n_cap);

  for (foi=0;foi<n_f;foi++) {
    f_off = f_search_set(foi);
    const double k_factor = (fc_requested-f_off)/fc_programmed;
    for (t = 0; t < 3;t++) {
#if 1
      xc_correlate_step(capbuf, xc_sqr, xc_incoherent_single, xc_incoherent, t, foi, f_off, fs_programmed*k_factor, ds_comb_arm);
#else
      temp = ROM_TABLES.pss_td[t];
      temp = fshift(temp,f_off,fs_programmed*k_factor);
      temp = conj(temp)/137;
#ifdef _OPENMP
#pragma omp parallel for shared(temp,capbuf,xc) private(k,acc,m)
#endif
      for (k=0;k<n_cap-136;k++) {
        acc=0;
        for (m=0;m<137;m++) {
          // Correlations are performed at the 2x rate which effectively
          // performs filtering and correlating at the same time. Thus,
          // this algorithm can handle huge frequency offsets limited only
          // by the bandwidth of the capture device.
          // Correlations can also be done at the 1x rate if filtering is
          // peformed first, but this will limit the set of frequency offsets
          // that this algorithm can detect. 1x rate correlations will,
          // however, be nearly twice as fast as the 2x correlations
          // performed here.
          acc+=temp(m)*capbuf(k+m);
        }
        xc[t][k][foi]=acc;
      }

      for (uint16 idx=0;idx<9600;idx++) {
        // Because of the large supported frequency offsets and the large
        // amount of time represented by the capture buffer, the length
        // in samples, of a frame varies by the frequency offset.
        //double actual_time_offset=m*.005*k_factor;
        //double actual_start_index=itpp::round_i(actual_time_offset*FS_LTE/16);
        xc_incoherent_single[t][idx][foi] = 0;
        for (uint16 m = 0; m < n_comb_xc; m++) {
          uint32 actual_start_index = itpp::round_i(m*.005*k_factor*fs_programmed);
          xc_incoherent_single[t][idx][foi] += xc[t][idx + actual_start_index][foi].real() * xc[t][idx + actual_start_index][foi].real() + xc[t][idx + actual_start_index][foi].imag() * xc[t][idx + actual_start_index][foi].imag();
        }
        xc_incoherent_single[t][idx][foi]/= n_comb_xc;
      }

      for (uint16 idx=0;idx<9600;idx++) {
        xc_incoherent[t][idx][foi] = xc_incoherent_single[t][idx][foi];
        for (uint8 k=1;k<=ds_comb_arm;k++) {
          xc_incoherent[t][idx][foi] += (xc_incoherent_single[t][itpp_ext::matlab_mod(idx-k,9600)][foi] + xc_incoherent_single[t][itpp_ext::matlab_mod(idx+k,9600)][foi]);
        }
        xc_incoherent[t][idx][foi] /= ds_com_arm_weight;
      }
#endif
    }
  }

  // Estimate received signal power
  // const uint32 n_cap=length(capbuf);
  n_comb_sp = floor_i((n_cap-136-137)/9600);
  const uint32 n_sp = n_comb_sp*9600;

  // Set aside space for the vector and initialize with NAN's.
  sp = vec(n_sp);
  xc_incoherent_collapsed_pow = mat(3,9600);
  xc_incoherent_collapsed_frq = imat(3,9600);
#ifndef NDEBUG
  sp = NAN;
  xc_incoherent_collapsed_pow = NAN;
  xc_incoherent_collapsed_frq = -1;
#endif
  sp[0] = 0;
  // Estimate power for first time offset
  for (uint16 t=0;t<274;t++) {
    sp[0] += pow(capbuf[t].real(),2) + pow(capbuf[t].imag(),2);
  }
  sp[0] = sp[0] / 274;
  // Estimate RX power for remaining time offsets.
  for (uint32 t=1;t<n_sp;t++) {
    sp[t] = sp[t-1] + (-pow(capbuf[t-1].real(),2)-pow(capbuf[t-1].imag(),2)+pow(capbuf[t+274-1].real(),2)+pow(capbuf[t+274-1].imag(),2))/274;
  }

  // Combine incoherently
  sp_incoherent = sp.left(9600);
  for (uint16 t=1; t < n_comb_sp;t++) {
    sp_incoherent += sp.mid(t*9600, 9600);
  }
  sp_incoherent = sp_incoherent / n_comb_sp;

  // Shift to the right by 137 samples to align with the correlation peaks.
  tshift(sp_incoherent, 137);

  // Search for peaks among all the frequency offsets.
  // const int n_f=xc_incoherent[0][0].size();

  for (uint8 t=0;t<3;t++) {
    for (uint16 k=0;k<9600;k++) {
      double best_pow=xc_incoherent[t][k][0];
      uint16 best_idx=0;
      for (uint16 foi=1;foi<n_f;foi++) {
        if (xc_incoherent[t][k][foi]>best_pow) {
          best_pow=xc_incoherent[t][k][foi];
          best_idx=foi;
        }
      }
      xc_incoherent_collapsed_pow(t,k)=best_pow;
      xc_incoherent_collapsed_frq(t,k)=best_idx;
    }
  }

  gettimeofday(&tv2, NULL);
  printf("xcorr_pss2 : %ld us\n", (tv2.tv_sec-tv1.tv_sec)*1000000+(tv2.tv_usec-tv1.tv_usec));
}



