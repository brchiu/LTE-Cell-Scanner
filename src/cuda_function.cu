#include "hip/hip_runtime.h"
#include <itpp/base/array.h>
#include <itpp/base/converters.h>
#include <itpp/base/itassert.h>
#include <itpp/base/mat.h>
#include <itpp/base/matfunc.h>
#include <itpp/base/random.h>
#include <itpp/base/vec.h>
#include <itpp/base/math/elem_math.h>
#include <boost/math/special_functions/gamma.hpp>
#include <sys/time.h>
#include <cmath>
#include <list>
#include <iomanip>
#include <algorithm>
#include <vector>

#include "macros.h"
#include "common.h"
#include "lte_lib.h"
#include "constants.h"
#include "dsp.h"
#include "itpp_ext.h"

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace itpp;

#define SIGNAL_SIZE 128
#define SQRT2_INV   (0.7071067817811865475)
#define SQRT12_INV  (0.0883883476483184)

__constant__ hipfftDoubleComplex pss_td[3][256];
__constant__ hipfftDoubleComplex d_tw128[SIGNAL_SIZE];
__constant__ short d_radix2_bitreverse[SIGNAL_SIZE];
__constant__ short d_radix4_bitreverse[SIGNAL_SIZE];

hipfftDoubleComplex h_tw128[SIGNAL_SIZE];
short h_radix2_bitreverse[SIGNAL_SIZE];
short h_radix4_bitreverse[SIGNAL_SIZE];

extern "C" __device__ void kernel_fft_radix2(hipfftDoubleComplex *c_io, int N);


extern "C" void cuda_reset_device()
{
    hipDeviceReset();
}

extern "C" void copy_pss_to_device()
{
    int i, t, len;
    hipfftDoubleComplex pss[3][256];

    for (t = 0; t < 3; t++) {
        len = ROM_TABLES.pss_td[t].length();
        for (i = 0; i < len; i++) {
            pss[t][i].x = ROM_TABLES.pss_td[t][i].real();
            pss[t][i].y = ROM_TABLES.pss_td[t][i].imag();
        }
        for (; i < 256; i++) {
            pss[t][i].x = 0.0f;
            pss[t][i].y = 0.0f;
        }
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pss_td), &pss, sizeof(pss)));
}

extern "C" unsigned int reverse_bit(unsigned int n, int nbits)
{
    unsigned int reverse_num = 0;

    for (int i = 0; i < nbits; i++) {
        if (n & (1 << i))
            reverse_num |= (1 << ((nbits - 1) - i));
    }

    return reverse_num;
}

extern "C" unsigned int reverse_radix_4_and_2(unsigned int n, int nbits, int s)
{
    unsigned int reverse_num = 0;
    int i = 0;

    if (nbits & 1) {
        if (s == 0) {
            reverse_num |= ((n >> (nbits - 1)) & 1);
        } else {
            i = 1;
        }
    }

    for (; i <= nbits - 2; i += 2) {
        reverse_num |= (((n >> i) & 3) << ((nbits - 2) - i));
    }

    if (nbits & 1) {
        if (s == 1) {
            reverse_num |= ((n & 1) << (nbits - 1));
        }
    }

    return reverse_num;
}

extern "C" void generate_twiddle_factor(int N)
{
    int nbits = ceil(log(1.0 * N) / log(2.0));

    for (int n = 0; n < N; n++) {
        double theta = (HIP_PI * 2 * n) / N;
        h_tw128[n].x = cos(theta);
        h_tw128[n].y = -sin(theta);
        h_radix2_bitreverse[n] = reverse_bit(n, nbits);
        h_radix4_bitreverse[n] = reverse_radix_4_and_2(n, nbits, 1);
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_tw128), &h_tw128, sizeof(h_tw128)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_radix2_bitreverse), &h_radix2_bitreverse, sizeof(h_radix2_bitreverse)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_radix4_bitreverse), &h_radix4_bitreverse, sizeof(h_radix4_bitreverse)));
}

#define COMPLEX_MUL_REAL(a, b)  ((a).x * (b).x - (a).y * (b).y)
#define COMPLEX_MUL_IMAG(a, b)  ((a).x * (b).y + (a).y * (b).x)

__global__ void xc_correlate_kernel(hipfftDoubleComplex *d_capbuf, double *d_xc_sqr,
                                    double *d_xc_incoherent_single, double *d_xc_incoherent,
                                    unsigned int n_cap, uint8 ds_comb_arm,
                                    unsigned int t, double f, double fs)
{
    __shared__ hipfftDoubleComplex s_fshift_pss[256], s_capbuf[256 + 137];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    double k = HIP_PI * f * 2 / fs;
    double shift = k * tid;
    double x1 = cos(shift), y1 = sin(shift);
    double x2 = pss_td[t][tid].x, y2 = pss_td[t][tid].y;
    unsigned int max_m = (n_cap - 100 - 136) / 9600;
    unsigned int i, m;

    s_fshift_pss[tid].x = x1*x2 - y1*y2;
    s_fshift_pss[tid].y = -x1*y2 - x2*y1;

    s_capbuf[tid] = d_capbuf[256 * bid + tid];

    if (tid < 137) {
        if (256 * bid + 256 + tid < n_cap) {
            s_capbuf[256 + tid] = d_capbuf[256 * bid + 256 + tid];
        } else {
            s_capbuf[256 + tid] = d_capbuf[tid];
        }
    }

    __syncthreads();

    double real, imag;

    real = COMPLEX_MUL_REAL(s_fshift_pss[0], s_capbuf[tid]);
    imag = COMPLEX_MUL_IMAG(s_fshift_pss[0], s_capbuf[tid]);
    for (i = 1; i < 137; i++) {
        real += COMPLEX_MUL_REAL(s_fshift_pss[i], s_capbuf[tid + i]);
        imag += COMPLEX_MUL_IMAG(s_fshift_pss[i], s_capbuf[tid + i]);
    }
    d_xc_sqr[256 * bid + tid] = (real * real + imag * imag) / (137.0*137.0);

    __syncthreads();

    if (tid < 16) {
        unsigned int index = 16 * bid + tid;
        double xc_incoherent_single_val = d_xc_sqr[index];
        for (m = 1; m < max_m; m++) {
            unsigned int span = m * 0.005 * fs;
            xc_incoherent_single_val += d_xc_sqr[index + span];
        }
        double xc_incoherent_value = d_xc_incoherent_single[index] = xc_incoherent_single_val / max_m;

        __syncthreads();

        for (i = 1; i <= ds_comb_arm; i++) {
            if (index + i < 9600) {
                xc_incoherent_value += d_xc_incoherent_single[index + i];
            } else {
                xc_incoherent_value += d_xc_incoherent_single[index + i - 9600];
            }
            if (index > i) {
                xc_incoherent_value += d_xc_incoherent_single[index - i];
            } else {
                xc_incoherent_value += d_xc_incoherent_single[index - i + 9600];
            }
        }
        d_xc_incoherent[index] = xc_incoherent_value / (ds_comb_arm * 2 + 1);
    }

    __syncthreads();
}


__global__ void xc_incoherent_collapsed_kernel(double *d_xc_incoherent,
                                               double *d_xc_incoherent_collapsed_pow, int *d_xc_incoherent_collapsed_frq,
                                               unsigned int n_f)
{
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    double best_pow = d_xc_incoherent[(0 * 3 + tid) * 9600 + bid];
    unsigned int best_index = 0;

    for (unsigned int foi = 1; foi < n_f; foi++) {
        if (d_xc_incoherent[(foi * 3 + tid) * 9600 + bid] > best_pow) {
            best_pow = d_xc_incoherent[(foi * 3 + tid) * 9600 + bid];
            best_index = foi;
        }
    }

    d_xc_incoherent_collapsed_pow[tid * 9600 + bid] = best_pow;
    d_xc_incoherent_collapsed_frq[tid * 9600 + bid] = best_index;
}

__global__ void sp_incoherent_kernel(hipfftDoubleComplex *d_capbuf, double *d_sp_incoherent, double *d_Z_th1, unsigned int n_cap, double Z_th1_factor)
{
    __shared__ double s_sqr[512];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int n_comb_sp = (n_cap - 136 - 137) / 9600;
    unsigned int index = bid * 16 + tid;
    double value;

    if (tid < 274 + 16) {
        value = d_capbuf[index].x * d_capbuf[index].x + d_capbuf[index].y * d_capbuf[index].y;
        for (unsigned int m = 1; m < n_comb_sp; m++) {
            value += (d_capbuf[index + 9600 * m].x * d_capbuf[index + 9600 * m].x + d_capbuf[index + 9600 * m].y * d_capbuf[index + 9600 * m].y);
        }
        s_sqr[tid] = value;
    } else {
        s_sqr[tid] = 0.0f;
    }

    __syncthreads();

    if (tid < 16) {
        value = s_sqr[tid];
        for (unsigned int k = 1; k < 274; k++) {
            value += s_sqr[tid + k];
        }
        index += 137;
        if (index >= 9600)
            index -= 9600;
        d_sp_incoherent[index] = value / (274.0 * n_comb_sp);
        d_Z_th1[index] = d_sp_incoherent[index] * Z_th1_factor;
    }

    __syncthreads();
}


void xcorr_pss2(const cvec & capbuf,
                const vec & f_search_set,
                const uint8 & ds_comb_arm,
                const double & fc_requested,
                const double & fc_programmed,
                const double & fs_programmed,
                // Outputs
                mat & xc_incoherent_collapsed_pow,
                imat & xc_incoherent_collapsed_frq,
                // Following used only for debugging...
                vf3d & xc_incoherent_single,
                vf3d & xc_incoherent,
                vec & sp_incoherent,
                vcf3d & xc,
                vec & sp,
                uint16 & n_comb_xc,
                uint16 & n_comb_sp)
{
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);

    unsigned int n_cap = capbuf.length();
    unsigned int n_f = f_search_set.length();
    n_comb_xc = (n_cap - 100) / 9600;
    n_comb_sp = (n_cap - 136 - 137) / 9600;

    hipfftDoubleComplex *h_capbuf = (hipfftDoubleComplex *)NULL, *d_capbuf = (hipfftDoubleComplex *)NULL;
    double *h_f = (double *)NULL, *d_f = (double *)NULL;
    double *h_xc_sqr = (double *)NULL, *d_xc_sqr = (double *)NULL;
    double *h_xc_incoherent_single = (double *)NULL, *d_xc_incoherent_single = (double *)NULL;
    double *h_xc_incoherent = (double *)NULL, *d_xc_incoherent = (double *)NULL;
    double *h_xc_incoherent_collapsed_pow = (double *)NULL, *d_xc_incoherent_collapsed_pow = (double *)NULL;
    int *h_xc_incoherent_collapsed_frq = (int *)NULL, *d_xc_incoherent_collapsed_frq = (int *)NULL;
    double *h_sp_incoherent = (double *)NULL, *d_sp_incoherent = (double *)NULL;
    double *h_Z_th1 = (double *)NULL, *d_Z_th1 = (double *)NULL;

    h_capbuf = (hipfftDoubleComplex *)malloc(n_cap * sizeof(hipfftDoubleComplex));
    h_f = (double *)malloc(n_f * sizeof(double));
    h_xc_incoherent_single = (double *)malloc(3 * n_f * 9600 * sizeof(double));
    h_xc_incoherent = (double *)malloc(3 * n_f * 9600 * sizeof(double));
    h_xc_incoherent_collapsed_pow = (double *)malloc(3 * 9600 * sizeof(double));
    h_xc_incoherent_collapsed_frq = (int *)malloc(3 * 9600 * sizeof(int));
    h_sp_incoherent = (double *)malloc(9600 * sizeof(double));
    h_Z_th1 = (double *)malloc(9600 * sizeof(double));
    h_xc_sqr = (double *)malloc(n_cap * sizeof(double));

    checkCudaErrors(hipMalloc(&d_capbuf, n_cap * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_f, n_f * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_sqr, n_cap * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_single, 3 * n_f * 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent, 3 * n_f * 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_collapsed_pow, 3 * 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_collapsed_frq, 3 * 9600 * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_sp_incoherent, 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_Z_th1, 9600 * sizeof(double)));

    for (unsigned int i = 0; i < n_cap; i++) {
        h_capbuf[i].x = capbuf[i].real();
        h_capbuf[i].y = capbuf[i].imag();
    }

    checkCudaErrors(hipMemcpy(d_capbuf, h_capbuf, n_cap * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

    for (unsigned int i = 0; i < n_f; i++) {
        h_f[i] = HIP_PI * 2 * f_search_set[i] * fc_programmed / (fs_programmed * (fc_requested - f_search_set[i]));
    }

    checkCudaErrors(hipMemcpy(d_f, h_f, n_f * sizeof(double), hipMemcpyHostToDevice));

    /* xc_correlate, xc_combine, xc_delay_spread */
    for (unsigned int foi = 0; foi < n_f; foi++) {
        for (unsigned int t = 0; t < 3; t++) {
            xc_correlate_kernel<<<600, 256>>>(d_capbuf, d_xc_sqr,
                                              &d_xc_incoherent_single[(foi * 3 + t)*9600], &d_xc_incoherent[(foi * 3 + t)*9600],
                                              n_cap, ds_comb_arm,
                                              t, f_search_set[foi], (fc_requested - f_search_set[foi]) * fs_programmed /fc_programmed);
            checkCudaErrors(hipDeviceSynchronize());
        }
    }
    checkCudaErrors(hipDeviceSynchronize());

    /* xc_peak_freq */
    xc_incoherent_collapsed_kernel<<<9600, 3>>>(d_xc_incoherent, d_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_frq, n_f);
    checkCudaErrors(hipDeviceSynchronize());

    /* sp_est, Z_th1 */
    const uint8 thresh1_n_nines = 12;
    double R_th1 = chi2cdf_inv(1 - pow(10.0, -thresh1_n_nines), 2 * n_comb_xc * (2 * ds_comb_arm + 1));
    double rx_cutoff = (6 * 12 * 15e3 / 2 + 4*15e3) / (FS_LTE / 16 / 2);
    double Z_th1_factor = R_th1 / rx_cutoff / 137 / 2 / n_comb_xc / (2 * ds_comb_arm + 1);

    sp_incoherent_kernel<<<600, 512>>>(d_capbuf, d_sp_incoherent, d_Z_th1, n_cap, Z_th1_factor);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_xc_incoherent_single, d_xc_incoherent_single, 3 * n_f * 9600 * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_pow, 3 * 9600 * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_collapsed_frq, d_xc_incoherent_collapsed_frq, 3 * 9600 * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_sp_incoherent, d_sp_incoherent, 9600 * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_Z_th1, d_Z_th1, 9600 * sizeof(double), hipMemcpyDeviceToHost));

    /* copy data for subsequent functions */
    sp_incoherent = vec(9600);
    xc_incoherent_collapsed_pow = mat(3, 9600);
    xc_incoherent_collapsed_frq = imat(3, 9600);
    xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));

    for (unsigned int foi = 0; foi < n_f; foi++) {
        for (unsigned int t = 0; t < 3; t++) {
            for (unsigned int k = 0; k < 9600; k++) {
                xc_incoherent_single[t][k][foi] = h_xc_incoherent_single[(foi*3+t)*9600+k];
            }
        }
    }

    for (unsigned int t = 0; t < 3; t++) {
        for (unsigned int k = 0; k < 9600; k++) {
            xc_incoherent_collapsed_pow(t,k) = h_xc_incoherent_collapsed_pow[t * 9600 + k];
            xc_incoherent_collapsed_frq(t,k) = h_xc_incoherent_collapsed_frq[t * 9600 + k];
        }
    }

    for (unsigned int i = 0; i < 9600; i++) {
        sp_incoherent[i] = h_sp_incoherent[i];
    }

    free(h_capbuf);
    free(h_f);
    free(h_xc_incoherent_single);
    free(h_xc_incoherent);
    free(h_xc_incoherent_collapsed_pow);
    free(h_xc_incoherent_collapsed_frq);
    free(h_sp_incoherent);
    free(h_xc_sqr);

    checkCudaErrors(hipFree(d_capbuf));
    checkCudaErrors(hipFree(d_f));
    checkCudaErrors(hipFree(d_xc_sqr));
    checkCudaErrors(hipFree(d_xc_incoherent_single));
    checkCudaErrors(hipFree(d_xc_incoherent));
    checkCudaErrors(hipFree(d_xc_incoherent_collapsed_pow));
    checkCudaErrors(hipFree(d_xc_incoherent_collapsed_frq));
    checkCudaErrors(hipFree(d_sp_incoherent));

    gettimeofday(&tv2, NULL);
    printf("xcorr_pss2 : %ld us\n", (tv2.tv_sec-tv1.tv_sec)*1000000+(tv2.tv_usec-tv1.tv_usec));
}


/**
 * Implement 36.211 7.2.
 * Generate Pseudo-random sequence and store the result into array rather than queue.
 * Bits are stored from LSB to MSB.
 *
 * \param init_in       Initial value of pseudo-random sequence generator
 * \param seqLn         How many consecutive pseudo-random number to be generated.
 * \param initOffset    The starting position of pseudo-random number to be generated
 * \param pSeqOut       Pointer to output array of UNSG32 which holds generated pseudo-random sequence
 */
__device__ void pn_seq_lsb_to_msb(unsigned int d_init_in, unsigned int d_seq_len, unsigned int d_init_offset, unsigned int *d_pseq_out)
{
    unsigned int x1, x2, tmp_val;
    unsigned int i;

    const unsigned int m2_v1600[31] = {
        0x0099110E, 0x004C8887, 0x40264444, 0x20132222, 0x10099111, 0x4804C88F, 0x64026440, 0x32013220,
        0x19009910, 0x0C804C88, 0x06402644, 0x03201322, 0x01900991, 0x40C804CF, 0x60640260, 0x30320130,
        0x18190098, 0x0C0C804C, 0x06064026, 0x03032013, 0x4181900E, 0x20C0C807, 0x50606404, 0x28303202,
        0x14181901, 0x4A0C0C87, 0x65060644, 0x32830322, 0x19418191, 0x4CA0C0CF, 0x66506060
    };

    unsigned int init_in = d_init_in;
    unsigned int seq_len = d_seq_len;
    unsigned int init_offset = d_init_offset;
    unsigned int *pseq_out = d_pseq_out;

    /* x1 is independent of c_init,
       so it can be pre-calculated at N=1600.
       x2 depends on c_init,
       so it need to multiply M^1570 to obtain its value at N=1600 */
    x1 = 0x54D21B24;
    x2 = 0;
    for (i = 0; i < 31; i++) {
        tmp_val = init_in & m2_v1600[i];

        /* determine there are even or odd number of bits set in tmp_val */
        tmp_val ^= (tmp_val >> 16);
        tmp_val ^= (tmp_val >> 8);
        tmp_val ^= (tmp_val >> 4);
        tmp_val &= 0xF;
        tmp_val = ((0x6996 >> tmp_val) & 1);

        x2 |= (tmp_val << (31 - i));
    }

    for (i = 0; i < init_offset; i++) {
        x1 >>= 1;
        x1  |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);   /* bit0 of 0x55AA is x(0)^x(3) of [3:0] */
        x2 >>= 1;
        x2  |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);   /* bit0 of 0x55AA is x(0)^x(3) */
    }

    tmp_val = 0;
    for (i = 0; i < seq_len; i++) {
        /* store from LSB to MSB */
        tmp_val |= ((x1 ^ x2) >> (31 - (i % 32)));
        if (((i + 1) % 32) == 0) {
            *pseq_out++ = tmp_val;
            tmp_val    = 0;
        }

        x1 >>= 1;
        x1 |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);
        x2 >>= 1;
        x2 |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);
    }

    if (seq_len && (seq_len % 32))
        *pseq_out = tmp_val;
}

/**
 * Implement 36.211 7.2.
 * Generate Pseudo-random sequence and store the result into array rather than queue.
 * Bits are stored from MSB to LSB.
 *
 * \param init_in       Initial value of pseudo-random sequence generator
 * \param seqLn         How many consecutive pseudo-random number to be generated.
 * \param initOffset    The starting position of pseudo-random number to be generated
 * \param pSeqOut       Pointer to output array of unsigned int which holds generated pseudo-random sequence
 */
__device__ void pn_seq_msb_to_lsb(unsigned int d_init_in, unsigned int d_seq_len, unsigned int d_init_offset, unsigned int *d_pseq_out)
{
    unsigned int x1, x2, tmp_val;
    unsigned int i;

    const unsigned int m2_v1600[31] = {
        0x0099110E, 0x004C8887, 0x40264444, 0x20132222, 0x10099111, 0x4804C88F, 0x64026440, 0x32013220,
        0x19009910, 0x0C804C88, 0x06402644, 0x03201322, 0x01900991, 0x40C804CF, 0x60640260, 0x30320130,
        0x18190098, 0x0C0C804C, 0x06064026, 0x03032013, 0x4181900E, 0x20C0C807, 0x50606404, 0x28303202,
        0x14181901, 0x4A0C0C87, 0x65060644, 0x32830322, 0x19418191, 0x4CA0C0CF, 0x66506060
    };

    unsigned int init_in = d_init_in;
    unsigned int seq_len = d_seq_len;
    unsigned int init_offset = d_init_offset;
    unsigned int *pseq_out = d_pseq_out;

    /* x1 is independent of c_init,
       so it can be pre-calculated at N=1600.
       x2 depends on c_init,
       so it need to multiply M^1570 to obtain its value at N=1600 */
    x1 = 0x54D21B24;
    x2 = 0;
    for (i = 0; i < 31; i++) {
        tmp_val = init_in & m2_v1600[i];

        /* determine there are even or odd number of bits set in tmp_val */
        tmp_val ^= (tmp_val >> 16);
        tmp_val ^= (tmp_val >> 8);
        tmp_val ^= (tmp_val >> 4);
        tmp_val &= 0xF;
        tmp_val = ((0x6996 >> tmp_val) & 1);

        x2 |= (tmp_val << (31 - i));
    }

    for (i = 0; i < init_offset; i++) {
        x1 >>= 1;
        x1 |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);
        x2 >>= 1;
        x2 |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);
    }

    tmp_val = 0;
    for (i = 0; i < seq_len; i++) {
        /* store from MSB to LSB */
        tmp_val |= (((x1 ^ x2) & 0x80000000) >> (i % 32));
        if (((i + 1) % 32) == 0) {
            *pseq_out++ = tmp_val;
            tmp_val    = 0;
        }

        x1 >>= 1;
        x1 |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);
        x2 >>= 1;
        x2 |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);
    }

    if (seq_len && (seq_len % 32))
        *pseq_out = tmp_val;
}

__device__ void kernel_fft_radix2(hipfftDoubleComplex *c_io, int N)
{
    int n, s, l, i;
    hipfftDoubleComplex *d_tw = &d_tw128[0];

    for (n = N >> 1, l = 1; n >= 1; n >>= 1, l <<= 1) {
        for (i = 0; i < l; i++) {
            for (s = 0; s < n; s++) {
                hipfftDoubleComplex a, b, aa, bb, tw;

                a = c_io[s + n*0 + i*n*2];
                b = c_io[s + n*1 + i*n*2];
                tw = d_tw[s * l];

                aa.x = a.x + b.x;
                aa.y = a.y + b.y;

                bb.x = (a.x - b.x) * tw.x - (a.y - b.y) * tw.y;
                bb.y = (a.y - b.y) * tw.x + (a.x - b.x) * tw.y;

                c_io[s + n*0 + i*n*2] = aa;
                c_io[s + n*1 + i*n*2] = bb;
            }
        }
    }

    // bit reverse
    for (n = 0; n < N; n++) {
        hipfftDoubleComplex c;
        int idx = d_radix2_bitreverse[n];

        if (idx <= n)
            continue;

        c = c_io[idx];
        c_io[idx] = c_io[n];
        c_io[n] = c;
    }
}

__device__ double angle(float real, float imag)
{
    if (real > 0.0) {
        return atan(imag / real);
    } else if (real < 0.0) {
        if (imag >= 0.0) {
            return atan(imag / real) + HIP_PI;
        } else {
            return atan(imag / real) - HIP_PI;
        }
    } else if (imag > 0.0) {
        return HIP_PI / 2;
    } else if (imag < 0.0) {
        return -HIP_PI / 2;
    } else {
        return HIP_NAN;
    }
}

__global__ void extract_tfg_kernel(hipfftDoubleComplex *d_capbuf, hipfftDoubleComplex *d_tfg, hipfftDoubleComplex *d_rs_extracted, double *d_tfg_timestamp,
                                   unsigned short n_id_cell, int n_symb_dl, double frame_start,
                                   double fc_requested, double fc_programmed, double fs_programmed, double freq_fine,
                                   // output
                                   double *d_residual_f)
{
    __shared__ unsigned int rs_dl[20 * 3];

    const unsigned int tid = threadIdx.x;

    hipfftDoubleComplex s_capbuf[128];
    int dft_location_i;
    const double k_factor = (fc_requested - freq_fine) / fc_programmed;
    double dft_location = frame_start + ((n_symb_dl == 6) ? 32 : 10) * 16 / FS_LTE * fs_programmed * k_factor;

    if (dft_location - .01 * fs_programmed * k_factor > -0.5) {
        dft_location = dft_location - .01 * fs_programmed * k_factor;
    }

    dft_location += ((tid / n_symb_dl) * 960 + (tid % n_symb_dl) * (n_symb_dl == 6 ? 160 : 137))  * 16 / FS_LTE * fs_programmed * k_factor;
    dft_location_i = lround(dft_location);
    d_tfg_timestamp[tid] = dft_location;

    // cvec capbuf = fshift(capbuf_raw, -freq_fine, fs_programmed * k_factor);
    double k = HIP_PI * (-freq_fine) / (fs_programmed * k_factor / 2);

    for (unsigned int t = dft_location_i, i = 0; i < 128; i++, t++) {
        hipfftDoubleComplex shift;

        shift.x = cos(k * t);
        shift.y = sin(k * t);

        s_capbuf[i].x = COMPLEX_MUL_REAL(d_capbuf[t], shift);
        s_capbuf[i].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift);
    }

    // DFT of 128 points
    // cvec dft_out = dft(capbuf.mid(round_i(dft_location), 128));

    kernel_fft_radix2(s_capbuf, 128);

    __syncthreads();

    // generate random sequences for symbol 0, 1, 3/4 of 20 slots

    if (tid < 20 * 3) {
        int slot = tid / 3;
        int l = (tid % 3 == 2) ? n_symb_dl - 3 : tid % 3;
        int cinit = ((7 * (slot + 1) + l + 1) * (2 * n_id_cell + 1) << 10) + 2 * n_id_cell + (n_symb_dl == 7 ? 1 : 0);

        pn_seq_lsb_to_msb(cinit, 6 * 2 * 2, (55 - 3) * 2 * 2, &rs_dl[tid]);
    }

    //  92,  93,  94, ... , 127,  1,  2,  3, ..., 36 -> concat(dft_out.right(36), dft_out.mid(1, 36))
    //   0,   1,   2,     ,  35, 36, 37, 38,    , 71
    // -36, -35, -34, ... ,  -1,  1,  2,  3, .... 36

    // concat(dft_out.right(36), dft_out.mid(1,36));
    // exp((-J * 2 * pi * late / 128) * cn)

    double late = dft_location_i - dft_location;

    for (unsigned int i = 1; i <= 36; i++) {
        hipfftDoubleComplex coeff;
        coeff.x =  cos(2 * HIP_PI * late * i / 128);
        coeff.y = -sin(2 * HIP_PI * late * i / 128);

        d_tfg[tid * 72 + 35 + i].x = SQRT12_INV * COMPLEX_MUL_REAL(s_capbuf[i], coeff);
        d_tfg[tid * 72 + 35 + i].y = SQRT12_INV * COMPLEX_MUL_IMAG(s_capbuf[i], coeff);

        coeff.y = -coeff.y;

        d_tfg[tid * 72 + 36 - i].x = SQRT12_INV * COMPLEX_MUL_REAL(s_capbuf[128 - i], coeff);
        d_tfg[tid * 72 + 36 - i].y = SQRT12_INV * COMPLEX_MUL_IMAG(s_capbuf[128 - i], coeff);
    }

    __syncthreads();
}


__global__ void tfoec_kernel(hipfftDoubleComplex *d_capbuf, hipfftDoubleComplex *d_tfg, hipfftDoubleComplex *d_rs_extracted, double *d_tfg_timestamp,
                             unsigned short n_id_cell, int n_symb_dl, double frame_start,
                             double fc_requested, double fc_programmed, double fs_programmed, double freq_fine,
                             // output
                             double *d_residual_f)
{
    __shared__ unsigned int rs_dl[20 * 3];
    __shared__ float foe_real, foe_imag;
    __shared__ float toe_real, toe_imag;

    const unsigned int tid = threadIdx.x;
    double dft_location = d_tfg_timestamp[tid];
    double late;

    // generate random sequences for symbol 0, 1, 3/4 of 20 slots

    if (tid < 20 * 3) {
        int slot = tid / 3;
        int l = (tid % 3 == 2) ? n_symb_dl - 3 : tid % 3;
        int cinit = ((7 * (slot + 1) + l + 1) * (2 * n_id_cell + 1) << 10) + 2 * n_id_cell + (n_symb_dl == 7 ? 1 : 0);

        pn_seq_lsb_to_msb(cinit, 6 * 2 * 2, (55 - 3) * 2 * 2, &rs_dl[tid]);
    }

    foe_real = 0.0; foe_imag = 0.0;
    toe_real = 0.0; toe_imag = 0.0;
    __syncthreads();

    if (tid < 122 * 2) {
        int slot = tid / 2;
        int l = (tid & 1) ? (n_symb_dl - 3) : 0;
        int rs_bits = rs_dl[((tid / 2) % 20) * 3 + ((l == 0) ? 0 : 2)];
        int v_offset = ((n_id_cell % 6) + ((l == 0) ? 0 : 3)) % 6;

        // elem_mult(rs_extracted.get_row(t), conj(rs_dl.get_rs(mod(t, 20), sym_num)))
        for (unsigned int i = 0; i < 12; i++, rs_bits >>= 2, v_offset += 6) {
            hipfftDoubleComplex std_rs, rcvd_rs;

            // rs_symb = 1 / sqrt(2) ((1 - 2 * c(0)) + j (1 - 2 * c(1))

            std_rs.x = SQRT2_INV * (1 - ((rs_bits & 1) * 2));
            std_rs.y = SQRT2_INV * (1 - ((rs_bits & 2)));

            rcvd_rs = d_tfg[(slot * n_symb_dl + l) * 72 + v_offset];

            std_rs.y = -std_rs.y;

            d_rs_extracted[((tid & 1) * 122 + (tid / 2)) * 12 + i].x = COMPLEX_MUL_REAL(rcvd_rs, std_rs);
            d_rs_extracted[((tid & 1) * 122 + (tid / 2)) * 12 + i].y = COMPLEX_MUL_IMAG(rcvd_rs, std_rs);
        }
    }

    __syncthreads();

    if (tid < 121) {
        // CUDA 2.0+ capability support atomic addition of 32 bit floating point numbers
        // CUDA 6.0+ capability support atomic addition of 64 bit floating point numbers

        // sum(elem_mult(conj(col(0,n_slot-2)), col(1,-1)));

        float real = 0.0, imag = 0.0;
        for (unsigned int i = 0; i < 12; i++) {
            hipfftDoubleComplex rs_1, rs_2;

            rs_1 = d_rs_extracted[(0 + tid + 0) * 12 + i];
            rs_2 = d_rs_extracted[(0 + tid + 1) * 12 + i];

            rs_1.y = -rs_1.y;
            real += COMPLEX_MUL_REAL(rs_1, rs_2);
            imag += COMPLEX_MUL_IMAG(rs_1, rs_2);

            rs_1 = d_rs_extracted[(122 + tid + 0) * 12 + i];
            rs_2 = d_rs_extracted[(122 + tid + 1) * 12 + i];

            rs_1.y = -rs_1.y;
            real += COMPLEX_MUL_REAL(rs_1, rs_2);
            imag += COMPLEX_MUL_IMAG(rs_1, rs_2);
        }

        atomicAdd(&foe_real, real);
        atomicAdd(&foe_imag, imag);

        __syncthreads();
    }

    __syncthreads();

    double residual_f = angle(foe_real, foe_imag) / (2*HIP_PI) / 0.0005;
    double k_factor_residual = (fc_requested - residual_f) / fc_programmed;
    late = dft_location - k_factor_residual * dft_location;

    *d_residual_f = residual_f;

    // -36, -35, -34, ... ,  -1,  1,  2,  3, .... 36
    // exp((-J * 2 * pi * late / 128) * cn)

    // tfg.get_row(t)*exp(J*2*pi* -residual_f*tfg_comp_timestamp(t)/(FS_LTE/16))
    // elem_mult(tfg_comp.get_row(t), exp((-J*2*pi*late/128)*cn))
    for (unsigned int i = 1; i <= 36; i++) {

        hipfftDoubleComplex coeff;
        double real, imag;

        coeff.x = cos(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) - (late * i / 128)));
        coeff.y = sin(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) - (late * i / 128)));

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 35 + i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 35 + i], coeff);

        d_tfg[tid * 72 + 35 + i].x = real;
        d_tfg[tid * 72 + 35 + i].y = imag;

        coeff.x = cos(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) + (late * i / 128)));
        coeff.y = sin(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) + (late * i / 128)));

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 36 - i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 36 - i], coeff);

        d_tfg[tid * 72 + 36 - i].x = real;
        d_tfg[tid * 72 + 36 - i].y = imag;
    }

    __syncthreads();

    // Perform TOE.
    // Implemented by comparing subcarrier k of one OFDM symbol with subcarrier
    // k+3 of another OFDM symbol. This is why FOE must be performed first.
    // Slightly less performance but faster execution time could be obtained
    // by comparing subcarrier k with subcarrier k+6 of the same OFDM symbol.

    if (tid < 2 * 122 - 1) {

        int slot1 = tid / 2;
        int l1 = (tid & 1) ? (n_symb_dl - 3) : 0;
        int rs_bits1 = rs_dl[(slot1 % 20) * 3 + ((l1 == 0) ? 0 : 2)];
        int v_offset1 = ((n_id_cell % 6) + ((l1 == 0) ? 0 : 3)) % 6;

        int slot2 = (tid + 1) / 2;
        int l2 = ((tid + 1) & 1) ? (n_symb_dl - 3) : 0;
        int rs_bits2 = rs_dl[(slot2 % 20) * 3 + ((l2 == 0) ? 0 : 2)];
        int v_offset2 = ((n_id_cell % 6) + ((l2 == 0) ? 0 : 3)) % 6;

        float real, imag;

        hipfftDoubleComplex toe1, toe2;
        hipfftDoubleComplex std_rs, rcvd_rs;
        hipfftDoubleComplex r1v, r2v, r2v_prev;

        toe1.x = 0.0; toe1.y = 0.0;
        toe2.x = 0.0; toe2.y = 0.0;
        r2v_prev.x = 0.0; r2v_prev.y = 0.0;

#define SWAP(x,y) \
    do { (tmp) = (x); (x) = (y); (y) = (tmp); \
    } while(0)

        if (v_offset2 < v_offset1) {
            int tmp;

            SWAP(slot1, slot2);
            SWAP(l1, l2);
            SWAP(rs_bits1, rs_bits2);
            SWAP(v_offset1, v_offset2);
        }

        for (unsigned int i = 0; i < 12; i++, rs_bits1 >>= 2, v_offset1 += 6, rs_bits2 >>= 2, v_offset2 += 6) {

            // rs_symb = 1 / sqrt(2) ((1 - 2 * c(0)) + j (1 - 2 * c(1))

            std_rs.x = SQRT2_INV * (1 - ((rs_bits1 & 1) * 2));
            std_rs.y = SQRT2_INV * (1 - ((rs_bits1 & 2)));

            rcvd_rs = d_tfg[(slot1 * n_symb_dl + l1) * 72 + v_offset1];

            std_rs.y = -std_rs.y;

            r1v.x = COMPLEX_MUL_REAL(rcvd_rs, std_rs);
            r1v.y = -COMPLEX_MUL_IMAG(rcvd_rs, std_rs); // this r1v is actually conj(r1v)

            std_rs.x = SQRT2_INV * (1 - ((rs_bits2 & 1) * 2));
            std_rs.y = SQRT2_INV * (1 - ((rs_bits2 & 2)));

            std_rs.y = -std_rs.y;

            rcvd_rs = d_tfg[(slot2 * n_symb_dl + l2) * 72 + v_offset2];

            r2v.x = COMPLEX_MUL_REAL(rcvd_rs, std_rs);
            r2v.y = COMPLEX_MUL_IMAG(rcvd_rs, std_rs);

            // elem_mult(conj(r1v), r2v)

            toe1.x += COMPLEX_MUL_REAL(r1v, r2v);
            toe1.y += COMPLEX_MUL_IMAG(r1v, r2v);

            r1v.y = -r1v.y;
            r2v.y = -r2v.y;   // this r2v is actually conj(r2v)

            // elem_mult(conj(r2v(i-1)), r1v(i))

            toe2.x += COMPLEX_MUL_REAL(r1v, r2v_prev);
            toe2.y += COMPLEX_MUL_IMAG(r1v, r2v_prev);

            r2v_prev = r2v;
        }

        real = 1.0 * (toe1.x + toe2.x);
        imag = 1.0 * (toe1.y + toe2.y);
        atomicAdd(&toe_real, real);
        atomicAdd(&toe_imag, imag);
    }

    __syncthreads();

    // double delay = -arg(toe)/3/(2*pi/128);
    double delay = -angle(toe_real, toe_imag) / 3 / (2 * HIP_PI / 128);

    // Perform TOC
    for (unsigned int i = 1; i <= 36; i++) {

        hipfftDoubleComplex coeff;
        double real, imag;

        coeff.x = cos(2 * HIP_PI * delay * i / 128);
        coeff.y = sin(2 * HIP_PI * delay * i / 128);

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 35 + i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 35 + i], coeff);

        d_tfg[tid * 72 + 35 + i].x = real;
        d_tfg[tid * 72 + 35 + i].y = imag;

        coeff.y = -coeff.y;

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 36 - i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 36 - i], coeff);

        d_tfg[tid * 72 + 36 - i].x = real;
        d_tfg[tid * 72 + 36 - i].y = imag;
    }

    __syncthreads();
}

extern "C" Cell extract_tfg_and_tfoec(
    const Cell & cell,
    const cvec & capbuf_raw,
    const double & fc_requested,
    const double & fc_programmed,
    const double & fs_programmed,
    // Output
    cmat & my_tfg_comp)
{
    const double frame_start = cell.frame_start;
    const int n_symb_dl = cell.n_symb_dl();
    const int n_ofdm_sym = (6*10*2+2)*n_symb_dl;

    unsigned int n_cap = capbuf_raw.length();
    hipfftDoubleComplex *h_capbuf = (hipfftDoubleComplex *)NULL, *d_capbuf = (hipfftDoubleComplex *)NULL;
    hipfftDoubleComplex *h_tfg = (hipfftDoubleComplex *)NULL, *d_tfg = (hipfftDoubleComplex *)NULL;
    hipfftDoubleComplex *d_rs_extracted = (hipfftDoubleComplex *)NULL;
    double h_residual_f, *d_residual_f = (double *)NULL;
    double *h_tfg_timestamp = (double *)NULL, *d_tfg_timestamp = (double *)NULL;

    h_capbuf = (hipfftDoubleComplex *)malloc(n_cap * sizeof(hipfftDoubleComplex));
    h_tfg = (hipfftDoubleComplex *)malloc(n_ofdm_sym * 12 * 6 * sizeof(hipfftDoubleComplex));
    h_tfg_timestamp = (double *)malloc(n_ofdm_sym * sizeof(double));

    checkCudaErrors(hipMalloc(&d_capbuf, n_cap * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_tfg, n_ofdm_sym * 12 * 6 * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_rs_extracted, 2 * 122 * 12 * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_residual_f, sizeof(double)));
    checkCudaErrors(hipMalloc(&d_tfg_timestamp, n_ofdm_sym * sizeof(double)));

    for (unsigned int i = 0; i < n_cap; i++) {
        h_capbuf[i].x = capbuf_raw[i].real();
        h_capbuf[i].y = capbuf_raw[i].imag();
    }
    checkCudaErrors(hipMemcpy(d_capbuf, h_capbuf, n_cap * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

    extract_tfg_kernel<<<1, n_ofdm_sym>>>(d_capbuf, d_tfg, d_rs_extracted, d_tfg_timestamp,
                                          cell.n_id_cell(), n_symb_dl, frame_start,
                                          fc_requested, fc_programmed, fs_programmed, cell.freq_fine,
                                          // output
                                          d_residual_f);

    tfoec_kernel<<<1, n_ofdm_sym>>>(d_capbuf, d_tfg, d_rs_extracted, d_tfg_timestamp,
                                    cell.n_id_cell(), cell.n_symb_dl(), frame_start,
                                    fc_requested, fc_programmed, fs_programmed, cell.freq_fine,
                                    // output
                                    d_residual_f);

    checkCudaErrors(hipMemcpy(h_tfg, d_tfg, n_ofdm_sym * 12 * 6 * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&h_residual_f, d_residual_f, sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_tfg_timestamp, d_tfg_timestamp, n_ofdm_sym * sizeof(double), hipMemcpyDeviceToHost));

    my_tfg_comp = cmat(n_ofdm_sym, 72);
    for (int i = 0; i < n_ofdm_sym; i++) {
        for (unsigned int j = 0; j < 72; j++) {
            my_tfg_comp.set(i, j, complex<double>(h_tfg[i * 72 + j].x, h_tfg[i * 72 + j].y));
        }
    }

    free(h_capbuf);
    free(h_tfg);

    checkCudaErrors(hipFree(d_capbuf));
    checkCudaErrors(hipFree(d_tfg));
    checkCudaErrors(hipFree(d_rs_extracted));
    checkCudaErrors(hipFree(d_residual_f));
    checkCudaErrors(hipFree(d_tfg_timestamp));

    Cell cell_out(cell);
    cell_out.freq_superfine = cell_out.freq_fine + h_residual_f;
    return cell_out;
}



