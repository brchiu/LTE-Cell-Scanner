#include "hip/hip_runtime.h"
#include <itpp/base/array.h>
#include <itpp/base/converters.h>
#include <itpp/base/itassert.h>
#include <itpp/base/mat.h>
#include <itpp/base/matfunc.h>
#include <itpp/base/random.h>
#include <itpp/base/vec.h>
#include <itpp/base/math/elem_math.h>
#include <boost/math/special_functions/gamma.hpp>
#include <sys/time.h>
#include <cmath>
#include <list>
#include <iomanip>
#include <algorithm>
#include <vector>

#include "macros.h"
#include "common.h"
#include "lte_lib.h"
#include "constants.h"
#include "dsp.h"
#include "itpp_ext.h"

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace itpp;

__constant__ hipfftComplex pss_td[3][256];

#define THREAD_DIM_X   (256)

extern "C" void copy_pss_to_device()
{
    int i, t, len;
    hipfftComplex pss[3][256];
  
    for (t = 0; t < 3; t++) {
        len = ROM_TABLES.pss_td[t].length();
        for (i = 0; i < len; i++) {
            pss[t][i].x = ROM_TABLES.pss_td[t][i].real();
            pss[t][i].y = ROM_TABLES.pss_td[t][i].imag();
        }
        for (; i < 256; i++) {
            pss[t][i].x = 0.0f;
            pss[t][i].y = 0.0f;
        }
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pss_td), &pss, sizeof(pss)));
}

#define COMPLEX_MUL_REAL(a, b)  ((a).x * (b).x - (a).y * (b).y)
#define COMPLEX_MUL_IMAG(a, b)  ((a).x * (b).y + (a).y * (b).x)

__global__ void xc_correlate_kernel(hipfftComplex *d_capbuf, float *d_xc_sqr, 
                                    float *d_xc_incoherent_single, float *d_xc_incoherent,
                                    unsigned int n_cap, uint8 ds_comb_arm, 
                                    unsigned int t, double f, double fs)
{
    __shared__ hipfftComplex s_fshift_pss[256], s_capbuf[256 + 137];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    double k = HIP_PI * f * 2 / fs;
    double shift = k * tid;
    double x1 = cos(shift), y1 = sin(shift);
    double x2 = pss_td[t][tid].x, y2 = pss_td[t][tid].y;
    unsigned int max_m = (n_cap - 100 - 136) / 9600;
    unsigned int i, m;
 
    s_fshift_pss[tid].x = x1*x2 - y1*y2;
    s_fshift_pss[tid].y = -x1*y2 - x2*y1;

    s_capbuf[tid] = d_capbuf[256 * bid + tid];

    if (tid < 137) {
        if (THREAD_DIM_X * bid + THREAD_DIM_X + tid < n_cap) {
            s_capbuf[THREAD_DIM_X + tid] = d_capbuf[THREAD_DIM_X * bid + THREAD_DIM_X + tid];
        } else {
            s_capbuf[THREAD_DIM_X + tid] = d_capbuf[tid];
        }
    }
  
    __syncthreads();

    float real, imag;

    real = COMPLEX_MUL_REAL(s_fshift_pss[0], s_capbuf[tid]);
    imag = COMPLEX_MUL_IMAG(s_fshift_pss[0], s_capbuf[tid]);
    for (i = 1; i < 137; i++) {
        real += COMPLEX_MUL_REAL(s_fshift_pss[i], s_capbuf[tid + i]);
        imag += COMPLEX_MUL_IMAG(s_fshift_pss[i], s_capbuf[tid + i]);
    }
    d_xc_sqr[THREAD_DIM_X * bid + tid] = (real * real + imag * imag) / (137.0*137.0);

    __syncthreads();

    if (tid < 16) {
        unsigned int index = 16 * bid + tid;
        float xc_incoherent_single_val = d_xc_sqr[index];
        for (m = 1; m < max_m; m++) {
            unsigned int span = m * 0.005 * fs;
            xc_incoherent_single_val += d_xc_sqr[index + span];
        }
        float xc_incoherent_value = d_xc_incoherent_single[index] = xc_incoherent_single_val / max_m;

        __syncthreads();

        for (i = 1; i <= ds_comb_arm; i++) {
            if (index + i < 9600) {
                xc_incoherent_value += d_xc_incoherent_single[index + i];
            } else {
                xc_incoherent_value += d_xc_incoherent_single[index + i - 9600];
            }
            if (index > i) {
                xc_incoherent_value += d_xc_incoherent_single[index - i];
            } else {
                xc_incoherent_value += d_xc_incoherent_single[index - i + 9600];
            }
        }
        d_xc_incoherent[index] = xc_incoherent_value / (ds_comb_arm * 2 + 1);
    }

    __syncthreads();
}


__global__ void xc_incoherent_collapsed_kernel(float *d_xc_incoherent, 
                                               float *d_xc_incoherent_collapsed_pow, int *d_xc_incoherent_collapsed_frq,
                                               unsigned int n_f)
{
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    float best_pow = d_xc_incoherent[(0 * 3 + tid) * 9600 + bid];
    unsigned int best_index = 0;

    for (unsigned int foi = 1; foi < n_f; foi++) {
        if (d_xc_incoherent[(foi * 3 + tid) * 9600 + bid] > best_pow) {
            best_pow = d_xc_incoherent[(foi * 3 + tid) * 9600 + bid];
            best_index = foi;
        }
    }

    d_xc_incoherent_collapsed_pow[tid * 9600 + bid] = best_pow;
    d_xc_incoherent_collapsed_frq[tid * 9600 + bid] = best_index;
}


__global__ void sp_incoherent_kernel(hipfftComplex *d_capbuf, float *d_sp_incoherent, unsigned int n_cap)
{
    __shared__ float s_sqr[512];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int n_comb_sp = (n_cap - 136 - 137) / 9600;
    unsigned int index = bid * 16 + tid;
    float value;

    if (tid < 274 + 16) {
       value = d_capbuf[index].x * d_capbuf[index].x + d_capbuf[index].y * d_capbuf[index].y;
       for (unsigned int m = 1; m < n_comb_sp; m++) {
           value += (d_capbuf[index + 9600 * m].x * d_capbuf[index + 9600 * m].x + d_capbuf[index + 9600 * m].y * d_capbuf[index + 9600 * m].y);
       }
       s_sqr[tid] = value;
    } else {
       s_sqr[tid] = 0.0f;
    }

    __syncthreads();

    if (tid < 16) {
        value = s_sqr[tid];
        for (unsigned int k = 1; k < 274; k++) {
             value += s_sqr[tid + k];
        }
        index += 137;
        if (index >= 9600)
            index -= 9600;
        d_sp_incoherent[index] = value / (274.0 * n_comb_sp);
    }

    __syncthreads();
}

void xcorr_pss2(
                       const cvec & capbuf,
                       const vec & f_search_set,
                       const uint8 & ds_comb_arm,
                       const double & fc_requested,
                       const double & fc_programmed,
                       const double & fs_programmed,
                       // Outputs
                       mat & xc_incoherent_collapsed_pow,
                       imat & xc_incoherent_collapsed_frq,
                       // Following used only for debugging...
                       vf3d & xc_incoherent_single,
                       vf3d & xc_incoherent,
                       vec & sp_incoherent,
                       vcf3d & xc,
                       vec & sp,
                       uint16 & n_comb_xc,
                       uint16 & n_comb_sp)
{
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);

    unsigned int n_cap = capbuf.length();
    unsigned int n_f = f_search_set.length();
    n_comb_xc = (n_cap - 100) / 9600;
    n_comb_sp = (n_cap - 136 - 137) / 9600;

    hipfftComplex *h_capbuf = (hipfftComplex *)NULL, *d_capbuf = (hipfftComplex *)NULL;
    double *h_f = (double *)NULL, *d_f = (double *)NULL;
    float *h_xc_sqr = (float *)NULL, *d_xc_sqr = (float *)NULL;
    float *h_xc_incoherent_single = (float *)NULL, *d_xc_incoherent_single = (float *)NULL;
    float *h_xc_incoherent = (float *)NULL, *d_xc_incoherent = (float *)NULL;
    float *h_xc_incoherent_collapsed_pow = (float *)NULL, *d_xc_incoherent_collapsed_pow = (float *)NULL;
    int *h_xc_incoherent_collapsed_frq = (int *)NULL, *d_xc_incoherent_collapsed_frq = (int *)NULL;
    float *h_sp_incoherent = (float *)NULL, *d_sp_incoherent = (float *)NULL;
 
    h_capbuf = (hipfftComplex *)malloc(n_cap * sizeof(hipfftComplex));
    h_f = (double *)malloc(n_f * sizeof(double));
    h_xc_incoherent_single = (float *)malloc(3 * n_f * 9600 * sizeof(float));
    h_xc_incoherent = (float *)malloc(3 * n_f * 9600 * sizeof(float));
    h_xc_incoherent_collapsed_pow = (float *)malloc(3 * 9600 * sizeof(float));
    h_xc_incoherent_collapsed_frq = (int *)malloc(3 * 9600 * sizeof(int));
    h_sp_incoherent = (float *)malloc(9600 * sizeof(float));
    h_xc_sqr = (float *)malloc(n_cap * sizeof(float));
 
    checkCudaErrors(hipMalloc(&d_capbuf, n_cap * sizeof(hipfftComplex)));
    checkCudaErrors(hipMalloc(&d_f, n_f * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_sqr, n_cap * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_single, 3 * n_f * 9600 * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent, 3 * n_f * 9600 * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_collapsed_pow, 3 * 9600 * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_collapsed_frq, 3 * 9600 * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_sp_incoherent, 9600 * sizeof(float)));

    for (unsigned int i = 0; i < n_cap; i++) {
        h_capbuf[i].x = capbuf[i].real();
        h_capbuf[i].y = capbuf[i].imag();
    }

    for (unsigned int i = 0; i < n_f; i++) {
        h_f[i] = HIP_PI * 2 * f_search_set[i] * fc_programmed / (fs_programmed * (fc_requested - f_search_set[i]));
    }

    checkCudaErrors(hipMemcpy(d_capbuf, h_capbuf, n_cap * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_f, h_f, n_f * sizeof(double), hipMemcpyHostToDevice));

    for (unsigned int foi = 0; foi < n_f; foi++) {
        for (unsigned int t = 0; t < 3; t++) {
            xc_correlate_kernel<<<600, 256>>>(d_capbuf, d_xc_sqr, 
                                              &d_xc_incoherent_single[(foi * 3 + t)*9600], &d_xc_incoherent[(foi * 3 + t)*9600],
                                              n_cap, ds_comb_arm, 
                                              t, f_search_set[foi], (fc_requested - f_search_set[foi]) * fs_programmed /fc_programmed);
            checkCudaErrors(hipDeviceSynchronize());
        }
    }
    checkCudaErrors(hipDeviceSynchronize());

    xc_incoherent_collapsed_kernel<<<9600, 3>>>(d_xc_incoherent, d_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_frq, n_f);
    checkCudaErrors(hipDeviceSynchronize());

    sp_incoherent_kernel<<<600, 512>>>(d_capbuf, d_sp_incoherent, n_cap);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_xc_incoherent_single, d_xc_incoherent_single, 3 * n_f * 9600 * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_pow, 3 * 9600 * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_collapsed_frq, d_xc_incoherent_collapsed_frq, 3 * 9600 * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_sp_incoherent, d_sp_incoherent, 9600 * sizeof(float), hipMemcpyDeviceToHost));

    sp_incoherent = vec(9600);
    xc_incoherent_collapsed_pow = mat(3, 9600);
    xc_incoherent_collapsed_frq = imat(3, 9600);
    xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));

    for (unsigned int foi = 0; foi < n_f; foi++) {
        for (unsigned int t = 0; t < 3; t++) {
            for (unsigned int k = 0; k < 9600; k++) {
                xc_incoherent_single[t][k][foi] = h_xc_incoherent_single[(foi*3+t)*9600+k];
            }
        }
    }

    for (unsigned int t = 0; t < 3; t++) {
        for (unsigned int k = 0; k < 9600; k++) {
            xc_incoherent_collapsed_pow(t,k) = h_xc_incoherent_collapsed_pow[t * 9600 + k];
            xc_incoherent_collapsed_frq(t,k) = h_xc_incoherent_collapsed_frq[t * 9600 + k];
        }
    }

    for (unsigned int i = 0; i < 9600; i++) {
        sp_incoherent[i] = h_sp_incoherent[i];
    }

    free(h_capbuf);
    free(h_f);
    free(h_xc_incoherent_single);
    free(h_xc_incoherent);
    free(h_xc_incoherent_collapsed_pow);
    free(h_xc_incoherent_collapsed_frq);
    free(h_sp_incoherent);
    free(h_xc_sqr);

    checkCudaErrors(hipFree(d_capbuf));
    checkCudaErrors(hipFree(d_f));
    checkCudaErrors(hipFree(d_xc_sqr));
    checkCudaErrors(hipFree(d_xc_incoherent_single));
    checkCudaErrors(hipFree(d_xc_incoherent));
    checkCudaErrors(hipFree(d_xc_incoherent_collapsed_pow));
    checkCudaErrors(hipFree(d_xc_incoherent_collapsed_frq));
    checkCudaErrors(hipFree(d_sp_incoherent));

    gettimeofday(&tv2, NULL);
    printf("xcorr_pss2 : %ld us\n", (tv2.tv_sec-tv1.tv_sec)*1000000+(tv2.tv_usec-tv1.tv_usec));
}


void xcorr_pss_orig(
  // Inputs
  const cvec & capbuf,
  const vec & f_search_set,
  const uint8 & ds_comb_arm,
  const double & fc_requested,
  const double & fc_programmed,
  const double & fs_programmed,
  // Outputs
  mat & xc_incoherent_collapsed_pow,
  imat & xc_incoherent_collapsed_frq,
  // Following used only for debugging...
  vf3d & xc_incoherent_single,
  vf3d & xc_incoherent,
  vec & sp_incoherent,
  vcf3d & xc,
  vec & sp,
  uint16 & n_comb_xc,
  uint16 & n_comb_sp
) 
{
  // Perform correlations
  const uint32 n_cap=length(capbuf);
  const uint16 n_f=length(f_search_set);

  // Set aside space for the vector and initialize with NAN's.
#ifndef NDEBUG
  xc = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f,NAN)));
  vcf3d xc2 = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f,NAN)));
  xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f,NAN)));
  xc_incoherent = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f,NAN)));
#else
  xc = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f)));
  vcf3d xc2 = vector < vector < vector < complex < float > > > > (3,vector< vector < complex < float > > >(n_cap-136, vector < complex < float > > (n_f)));
  xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));
  xc_incoherent = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));
#endif
  vec xc_sqr = vec(n_cap-136);

  // Local variables declared outside of the loop.
  double f_off;
  cvec temp;
  complex <double> acc;
  uint16 foi;
  uint8 t;
  uint32 k;
  uint8 m;

  struct timeval tv1, tv2;
  gettimeofday(&tv1, NULL);

  // Loop and perform correlations.
  // Incoherently combine correlations
  n_comb_xc = floor_i((xc[0].size()-100)/9600);
  const uint8 ds_com_arm_weight = (2*ds_comb_arm+1);
  printf("n_cap=%d\n", n_cap);

#if 0
  sp_incoherent = vec(9600);
  xc_incoherent_collapsed_pow = mat(3,9600);
  xc_incoherent_collapsed_frq = imat(3,9600);

  xc_correlate_step(capbuf,
                    f_search_set,
                    ds_comb_arm,
                    fc_requested,
                    fc_programmed,
                    fs_programmed,
                    // Outputs
                    xc_incoherent_collapsed_pow,
                    xc_incoherent_collapsed_frq,
                    // Following used only for debugging...
                    xc_incoherent_single,
                    xc_incoherent,
                    sp_incoherent,
                    xc,
                    sp,
                    n_comb_xc,
                    n_comb_sp);
#else
  for (foi=0;foi<n_f;foi++) {
    f_off = f_search_set(foi);
    const double k_factor = (fc_requested-f_off)/fc_programmed;
    for (t = 0; t < 3;t++) {
      temp = ROM_TABLES.pss_td[t];
      temp = fshift(temp,f_off,fs_programmed*k_factor);
      temp = conj(temp)/137;
#ifdef _OPENMP
#pragma omp parallel for shared(temp,capbuf,xc) private(k,acc,m)
#endif
      for (k=0;k<n_cap-136;k++) {
        acc=0;
        for (m=0;m<137;m++) {
          // Correlations are performed at the 2x rate which effectively
          // performs filtering and correlating at the same time. Thus,
          // this algorithm can handle huge frequency offsets limited only
          // by the bandwidth of the capture device.
          // Correlations can also be done at the 1x rate if filtering is
          // peformed first, but this will limit the set of frequency offsets
          // that this algorithm can detect. 1x rate correlations will,
          // however, be nearly twice as fast as the 2x correlations
          // performed here.
          acc+=temp(m)*capbuf(k+m);
        }
        xc[t][k][foi]=acc;
      }

      for (uint16 idx=0;idx<9600;idx++) {
        // Because of the large supported frequency offsets and the large
        // amount of time represented by the capture buffer, the length
        // in samples, of a frame varies by the frequency offset.
        //double actual_time_offset=m*.005*k_factor;
        //double actual_start_index=itpp::round_i(actual_time_offset*FS_LTE/16);
        xc_incoherent_single[t][idx][foi] = 0;
        for (uint16 m = 0; m < n_comb_xc; m++) {
          uint32 actual_start_index = itpp::round_i(m*.005*k_factor*fs_programmed);
          xc_incoherent_single[t][idx][foi] += xc[t][idx + actual_start_index][foi].real() * xc[t][idx + actual_start_index][foi].real() + 
                                               xc[t][idx + actual_start_index][foi].imag() * xc[t][idx + actual_start_index][foi].imag();
        }
        xc_incoherent_single[t][idx][foi]/= n_comb_xc;
      }

      for (uint16 idx=0;idx<9600;idx++) {
        xc_incoherent[t][idx][foi] = xc_incoherent_single[t][idx][foi];
        for (uint8 k=1;k<=ds_comb_arm;k++) {
          xc_incoherent[t][idx][foi] += (xc_incoherent_single[t][itpp_ext::matlab_mod(idx-k,9600)][foi] + xc_incoherent_single[t][itpp_ext::matlab_mod(idx+k,9600)][foi]);
        }
        xc_incoherent[t][idx][foi] /= ds_com_arm_weight;
      }
    }
  }
#endif

  // Estimate received signal power
  // const uint32 n_cap=length(capbuf);

#if 0
  n_comb_sp = floor_i((n_cap-136-137)/9600);
  const uint32 n_sp = n_comb_sp*9600;

  // Set aside space for the vector and initialize with NAN's.
  sp = vec(n_sp);
#ifndef NDEBUG
  sp = NAN;
#endif
  sp[0] = 0;
  // Estimate power for first time offset
  for (uint16 t=0;t<274;t++) {
    sp[0] += pow(capbuf[t].real(),2) + pow(capbuf[t].imag(),2);
  }
  sp[0] = sp[0] / 274;
  // Estimate RX power for remaining time offsets.
  for (uint32 t=1;t<n_sp;t++) {
    sp[t] = sp[t-1] + (-pow(capbuf[t-1].real(),2)-pow(capbuf[t-1].imag(),2)+pow(capbuf[t+274-1].real(),2)+pow(capbuf[t+274-1].imag(),2))/274;
  }

  // Combine incoherently
  sp_incoherent = sp.left(9600);
  for (uint16 t=1; t < n_comb_sp;t++) {
    sp_incoherent += sp.mid(t*9600, 9600);
  }
  sp_incoherent = sp_incoherent / n_comb_sp;

  // Shift to the right by 137 samples to align with the correlation peaks.
  tshift(sp_incoherent, 137);

  // Search for peaks among all the frequency offsets.
  // const int n_f=xc_incoherent[0][0].size();

  xc_incoherent_collapsed_pow = mat(3,9600);
  xc_incoherent_collapsed_frq = imat(3,9600);
#ifndef NDEBUG
  xc_incoherent_collapsed_pow = NAN;
  xc_incoherent_collapsed_frq = -1;
#endif
  for (uint8 t=0;t<3;t++) {
    for (uint16 k=0;k<9600;k++) {
      double best_pow=xc_incoherent[t][k][0];
      uint16 best_idx=0;
      for (uint16 foi=1;foi<n_f;foi++) {
        if (xc_incoherent[t][k][foi]>best_pow) {
          best_pow=xc_incoherent[t][k][foi];
          best_idx=foi;
        }
      }
      xc_incoherent_collapsed_pow(t,k)=best_pow;
      xc_incoherent_collapsed_frq(t,k)=best_idx;
    }
  }
#endif

  gettimeofday(&tv2, NULL);
  printf("xcorr_pss2 : %ld us\n", (tv2.tv_sec-tv1.tv_sec)*1000000+(tv2.tv_usec-tv1.tv_usec));
}



