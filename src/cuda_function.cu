#include "hip/hip_runtime.h"
#include <itpp/base/array.h>
#include <itpp/base/converters.h>
#include <itpp/base/itassert.h>
#include <itpp/base/mat.h>
#include <itpp/base/matfunc.h>
#include <itpp/base/random.h>
#include <itpp/base/vec.h>
#include <itpp/base/math/elem_math.h>
#include <boost/math/special_functions/gamma.hpp>
#include <sys/time.h>
#include <cmath>
#include <list>
#include <iomanip>
#include <algorithm>
#include <vector>

#include "macros.h"
#include "common.h"
#include "lte_lib.h"
#include "constants.h"
#include "dsp.h"
#include "itpp_ext.h"

#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace itpp;

#define SIGNAL_SIZE 128
#define SQRT2_INV               (0.7071067817811865475)
#define SQRT62_INV              (0.1270001270001905)
#define SQRT128_INV             (0.0883883476483184)
#define COMPLEX_MUL_REAL(a, b)  ((a).x * (b).x - (a).y * (b).y)
#define COMPLEX_MUL_IMAG(a, b)  ((a).x * (b).y + (a).y * (b).x)

#undef SWAP
#define SWAP(x,y)               do { typeof (x) __tmp = (x); (x) = (y); (y) = (__tmp); } while (0)

__constant__ hipfftDoubleComplex pss_fd[3][62];
__constant__ hipfftDoubleComplex pss_td[3][256];
__constant__ unsigned int sss_fd[168][3][2][2];

__constant__ hipfftDoubleComplex d_tw128[SIGNAL_SIZE];
__constant__ short d_radix2_bitreverse[SIGNAL_SIZE];
__constant__ short d_radix4_bitreverse[SIGNAL_SIZE];

hipfftDoubleComplex h_pss_fd[3][62];
hipfftDoubleComplex h_pss_td[3][256];
unsigned int h_sss_fd[168][3][2][2];

hipfftDoubleComplex h_tw128[SIGNAL_SIZE];
short h_radix2_bitreverse[SIGNAL_SIZE];
short h_radix4_bitreverse[SIGNAL_SIZE];

__device__ void kernel_fft_radix2(hipfftDoubleComplex *c_io, int N);

/*
 *  Wrapper of hipDeviceReset()
 */
extern "C" void cuda_reset_device()
{
    hipDeviceReset();
}


/*
 *  Copy constant data to CUDA device.
 *
 *  To-Do : use CUDA to generate PSS and SSS.
 */
extern "C" void cuda_copy_constant_data_to_device()
{
    int i, j, k, t, len;

    for (t = 0; t < 3; t++) {
        len = ROM_TABLES.pss_td[t].length();
        for (i = 0; i < len; i++) {
            h_pss_td[t][i].x = ROM_TABLES.pss_td[t][i].real();
            h_pss_td[t][i].y = ROM_TABLES.pss_td[t][i].imag();
        }
        for (; i < 256; i++) {
            h_pss_td[t][i].x = 0.0f;
            h_pss_td[t][i].y = 0.0f;
        }
    }

    // PSS frequency domain data

    for (t = 0; t < 3; t++) {
        for (i = 0; i < 62; i++) {
            h_pss_fd[t][i].x = ROM_TABLES.pss_fd[t][i].real();
            h_pss_fd[t][i].y = ROM_TABLES.pss_fd[t][i].imag();
        }
    }

    for (t = 0; t < 168; t++) {
        for (i = 0; i < 3; i++) {
            for (j = 0; j < 2; j++) {
                unsigned int word1 = 0, word2 = 0;
                for (k = 0; k < 31; k++) {
                    if (ROM_TABLES.sss_fd(t,i,j)[k] == -1)
                        word1 |= (1 << k);
                    if (ROM_TABLES.sss_fd(t,i,j)[k+31] == -1)
                        word2 |= (1 << k);
                }
                h_sss_fd[t][i][j][0] = word1;
                h_sss_fd[t][i][j][1] = word2;
            }
        }
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pss_td), h_pss_td, sizeof(h_pss_td)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pss_fd), h_pss_fd, sizeof(h_pss_fd)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sss_fd), h_sss_fd, sizeof(h_sss_fd)));
}

/*
 * Return bit-reverse of number n in nbits bits.
 */
extern "C" unsigned int reverse_bit(unsigned int n, int nbits)
{
    unsigned int reverse_num = 0;

    for (int i = 0; i < nbits; i++) {
        if (n & (1 << i))
            reverse_num |= (1 << ((nbits - 1) - i));
    }

    return reverse_num;
}

/*
 * Returns radix-4/2 reverse of number n in nbits bits.
 * s denotes the radix-2 stage is in the last one (1) or the first one (0).
 *
 * Example : s=1, assuming 4442 -> 2444
 *           s=0, assuming 2444 -> 4442
 */
extern "C" unsigned int reverse_radix_4_and_2(unsigned int n, int nbits, int s)
{
    unsigned int reverse_num = 0;
    int i = 0;

    if (nbits & 1) {
        if (s == 0) {
            reverse_num |= ((n >> (nbits - 1)) & 1);
        } else {
            i = 1;
        }
    }

    for (; i <= nbits - 2; i += 2) {
        reverse_num |= (((n >> i) & 3) << ((nbits - 2) - i));
    }

    if (nbits & 1) {
        if (s == 1) {
            reverse_num |= ((n & 1) << (nbits - 1));
        }
    }

    return reverse_num;
}


/*
 * Generate twiddle factor of length N and copy them to CUDA device.
 * Generate bit reverse table of length N and copy them to CUDA device.
 *
 * Now support data of length N less or equal to 128.
 */
extern "C" void generate_twiddle_factor(int N)
{
    int nbits = ceil(log(1.0 * N) / log(2.0));

    for (int n = 0; n < N; n++) {
        double theta = (HIP_PI * 2 * n) / N;
        h_tw128[n].x = cos(theta);
        h_tw128[n].y = -sin(theta);
        h_radix2_bitreverse[n] = reverse_bit(n, nbits);
        h_radix4_bitreverse[n] = reverse_radix_4_and_2(n, nbits, 1);
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_tw128), &h_tw128, sizeof(h_tw128)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_radix2_bitreverse), &h_radix2_bitreverse, sizeof(h_radix2_bitreverse)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_radix4_bitreverse), &h_radix4_bitreverse, sizeof(h_radix4_bitreverse)));
}



/*
 * Calculate angle of complex number with real and imag.
 */
__device__ double angle(float real, float imag)
{
    if (real > 0.0) {
        return atan(imag / real);
    } else if (real < 0.0) {
        if (imag >= 0.0) {
            return atan(imag / real) + HIP_PI;
        } else {
            return atan(imag / real) - HIP_PI;
        }
    } else if (imag > 0.0) {
        return HIP_PI / 2;
    } else if (imag < 0.0) {
        return -HIP_PI / 2;
    } else {
        return HIP_NAN;
    }
}



/*
 *  Step 1 of xc_correlate()
 */
__global__ void xc_correlate_step1_kernel(hipfftDoubleComplex *d_capbuf, double *d_xc_sqr,
                                          const unsigned int n_cap, const unsigned int t,
                                          const double f_off, const double fc_requested, const double fc_programmed, const double fs_programmed)
{
    __shared__ hipfftDoubleComplex s_fshift_pss[256], s_capbuf[256 + 137];

    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;

    const double k_factor = (fc_requested - f_off)/fc_programmed;
    const double k = HIP_PI * f_off / (fs_programmed * k_factor / 2);
    double shift = k * tid;
    double x1 = cos(shift), y1 = sin(shift);
    double x2 = pss_td[t][tid].x, y2 = pss_td[t][tid].y;
    double real, imag;

    s_fshift_pss[tid].x = x1*x2 - y1*y2;
    s_fshift_pss[tid].y = -x1*y2 - x2*y1;

    s_capbuf[tid] = d_capbuf[256 * bid + tid];

    if (tid < 137) {
        if (256 * bid + 256 + tid < n_cap) {
            s_capbuf[256 + tid] = d_capbuf[256 * bid + 256 + tid];
        } else {
            s_capbuf[256 + tid] = d_capbuf[tid];
        }
    }

    __syncthreads();

    real = COMPLEX_MUL_REAL(s_fshift_pss[0], s_capbuf[tid]);
    imag = COMPLEX_MUL_IMAG(s_fshift_pss[0], s_capbuf[tid]);
    for (unsigned int i = 1; i < 137; i++) {
        real += COMPLEX_MUL_REAL(s_fshift_pss[i], s_capbuf[tid + i]);
        imag += COMPLEX_MUL_IMAG(s_fshift_pss[i], s_capbuf[tid + i]);
    }
    d_xc_sqr[256 * bid + tid] = (real * real + imag * imag) / (137.0*137.0);
}



/*
 * Step 2 of xc_correlate()
 */
__global__ void xc_correlate_step2_kernel(double *d_xc_sqr, double *d_xc_incoherent_single, const unsigned int n_cap,
                                          const double f_off, const double fc_requested, const double fc_programmed, const double fs_programmed)
{
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const double k_factor = (fc_requested - f_off)/fc_programmed;
    const unsigned int index = 16 * bid + tid;
    const unsigned int max_m = (n_cap - 100 - 136) / 9600;
    double xc_incoherent_single_val;

    xc_incoherent_single_val = d_xc_sqr[index];
    for (unsigned int m = 1; m < max_m; m++) {
        unsigned int span = lround(m * 0.005 * k_factor * fs_programmed);
        xc_incoherent_single_val += d_xc_sqr[index + span];
    }
    d_xc_incoherent_single[index] = xc_incoherent_single_val / max_m;
}



/*
 *  Step 3 of xc_correlate()
 */
__global__ void xc_correlate_step3_kernel(double *d_xc_incoherent_single, double *d_xc_incoherent, uint8 ds_comb_arm)
{
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int index = 16 * bid + tid;
    double xc_incoherent_value = 0.0;

    if (index < ds_comb_arm) {
       for (unsigned int i = 0; i <= index + ds_comb_arm; i++) {
           xc_incoherent_value += d_xc_incoherent_single[i];
       }
       for (unsigned int i = 9600 + index - ds_comb_arm; i < 9600; i++) {
           xc_incoherent_value += d_xc_incoherent_single[i];
       }
    } else if (index < 9600 - ds_comb_arm) {
       for (unsigned int i = index - ds_comb_arm; i <= index + ds_comb_arm; i++) {
           xc_incoherent_value += d_xc_incoherent_single[i];
       }
    } else {
       for (unsigned int i = index - ds_comb_arm; i < 9600; i++) {
           xc_incoherent_value += d_xc_incoherent_single[i];
       }
       for (unsigned int i = 0; i <= index + ds_comb_arm - 9600; i++) {
           xc_incoherent_value += d_xc_incoherent_single[i];
       }
    }

    d_xc_incoherent[index] = xc_incoherent_value / (ds_comb_arm * 2 + 1);
}



/*
 *
 */
__global__ void xc_incoherent_collapsed_kernel(double *d_xc_incoherent,
                                               double *d_xc_incoherent_collapsed_pow, int *d_xc_incoherent_collapsed_frq,
                                               unsigned int n_f)
{
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    double best_pow = d_xc_incoherent[(0 * 3 + tid) * 9600 + bid];
    unsigned int best_index = 0;

    for (unsigned int foi = 1; foi < n_f; foi++) {
        if (d_xc_incoherent[(foi * 3 + tid) * 9600 + bid] > best_pow) {
            best_pow = d_xc_incoherent[(foi * 3 + tid) * 9600 + bid];
            best_index = foi;
        }
    }

    d_xc_incoherent_collapsed_pow[tid * 9600 + bid] = best_pow;
    d_xc_incoherent_collapsed_frq[tid * 9600 + bid] = best_index;
}



/*
 *
 */
__global__ void sp_incoherent_kernel(hipfftDoubleComplex *d_capbuf, double *d_sp_incoherent, double *d_Z_th1, unsigned int n_cap, double Z_th1_factor)
{
    __shared__ double s_sqr[512];

    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int n_comb_sp = (n_cap - 136 - 137) / 9600;
    unsigned int index = bid * 16 + tid;
    double value;

    if (tid < 274 + 16) {
        value = d_capbuf[index].x * d_capbuf[index].x + d_capbuf[index].y * d_capbuf[index].y;
        for (unsigned int m = 1; m < n_comb_sp; m++) {
            value += (d_capbuf[index + 9600 * m].x * d_capbuf[index + 9600 * m].x + d_capbuf[index + 9600 * m].y * d_capbuf[index + 9600 * m].y);
        }
        s_sqr[tid] = value;
    } else {
        s_sqr[tid] = 0.0;
    }

    __syncthreads();

    if (tid < 16) {
        value = s_sqr[tid];
        for (unsigned int k = 1; k < 274; k++) {
            value += s_sqr[tid + k];
        }
        index += 137;
        if (index >= 9600)
            index -= 9600;
        d_sp_incoherent[index] = value / (274.0 * n_comb_sp);
        d_Z_th1[index] = d_sp_incoherent[index] * Z_th1_factor;
    }
}



/*
 *
 */
__global__ void peak_search_kernel(double *d_xc_incoherent_collapsed_pow, int *d_xc_incoherent_collapsed_frq, double *d_f_search_set,
                                   double *d_Z_th1, double *d_xc_incoherent_single, short *d_aux, int ds_comb_arm)
{
    __shared__ unsigned int finished;
    __shared__ double thresh1, thresh2, peak_pow;
    __shared__ short peak_pos, peak_ind, peak_n_id_2;

    const int tid = threadIdx.x;
    short pos, pos_ind, pos_n_id_2;
    double pos_pow;

    for (unsigned int i = tid; i < 9600 * 3; i += 1024) {
        d_aux[i] = i;
    }

    __syncthreads();

    do {
        for (unsigned int k = 0; k < 9600 * 3; k += 2048) {
            for (unsigned int s = 1024; s > 0; s >>= 1) {
                if ((tid < s) && (k + tid + s < 9600 * 3)) {
                    int pos1 = d_aux[k + tid];
                    int pos2 = d_aux[k + tid + s];

                    if (d_xc_incoherent_collapsed_pow[pos1] < d_xc_incoherent_collapsed_pow[pos2]) {
                        d_aux[k + tid] = pos2;
                        d_aux[k + tid + s] = pos1;
                    }
                }
            }
            __syncthreads();
        }

        for (unsigned int s = 8; s > 0; s >>= 1) {
            if ((tid < s) && ((tid + s) * 2048 < 9600 * 3)) {
                int pos1 = d_aux[tid * 2048];
                int pos2 = d_aux[(tid + s) * 2048];

                if (d_xc_incoherent_collapsed_pow[pos1] < d_xc_incoherent_collapsed_pow[pos2]) {
                    d_aux[tid * 2048] = pos2;
                    d_aux[(tid + s) * 2048] = pos1;
                }
            }
            __syncthreads();
        }

        if (tid == 0) {
            peak_pos = d_aux[0];
            peak_pow = d_xc_incoherent_collapsed_pow[peak_pos];

            if (peak_pow < d_Z_th1[peak_ind]) {
                finished = 1;
            } else {

                finished = 0;
                peak_n_id_2 = peak_pos / 9600;
                peak_ind = peak_pos - peak_n_id_2 * 9600;

                int freq_idx = d_xc_incoherent_collapsed_frq[peak_pos];
                double freq = d_f_search_set[freq_idx];
                double best_pow = -HIP_INF;
                short best_ind = -1;
                int t = peak_ind - ds_comb_arm;

                if (t < 0) t += 9600;

                for (unsigned int i = 0; i < 2 * ds_comb_arm + 1; i++) {
                    if (d_xc_incoherent_single[(freq_idx * 3 + peak_n_id_2) * 9600 + t] > best_pow) {
                        best_ind = t;
                        best_pow = d_xc_incoherent_single[(freq_idx * 3 + peak_n_id_2) * 9600 + t];
                    }
                    t++;
                    if (t >= 9600) t-= 9600;
                }

                thresh1 = peak_pow * 0.1584893192461113; // udb10(-8.0) = pow(10.0,-8.0/10.0);
                thresh2 = peak_pow * 0.0630957344480193; // udb10(-12.0) = pow(10.0,-12.0/10.0);
            }
        }

        __syncthreads();

        if (!finished) {

            for (unsigned int i = tid; i < 3 * 9600; i += 1024) {

                pos = d_aux[i];

                pos_n_id_2 = pos / 9600;
                pos_ind = pos - pos_n_id_2 * 9600;
                pos_pow = d_xc_incoherent_collapsed_pow[pos];

                if (pos_pow < thresh2) {
                    d_xc_incoherent_collapsed_pow[pos] = 0.0;
                }

                /* 9600 - 274 <= pos_ind - peak_ind + 9600 <= 9600 + 274 */

                if (((9600 - 274) <= (pos_ind - peak_ind + 9600)) && 
                    ((pos_ind - peak_ind + 9600) <= (9600 + 274))) {

                    if (peak_n_id_2 == pos_n_id_2) {
                       d_xc_incoherent_collapsed_pow[pos] = 0.0;
                    } else if (pos_pow < thresh1) {
                       d_xc_incoherent_collapsed_pow[pos] = 0.0;
                    }
                }
            }
        }

        __syncthreads();

    } while (!finished);
}



/*
 *
 */
void xcorr_pss2(const cvec & capbuf,
                const vec & f_search_set,
                const uint8 & ds_comb_arm,
                const double & fc_requested,
                const double & fc_programmed,
                const double & fs_programmed,
                // Outputs
                mat & xc_incoherent_collapsed_pow,
                imat & xc_incoherent_collapsed_frq,
                // Following used only for debugging...
                vf3d & xc_incoherent_single,
                vf3d & xc_incoherent,
                vec & sp_incoherent,
                vcf3d & xc,
                vec & sp,
                uint16 & n_comb_xc,
                uint16 & n_comb_sp)
{
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);

    unsigned int n_cap = capbuf.length();
    unsigned int n_f = f_search_set.length();
    n_comb_xc = (n_cap - 100) / 9600;
    n_comb_sp = (n_cap - 136 - 137) / 9600;

    hipfftDoubleComplex *h_capbuf = (hipfftDoubleComplex *)NULL, *d_capbuf = (hipfftDoubleComplex *)NULL;
    double *h_f = (double *)NULL, *d_f = (double *)NULL;
    double *h_f_search_set = (double *)NULL, *d_f_search_set = (double *)NULL;
    double *h_xc_sqr = (double *)NULL, *d_xc_sqr = (double *)NULL;
    double *h_xc_incoherent_single = (double *)NULL, *d_xc_incoherent_single = (double *)NULL;
    double *h_xc_incoherent = (double *)NULL, *d_xc_incoherent = (double *)NULL;
    double *h_xc_incoherent_collapsed_pow = (double *)NULL, *d_xc_incoherent_collapsed_pow = (double *)NULL;
    int *h_xc_incoherent_collapsed_frq = (int *)NULL, *d_xc_incoherent_collapsed_frq = (int *)NULL;
    double *h_sp_incoherent = (double *)NULL, *d_sp_incoherent = (double *)NULL;
    double *h_Z_th1 = (double *)NULL, *d_Z_th1 = (double *)NULL;
    short *d_aux = (short *)NULL;

    h_capbuf = (hipfftDoubleComplex *)malloc(n_cap * sizeof(hipfftDoubleComplex));
    h_f = (double *)malloc(n_f * sizeof(double));
    h_f_search_set = (double *)malloc(n_f * sizeof(double));
    h_xc_incoherent_single = (double *)malloc(3 * n_f * 9600 * sizeof(double));
    h_xc_incoherent = (double *)malloc(3 * n_f * 9600 * sizeof(double));
    h_xc_incoherent_collapsed_pow = (double *)malloc(3 * 9600 * sizeof(double));
    h_xc_incoherent_collapsed_frq = (int *)malloc(3 * 9600 * sizeof(int));
    h_sp_incoherent = (double *)malloc(9600 * sizeof(double));
    h_Z_th1 = (double *)malloc(9600 * sizeof(double));
    h_xc_sqr = (double *)malloc(n_cap * sizeof(double));

    checkCudaErrors(hipMalloc(&d_capbuf, n_cap * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_f, n_f * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_f_search_set, n_f * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_sqr, n_cap * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_single, 3 * n_f * 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent, 3 * n_f * 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_collapsed_pow, 3 * 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_xc_incoherent_collapsed_frq, 3 * 9600 * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_sp_incoherent, 9600 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_Z_th1, 9600 * sizeof(double)));

    checkCudaErrors(hipMalloc(&d_aux, 9600 * 3 * sizeof(short)));

    for (unsigned int i = 0; i < n_cap; i++) {
        h_capbuf[i].x = capbuf[i].real();
        h_capbuf[i].y = capbuf[i].imag();
    }

    checkCudaErrors(hipMemcpy(d_capbuf, h_capbuf, n_cap * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

    for (unsigned int i = 0; i < n_f; i++) {
        h_f[i] = HIP_PI * 2 * f_search_set[i] * fc_programmed / (fs_programmed * (fc_requested - f_search_set[i]));
        h_f_search_set[i] = f_search_set[i];
    }

    checkCudaErrors(hipMemcpy(d_f, h_f, n_f * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_f_search_set, h_f_search_set, n_f * sizeof(double), hipMemcpyHostToDevice));

    /* xc_correlate, xc_combine, xc_delay_spread */
    for (unsigned int foi = 0; foi < n_f; foi++) {
        for (unsigned int t = 0; t < 3; t++) {

            xc_correlate_step1_kernel<<<600, 256>>>(d_capbuf, d_xc_sqr, n_cap, t,
                                                    f_search_set[foi], fc_requested, fc_programmed, fs_programmed);
            checkCudaErrors(hipDeviceSynchronize());

            xc_correlate_step2_kernel<<<600, 16>>>(d_xc_sqr, &d_xc_incoherent_single[(foi * 3 + t)*9600], n_cap,
                                                   f_search_set[foi], fc_requested, fc_programmed, fs_programmed);
            checkCudaErrors(hipDeviceSynchronize());

            xc_correlate_step3_kernel<<<600, 16>>>(&d_xc_incoherent_single[(foi * 3 + t)*9600], &d_xc_incoherent[(foi * 3 + t)*9600],
                                                   ds_comb_arm);
            checkCudaErrors(hipDeviceSynchronize());
        }
    }
    checkCudaErrors(hipDeviceSynchronize());

    /* xc_peak_freq */
    xc_incoherent_collapsed_kernel<<<9600, 3>>>(d_xc_incoherent, d_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_frq, n_f);
    checkCudaErrors(hipDeviceSynchronize());

    /* sp_est, Z_th1 */
    const uint8 thresh1_n_nines = 12;
    double R_th1 = chi2cdf_inv(1 - pow(10.0, -thresh1_n_nines), 2 * n_comb_xc * (2 * ds_comb_arm + 1));
    double rx_cutoff = (6 * 12 * 15e3 / 2 + 4*15e3) / (FS_LTE / 16 / 2);
    double Z_th1_factor = R_th1 / rx_cutoff / 137 / 2 / n_comb_xc / (2 * ds_comb_arm + 1);

    sp_incoherent_kernel<<<600, 512>>>(d_capbuf, d_sp_incoherent, d_Z_th1, n_cap, Z_th1_factor);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_xc_incoherent_single, d_xc_incoherent_single, 3 * n_f * 9600 * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_pow, 3 * 9600 * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_xc_incoherent_collapsed_frq, d_xc_incoherent_collapsed_frq, 3 * 9600 * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_sp_incoherent, d_sp_incoherent, 9600 * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_Z_th1, d_Z_th1, 9600 * sizeof(double), hipMemcpyDeviceToHost));

    peak_search_kernel<<<1, 1024>>>(d_xc_incoherent_collapsed_pow, d_xc_incoherent_collapsed_frq, d_f_search_set,
                                    d_Z_th1, d_xc_incoherent_single, d_aux, ds_comb_arm);
    checkCudaErrors(hipDeviceSynchronize());

    /* copy data for subsequent functions */
    sp_incoherent = vec(9600);
    xc_incoherent_collapsed_pow = mat(3, 9600);
    xc_incoherent_collapsed_frq = imat(3, 9600);
    xc_incoherent_single = vector < vector < vector < float > > > (3,vector< vector < float > >(9600, vector < float > (n_f)));

    for (unsigned int foi = 0; foi < n_f; foi++) {
        for (unsigned int t = 0; t < 3; t++) {
            for (unsigned int k = 0; k < 9600; k++) {
                xc_incoherent_single[t][k][foi] = h_xc_incoherent_single[(foi*3+t)*9600+k];
            }
        }
    }

    for (unsigned int t = 0; t < 3; t++) {
        for (unsigned int k = 0; k < 9600; k++) {
            xc_incoherent_collapsed_pow(t,k) = h_xc_incoherent_collapsed_pow[t * 9600 + k];
            xc_incoherent_collapsed_frq(t,k) = h_xc_incoherent_collapsed_frq[t * 9600 + k];
        }
    }

    for (unsigned int i = 0; i < 9600; i++) {
        sp_incoherent[i] = h_sp_incoherent[i];
    }

    free(h_capbuf);
    free(h_f);
    free(h_f_search_set);
    free(h_xc_incoherent_single);
    free(h_xc_incoherent);
    free(h_xc_incoherent_collapsed_pow);
    free(h_xc_incoherent_collapsed_frq);
    free(h_sp_incoherent);
    free(h_xc_sqr);

    checkCudaErrors(hipFree(d_capbuf));
    checkCudaErrors(hipFree(d_f));
    checkCudaErrors(hipFree(d_f_search_set));
    checkCudaErrors(hipFree(d_xc_sqr));
    checkCudaErrors(hipFree(d_xc_incoherent_single));
    checkCudaErrors(hipFree(d_xc_incoherent));
    checkCudaErrors(hipFree(d_xc_incoherent_collapsed_pow));
    checkCudaErrors(hipFree(d_xc_incoherent_collapsed_frq));
    checkCudaErrors(hipFree(d_sp_incoherent));

    checkCudaErrors(hipFree(d_aux));
}



/*
 *  Step 1 of sss_detect_getce_sss()
 */
__global__ void sss_detect_getce_sss_multiblocks_step1_kernel(hipfftDoubleComplex *d_capbuf, int n_pss,
                                                              unsigned short n_id_2_est, double peak_loc,
                                                              double fc_requested, double fc_programmed, double fs_programmed, double peak_freq,
                                                              // output
                                                              hipfftDoubleComplex *d_h_sm, double *d_pss_np, hipfftDoubleComplex *d_sss_nrm_raw, hipfftDoubleComplex *d_sss_ext_raw)
{
    __shared__ hipfftDoubleComplex s_pss_dft[128], s_nrm_sss_dft[128], s_ext_sss_dft[128];
    __shared__ hipfftDoubleComplex h_raw[62], *p_pss_fd;
    __shared__ float pss_np;

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;
    const unsigned int output_offset = bid * 62;
    const double k_factor = (fc_requested - peak_freq) / fc_programmed;
    const unsigned int pss_dft_location = lround(peak_loc + bid * k_factor * 9600 + 9 - 2);
    const unsigned int nrm_sss_dft_location = pss_dft_location - 128 - 9;
    const unsigned int ext_sss_dft_location = pss_dft_location - 128 - 32;
    hipfftDoubleComplex shift, acc;
    double noise_r, noise_i;

    double k = HIP_PI * (-peak_freq) / ((fs_programmed * k_factor) / 2);

    if (tid == 0) {
        p_pss_fd = (hipfftDoubleComplex *)&pss_fd[n_id_2_est];
        pss_np = 0.0;
    }

    shift.x = cos(k * tid);
    shift.y = sin(k * tid);

    // implement extract_psss(capbuf.mid(pss_dft_location, 128), -cell_in.freq, k_factor, fs_programmed)
    if (tid < 2) {
        s_pss_dft[126 + tid].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location + tid], shift);
        s_pss_dft[126 + tid].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location + tid], shift);
        s_nrm_sss_dft[126 + tid].x = COMPLEX_MUL_REAL(d_capbuf[nrm_sss_dft_location + tid], shift);
        s_nrm_sss_dft[126 + tid].y = COMPLEX_MUL_IMAG(d_capbuf[nrm_sss_dft_location + tid], shift);
        s_ext_sss_dft[126 + tid].x = COMPLEX_MUL_REAL(d_capbuf[ext_sss_dft_location + tid], shift);
        s_ext_sss_dft[126 + tid].y = COMPLEX_MUL_IMAG(d_capbuf[ext_sss_dft_location + tid], shift);
    } else if (tid < 128) {
        s_pss_dft[tid - 2].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location + tid], shift);
        s_pss_dft[tid - 2].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location + tid], shift);
        s_nrm_sss_dft[tid - 2].x = COMPLEX_MUL_REAL(d_capbuf[nrm_sss_dft_location + tid], shift);
        s_nrm_sss_dft[tid - 2].y = COMPLEX_MUL_IMAG(d_capbuf[nrm_sss_dft_location + tid], shift);
        s_ext_sss_dft[tid - 2].x = COMPLEX_MUL_REAL(d_capbuf[ext_sss_dft_location + tid], shift);
        s_ext_sss_dft[tid - 2].y = COMPLEX_MUL_IMAG(d_capbuf[ext_sss_dft_location + tid], shift);
    }

    __syncthreads();

    if (tid == 0) {
        kernel_fft_radix2(s_pss_dft, 128);
    } else if (tid == 32) {
        kernel_fft_radix2(s_nrm_sss_dft, 128);
    } else if (tid == 64) {
        kernel_fft_radix2(s_ext_sss_dft, 128);
    }

    __syncthreads();

    // index   :  0,  1,       30, 31, 32, 33,    , 61
    // dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    // pss_fd  :  0,  1, ....  30, 31, 32, 33, ..., 61

    if (tid < 31) {
        h_raw[tid].x = SQRT128_INV * (s_pss_dft[tid + 97].x * p_pss_fd[tid].x + s_pss_dft[tid + 97].y * p_pss_fd[tid].y);
        h_raw[tid].y = SQRT128_INV * (s_pss_dft[tid + 97].y * p_pss_fd[tid].x - s_pss_dft[tid + 97].x * p_pss_fd[tid].y);
    } else if (tid < 62) {
        // concat(dft_out.right(31), dft_out.mid(1,31)) * conj(pss_fd)
        h_raw[tid].x = SQRT128_INV * (s_pss_dft[tid - 30].x * p_pss_fd[tid].x + s_pss_dft[tid - 30].y * p_pss_fd[tid].y);
        h_raw[tid].y = SQRT128_INV * (s_pss_dft[tid - 30].y * p_pss_fd[tid].x - s_pss_dft[tid - 30].x * p_pss_fd[tid].y);
    }

    __syncthreads();

    // Smoothing... Basic...

    //                  t  = 0, 1, 2, 3,  4,  5,  6,  7,  8,  9, 10,        , 53, 54, 55, 56, 57, 58, 59, 60, 61
    // lt  = MAX(0, t - 6) = 0, 0, 0, 0,  0,  0,  0,  1,  2,  3,  4, ...... , 47, 48, 49, 50, 51, 52, 53, 54, 55
    // rt  = MIN(61,t + 6) = 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, ...... , 59, 60, 61, 61, 61, 61, 61, 61, 61
    // len = rt-lt + 1     = 7, 8, 9,10, 11, 12, 13, 13, 13, 13, 13, ...... , 13, 13, 13, 12, 11, 10,  9,  8,  7

    acc.x = 0.0; acc.y = 0.0;
    if (tid < 6) {
        for (unsigned int i = 0; i <= tid + 6; i++) {
            acc.x += h_raw[i].x;
            acc.y += h_raw[i].y;
        }
        acc.x /= (tid + 7);
        acc.y /= (tid + 7);
    } else if (tid < 56) {
        for (unsigned int i = tid - 6; i <= tid + 6; i++) {
            acc.x += h_raw[i].x;
            acc.y += h_raw[i].y;
        }
        acc.x /= 13;
        acc.y /= 13;
    } else if (tid < 62) {
        for (unsigned int i = tid - 6; i <= 61; i++) {
            acc.x += h_raw[i].x;
            acc.y += h_raw[i].y;
        }
        acc.x /= (68 - tid);
        acc.y /= (68 - tid);
    }

    // Estimate noise power.
    // pss_np = sigpower(h_sm.get_row(tid) - h_raw_fo_pss.get_row(tid));

    if (tid < 62) {
        d_h_sm[output_offset + tid].x = acc.x;
        d_h_sm[output_offset + tid].y = acc.y;

        noise_r = acc.x - h_raw[tid].x;
        noise_i = acc.y - h_raw[tid].y;
        atomicAdd(&pss_np, (float)(noise_r * noise_r + noise_i * noise_i));
    }

    __syncthreads();

    if (tid == 0) {
        d_pss_np[bid] = pss_np / 62;
    }

    __syncthreads();

    // implment : extract_psss(capbuf.mid(ext_sss_dft_location,128),-peak_freq,k_factor,fs_programmed)

    //  index   :  0,  1,       30, 31, 32, 33,    , 61
    //  dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31

    if (tid < 31) {
        d_sss_nrm_raw[output_offset + tid].x = SQRT128_INV * s_nrm_sss_dft[tid + 97].x;
        d_sss_nrm_raw[output_offset + tid].y = SQRT128_INV * s_nrm_sss_dft[tid + 97].y; 
        d_sss_ext_raw[output_offset + tid].x = SQRT128_INV * s_ext_sss_dft[tid + 97].x;
        d_sss_ext_raw[output_offset + tid].y = SQRT128_INV * s_ext_sss_dft[tid + 97].y; 
    } else if (tid < 62) {
        d_sss_nrm_raw[output_offset + tid].x = SQRT128_INV * s_nrm_sss_dft[tid - 30].x;
        d_sss_nrm_raw[output_offset + tid].y = SQRT128_INV * s_nrm_sss_dft[tid - 30].y;
        d_sss_ext_raw[output_offset + tid].x = SQRT128_INV * s_ext_sss_dft[tid - 30].x;
        d_sss_ext_raw[output_offset + tid].y = SQRT128_INV * s_ext_sss_dft[tid - 30].y;
    }
}



/*
 *  Step 2 of sss_detect_getce_sss()
 */
__global__ void sss_detect_getce_sss_multiblocks_step2_kernel(int n_pss,
                                                              hipfftDoubleComplex *d_h_sm, double *d_pss_np, hipfftDoubleComplex *d_sss_nrm_raw, hipfftDoubleComplex *d_sss_ext_raw,
                                                              // output
                                                              float *d_sss_h12_np_est, hipfftDoubleComplex *d_sss_h12_nrm_est, hipfftDoubleComplex *d_sss_h12_ext_est)
{
    __shared__ hipfftDoubleComplex h_sm[50], sss_nrm_raw[50], sss_ext_raw[50];
    __shared__ double pss_np_inv[50];
    __shared__ double sss_np_est[2];
    __shared__ float sss_h12_nrm_est_real[2], sss_h12_nrm_est_imag[2];
    __shared__ float sss_h12_ext_est_real[2], sss_h12_ext_est_imag[2];
    __shared__ float sum_of_sqr_sm_pss_np_inv[2];

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;

    if (tid > 50) {
        printf("Caution : number of thread is larger than reserved space !\n");
        return;
    }

    pss_np_inv[tid] = 1.0 / d_pss_np[tid];
    h_sm[tid] = d_h_sm[tid * 62 + bid];
    sss_nrm_raw[tid] = d_sss_nrm_raw[tid * 62 + bid];
    sss_ext_raw[tid] = d_sss_ext_raw[tid * 62 + bid];

    if (tid < 2) {
        sss_h12_nrm_est_real[tid] = 0.0;
        sss_h12_nrm_est_imag[tid] = 0.0;
        sss_h12_ext_est_real[tid] = 0.0;
        sss_h12_ext_est_imag[tid] = 0.0;
        sum_of_sqr_sm_pss_np_inv[tid] = 0.0;
    }

    __syncthreads();

    //  tid = 0 ... (n_pss-1)

    //  vec pss_np_inv_h1 = 1.0 / pss_np(itpp_ext::matlab_range(0, 2, n_pss - 1));
    //  vec pss_np_inv_h2 = 1.0 / pss_np(itpp_ext::matlab_range(1, 2, n_pss - 1));
    //
    //  for (uint8 t = 0; t < 62; t++) {
    //
    //      // First half (h1) and second half (h2) channel estimates.
    // 
    //      cvec h_sm_h1 = h_sm.get_col(t).get(itpp_ext::matlab_range(0, 2, n_pss - 1));
    //      cvec h_sm_h2 = h_sm.get_col(t).get(itpp_ext::matlab_range(1, 2, n_pss - 1));
    //
    //      sum(elem_mult(sqr(h_sm_h1), pss_np_inv_h1))
    //      sum(elem_mult(sqr(h_sm_h2), pss_np_inv_h2))
    //
    //      ....
    //  }

    double sqr_sm_pss_np_inv = (h_sm[tid].x * h_sm[tid].x + h_sm[tid].y * h_sm[tid].y) * pss_np_inv[tid];

    atomicAdd(&sum_of_sqr_sm_pss_np_inv[tid & 1], (float)(sqr_sm_pss_np_inv));

    __syncthreads();

    //  Implement the following expressions :
    //
    //  sss_h1_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h1), pss_np_inv_h1)));
    //  sss_h2_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h2), pss_np_inv_h2)));

    if (tid < 2) {
        d_sss_h12_np_est[bid + tid * 62] = sss_np_est[tid] = 1.0 / (1 + sum_of_sqr_sm_pss_np_inv[tid]);
    }

    __syncthreads();

    //  index   :  0,  1,       30, 31, 32, 33,    , 61
    //  dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    //
    //  vec pss_np_inv_h1 = 1.0 / pss_np(itpp_ext::matlab_range(0,2,n_pss-1));
    //  vec pss_np_inv_h2 = 1.0 / pss_np(itpp_ext::matlab_range(1,2,n_pss-1));
    //
    //  for (uint8 t = 0; t < 62; t++) {
    //
    //      sss_h1_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h1), pss_np_inv_h1)));
    //      sss_h2_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h2), pss_np_inv_h2)));
    //
    //      ...
    //      sss_h1_ext_est(t) = sss_h1_np_est(t) * sum(elem_mult(conj(h_sm_h1), to_cvec(pss_np_inv_h1), sss_ext_raw.get_col(t).get(itpp_ext::matlab_range(0,2,n_pss-1))));
    //      sss_h2_ext_est(t) = sss_h2_np_est(t) * sum(elem_mult(conj(h_sm_h2), to_cvec(pss_np_inv_h2), sss_ext_raw.get_col(t).get(itpp_ext::matlab_range(1,2,n_pss-1))));
    //  }

    double nrm_real, nrm_imag, ext_real, ext_imag;

    nrm_real = nrm_imag = ext_real = ext_imag = sss_np_est[tid & 1] * pss_np_inv[tid];

    nrm_real *= (h_sm[tid].x * sss_nrm_raw[tid].x + h_sm[tid].y * sss_nrm_raw[tid].y);
    nrm_imag *= (h_sm[tid].x * sss_nrm_raw[tid].y - h_sm[tid].y * sss_nrm_raw[tid].x);
    ext_real *= (h_sm[tid].x * sss_ext_raw[tid].x + h_sm[tid].y * sss_ext_raw[tid].y);
    ext_imag *= (h_sm[tid].x * sss_ext_raw[tid].y - h_sm[tid].y * sss_ext_raw[tid].x);

    atomicAdd(&sss_h12_nrm_est_real[tid & 1], (float)nrm_real);
    atomicAdd(&sss_h12_nrm_est_imag[tid & 1], (float)nrm_imag);
    atomicAdd(&sss_h12_ext_est_real[tid & 1], (float)ext_real);
    atomicAdd(&sss_h12_ext_est_imag[tid & 1], (float)ext_imag);

    __syncthreads(); 

    if (tid < 2) {
        d_sss_h12_nrm_est[bid + tid * 62].x = sss_h12_nrm_est_real[tid];
        d_sss_h12_nrm_est[bid + tid * 62].y = sss_h12_nrm_est_imag[tid];
 
        d_sss_h12_ext_est[bid + tid * 62].x = sss_h12_ext_est_real[tid];
        d_sss_h12_ext_est[bid + tid * 62].y = sss_h12_ext_est_imag[tid];
    }
}



/*
 *  Use one thread-block to implement function of sss_detect_getce_sss()
 */
__global__ void sss_detect_getce_sss_singleblock_kernel(hipfftDoubleComplex *d_capbuf, int n_pss,
                                                        unsigned short n_id_2_est, int n_symb_dl, double peak_loc,
                                                        double fc_requested, double fc_programmed, double fs_programmed, double peak_freq,
                                                        // output
                                                        float *d_sss_h12_np_est, hipfftDoubleComplex *d_sss_h12_nrm_est, hipfftDoubleComplex *d_sss_h12_ext_est)
{
    __shared__ hipfftDoubleComplex shift[128], *p_pss_fd;
    __shared__ hipfftComplex sss_h12_nrm_est[62 * 2], sss_h12_ext_est[62 * 2];
    __shared__ float sss_h12_np_est[62 * 2];

    const unsigned int tid = threadIdx.x;
    const double k_factor = (fc_requested - peak_freq) / fc_programmed;
    const unsigned int pss_dft_location = lround(peak_loc + tid * k_factor * 9600 + 9 - 2);
    hipfftDoubleComplex s_capbuf[128], h_raw[62], h_sm[62], acc;
    double pss_np;
    float real, imag;

    double k = HIP_PI * (-peak_freq) / ((fs_programmed * k_factor) / 2);

    if (tid == 0) {
        p_pss_fd = (hipfftDoubleComplex *)&pss_fd[n_id_2_est];
    }

    for (unsigned int i = tid; i < 128; i += n_pss) {
        shift[i].x = cos(k * i);
        shift[i].y = sin(k * i);
    }

    for (unsigned int i = tid; i < 62 * 2; i += n_pss) {
        sss_h12_np_est[i] = 0.0;
        sss_h12_nrm_est[i].x = 0.0; sss_h12_nrm_est[i].y = 0.0;
        sss_h12_ext_est[i].x = 0.0; sss_h12_ext_est[i].y = 0.0;
    }

    __syncthreads();

    // implement extract_psss(capbuf.mid(pss_dft_location, 128), -cell_in.freq, k_factor, fs_programmed)
    for (unsigned int t = pss_dft_location + 2, i = 2; i < 128; i++, t++) {
        /* 128 points data shift, so k multiply from 0 to 125 */

        s_capbuf[i - 2].x = COMPLEX_MUL_REAL(d_capbuf[t], shift[i]);
        s_capbuf[i - 2].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift[i]);
    }

    // time shift 2 points
    s_capbuf[126].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location], shift[0]);
    s_capbuf[126].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location], shift[0]);
    s_capbuf[127].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location + 1], shift[1]);
    s_capbuf[127].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location + 1], shift[1]);

    kernel_fft_radix2(s_capbuf, 128);

    // index   :  0,  1,       30, 31, 32, 33,    , 61
    // dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    // pss_fd  :  0,  1, ....  30, 31, 32, 33, ..., 61

    for (unsigned int i = 1; i <= 31; i++) {
        // concat(dft_out.right(31), dft_out.mid(1,31)) * conj(pss_fd)

        h_raw[i + 30].x = SQRT128_INV * (s_capbuf[i].x * p_pss_fd[i + 30].x + s_capbuf[i].y * p_pss_fd[i + 30].y);
        h_raw[i + 30].y = SQRT128_INV * (s_capbuf[i].y * p_pss_fd[i + 30].x - s_capbuf[i].x * p_pss_fd[i + 30].y);

        h_raw[i - 1].x = SQRT128_INV * (s_capbuf[i + 96].x * p_pss_fd[i - 1].x + s_capbuf[i + 96].y * p_pss_fd[i - 1].y);
        h_raw[i - 1].y = SQRT128_INV * (s_capbuf[i + 96].y * p_pss_fd[i - 1].x - s_capbuf[i + 96].x * p_pss_fd[i - 1].y);
    }

    // Smoothing... Basic...

    //                  t  = 0, 1, 2, 3,  4,  5,  6,  7,  8,  9, 10,        , 53, 54, 55, 56, 57, 58, 59, 60, 61
    // lt  = MAX(0, t - 6) = 0, 0, 0, 0,  0,  0,  0,  1,  2,  3,  4, ...... , 47, 48, 49, 50, 51, 52, 53, 54, 55
    // rt  = MIN(61,t + 6) = 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, ...... , 59, 60, 61, 61, 61, 61, 61, 61, 61
    // len = rt-lt + 1     = 7, 8, 9,10, 11, 12, 13, 13, 13, 13, 13, ...... , 13, 13, 13, 12, 11, 10,  9,  8,  7

    /* To be optimized in future */

    acc.x = 0.0; acc.y = 0.0;
    for (unsigned int i = 0; i < 6; i++) {
        acc.x += h_raw[i].x;
        acc.y += h_raw[i].y;
    }

    for (unsigned int i = 0; i <= 6; i++) {
        acc.x += h_raw[i + 6].x;
        acc.y += h_raw[i + 6].y;

        h_sm[i].x = acc.x / (i + 7);
        h_sm[i].y = acc.y / (i + 7);
    }

    for (unsigned int i = 7; i <= 55; i++) {
        acc.x += (h_raw[i + 6].x - h_raw[i - 7].x);
        acc.y += (h_raw[i + 6].y - h_raw[i - 7].y);

        h_sm[i].x = acc.x / 13;
        h_sm[i].y = acc.y / 13;
    }

    for (unsigned int i = 56; i < 62; i++) {
        acc.x -= h_raw[i - 7].x;
        acc.y -= h_raw[i - 7].y;

        h_sm[i].x = acc.x / (61 + 7 - i);
        h_sm[i].y = acc.y / (61 + 7 - i);
    }

    // Estimate noise power.
    // pss_np = sigpower(h_sm.get_row(tid) - h_raw_fo_pss.get_row(tid));
    pss_np = 0.0;
    for (unsigned int i = 0; i < 62; i++) {
        double noise_r = h_sm[i].x - h_raw[i].x;
        double noise_i = h_sm[i].y - h_raw[i].y;
        pss_np += (noise_r * noise_r + noise_i * noise_i);
    }

    pss_np = pss_np / 62.0;

    __syncthreads();

    //  vec pss_np_inv_h1 = 1.0 / pss_np(itpp_ext::matlab_range(0, 2, n_pss - 1));
    //  vec pss_np_inv_h2 = 1.0 / pss_np(itpp_ext::matlab_range(1, 2, n_pss - 1));
    //
    //  for (uint8 t = 0; t < 62; t++) {
    //
    //      // First half (h1) and second half (h2) channel estimates.
    // 
    //      cvec h_sm_h1 = h_sm.get_col(t).get(itpp_ext::matlab_range(0, 2, n_pss - 1));
    //      cvec h_sm_h2 = h_sm.get_col(t).get(itpp_ext::matlab_range(1, 2, n_pss - 1));
    //
    //      sum(elem_mult(sqr(h_sm_h1), pss_np_inv_h1))
    //      sum(elem_mult(sqr(h_sm_h2), pss_np_inv_h2))
    //
    //      ....
    //  }

    for (unsigned int i = 0, col = (tid & 1) * 62; i < 62; i++, col++) {
         float sss_np_est_value;

         sss_np_est_value = ((h_sm[i].x * h_sm[i].x) + (h_sm[i].y * h_sm[i].y)) / pss_np;
         atomicAdd(&sss_h12_np_est[col], sss_np_est_value);
    }

    __syncthreads();

    //  Implement the following expressions :
    //
    //  sss_h1_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h1), pss_np_inv_h1)));
    //  sss_h2_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h2), pss_np_inv_h2)));

    for (unsigned int i = tid; i < 62 * 2; i += n_pss) {
        sss_h12_np_est[i] = 1.0 / (1.0 + sss_h12_np_est[i]);
    }

    __syncthreads();

    // implment : extract_psss(capbuf.mid(ext_sss_dft_location,128),-peak_freq,k_factor,fs_programmed)

    //  index   :  0,  1,       30, 31, 32, 33,    , 61
    //  dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31

    const unsigned int ext_sss_dft_location = pss_dft_location - 128 - 32;
    for (unsigned int t = ext_sss_dft_location + 2, i = 2; i < 128; i++, t++) {
        /* 128 points data shift, so k multiply from 0 to 125 */

        s_capbuf[i - 2].x = COMPLEX_MUL_REAL(d_capbuf[t], shift[i]);
        s_capbuf[i - 2].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift[i]);
    }

    // time shift 2 points
    s_capbuf[126].x = COMPLEX_MUL_REAL(d_capbuf[ext_sss_dft_location], shift[0]);
    s_capbuf[126].y = COMPLEX_MUL_IMAG(d_capbuf[ext_sss_dft_location], shift[0]);
    s_capbuf[127].x = COMPLEX_MUL_REAL(d_capbuf[ext_sss_dft_location + 1], shift[1]);
    s_capbuf[127].y = COMPLEX_MUL_IMAG(d_capbuf[ext_sss_dft_location + 1], shift[1]);

    kernel_fft_radix2(s_capbuf, 128);

    //  index   :  0,  1,       30, 31, 32, 33,    , 61
    //  dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    //
    //  vec pss_np_inv_h1 = 1.0 / pss_np(itpp_ext::matlab_range(0,2,n_pss-1));
    //  vec pss_np_inv_h2 = 1.0 / pss_np(itpp_ext::matlab_range(1,2,n_pss-1));
    //
    //  for (uint8 t = 0; t < 62; t++) {
    //
    //      sss_h1_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h1), pss_np_inv_h1)));
    //      sss_h2_np_est(t) = 1 / (1 + sum(elem_mult(sqr(h_sm_h2), pss_np_inv_h2)));
    //
    //      ...
    //      sss_h1_ext_est(t) = sss_h1_np_est(t) * sum(elem_mult(conj(h_sm_h1), to_cvec(pss_np_inv_h1), sss_ext_raw.get_col(t).get(itpp_ext::matlab_range(0,2,n_pss-1))));
    //      sss_h2_ext_est(t) = sss_h2_np_est(t) * sum(elem_mult(conj(h_sm_h2), to_cvec(pss_np_inv_h2), sss_ext_raw.get_col(t).get(itpp_ext::matlab_range(1,2,n_pss-1))));
    //  }

    const double pss_np_inv_SQRT128_INV = (1 / pss_np) * SQRT128_INV;

    for (unsigned int i = 0, col = (tid & 1) * 62; i < 31; i++, col++) {

         real = imag = sss_h12_np_est[col] * pss_np_inv_SQRT128_INV;

         real *= (h_sm[i].x * s_capbuf[97 + i].x + h_sm[i].y * s_capbuf[97 + i].y);
         imag *= (h_sm[i].x * s_capbuf[97 + i].y - h_sm[i].y * s_capbuf[97 + i].x);

         atomicAdd(&sss_h12_ext_est[col].x, real);
         atomicAdd(&sss_h12_ext_est[col].y, imag);

         real = imag = sss_h12_np_est[col + 31] * pss_np_inv_SQRT128_INV;

         real *= (h_sm[i + 31].x * s_capbuf[1 + i].x + h_sm[i + 31].y * s_capbuf[1 + i].y);
         imag *= (h_sm[i + 31].x * s_capbuf[1 + i].y - h_sm[i + 31].y * s_capbuf[1 + i].x);

         atomicAdd(&sss_h12_ext_est[col + 31].x, real);
         atomicAdd(&sss_h12_ext_est[col + 31].y, imag);
    }

    __syncthreads();

    // implment : extract_psss(capbuf.mid(nrm_sss_dft_location,128),-peak_freq,k_factor,fs_programmed)

    const unsigned int nrm_sss_dft_location = pss_dft_location - 128 - 9;
    for (unsigned int t = nrm_sss_dft_location + 2, i = 2; i < 128; i++, t++) {
        s_capbuf[i - 2].x = COMPLEX_MUL_REAL(d_capbuf[t], shift[i]);
        s_capbuf[i - 2].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift[i]);
    }

    // time shift 2 points
    s_capbuf[126].x = COMPLEX_MUL_REAL(d_capbuf[nrm_sss_dft_location], shift[0]);
    s_capbuf[126].y = COMPLEX_MUL_IMAG(d_capbuf[nrm_sss_dft_location], shift[0]);
    s_capbuf[127].x = COMPLEX_MUL_REAL(d_capbuf[nrm_sss_dft_location + 1], shift[1]);
    s_capbuf[127].y = COMPLEX_MUL_IMAG(d_capbuf[nrm_sss_dft_location + 1], shift[1]);

    kernel_fft_radix2(s_capbuf, 128);

    // sss_h1_nrm_est(t) = sss_h1_np_est(t)*sum(elem_mult(conj(h_sm_h1), to_cvec(pss_np_inv_h1), sss_nrm_raw.get_col(t).get(itpp_ext::matlab_range(0,2,n_pss-1))));
    // sss_h2_nrm_est(t) = sss_h2_np_est(t)*sum(elem_mult(conj(h_sm_h2), to_cvec(pss_np_inv_h2), sss_nrm_raw.get_col(t).get(itpp_ext::matlab_range(1,2,n_pss-1))));

    for (unsigned int i = 0, col = (tid & 1) * 62; i < 31; i++, col++) {

         real = imag = sss_h12_np_est[col] * pss_np_inv_SQRT128_INV;

         real *= (h_sm[i].x * s_capbuf[97 + i].x + h_sm[i].y * s_capbuf[97 + i].y);
         imag *= (h_sm[i].x * s_capbuf[97 + i].y - h_sm[i].y * s_capbuf[97 + i].x);

         atomicAdd(&sss_h12_nrm_est[col].x, real);
         atomicAdd(&sss_h12_nrm_est[col].y, imag);

         real = imag = sss_h12_np_est[col + 31] * pss_np_inv_SQRT128_INV;

         real *= (h_sm[i + 31].x * s_capbuf[1 + i].x + h_sm[i + 31].y * s_capbuf[1 + i].y);
         imag *= (h_sm[i + 31].x * s_capbuf[1 + i].y - h_sm[i + 31].y * s_capbuf[1 + i].x);

         atomicAdd(&sss_h12_nrm_est[col + 31].x, real);
         atomicAdd(&sss_h12_nrm_est[col + 31].y, imag);
    }

    /* perhaps in the future, we can combine sss_detect_ml_kernel() into this one */
    for (unsigned int i = tid; i < 62 * 2; i += n_pss) {
        d_sss_h12_np_est[i]  = sss_h12_np_est[i];
        d_sss_h12_ext_est[i].x = sss_h12_ext_est[i].x;
        d_sss_h12_ext_est[i].y = sss_h12_ext_est[i].y;
    }

    __syncthreads();

    for (unsigned int i = tid; i < 62 * 2; i += n_pss) {
        d_sss_h12_nrm_est[i].x = sss_h12_nrm_est[i].x;
        d_sss_h12_nrm_est[i].y = sss_h12_nrm_est[i].y;
    }
}



/*
 *
 */
__device__ void sss_detect_ml_helper_function(float *sss_np_est, hipfftDoubleComplex *sss_est, int *sss_try_orig,
                                              /* output */
                                              double *log_lik)
{
    __shared__ hipfftDoubleComplex buffer1[124], coeff;
    __shared__ double buffer2[124];

    const unsigned int tid = threadIdx.x;
    double real, imag, r;

    /* elem_mult(conj(sss_h12_est), sss_h12_try) */
    buffer1[tid].x =   sss_est[tid].x * sss_try_orig[tid];
    buffer1[tid].y = - sss_est[tid].y * sss_try_orig[tid];
    
    __syncthreads();

    for (int s = 128 / 2; s > 0; s >>= 1) {
        if ((tid < s) && (tid + s < 124)) {
            buffer1[tid].x += buffer1[tid + s].x;
            buffer1[tid].y += buffer1[tid + s].y;
        }
        __syncthreads();
    }

    if (tid == 0) {
        real = buffer1[0].x; imag = buffer1[0].y;
        r = sqrt(real * real + imag * imag);
        coeff.x =  real / r;
        coeff.y = -imag / r;
    }

    __syncthreads();

    real = sss_try_orig[tid] * coeff.x - sss_est[tid].x;
    imag = sss_try_orig[tid] * coeff.y - sss_est[tid].y;
    buffer2[tid] = - (real * real + imag * imag) / sss_np_est[tid];

    __syncthreads();

    for (unsigned int s = 128 / 2; s > 0; s >>= 1) {
        if ((tid < s) && (tid + s < 124)) {
            buffer2[tid] += buffer2[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        *log_lik = buffer2[0];
    }
}



/*
 *
 */
__global__ void sss_detect_ml_kernel(float *d_sss_h12_np_est, hipfftDoubleComplex *d_sss_h12_nrm_est, hipfftDoubleComplex *d_sss_h12_ext_est,
                                     int n_id_2,
                                     // output
                                     double *d_log_lik_nrm, double *d_log_lik_ext)
{
    __shared__ float sss_h12_np_est[124];
    __shared__ int sss_h12_try[124], sss_h21_try[124], *sss_try;
    __shared__ hipfftDoubleComplex sss_h12_nrm_est[124], sss_h12_ext_est[124], *sss_h12_est;
    __shared__ unsigned int word12[4], word21[4];
    __shared__ double *d_log_lik;

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;

    if (tid == 0) {
        const int bid_mod4 = bid % 4;
        const int bid_div4 = bid / 4;

        word12[0] = word21[2] = sss_fd[bid_div4][n_id_2][0][0];
        word12[1] = word21[3] = sss_fd[bid_div4][n_id_2][0][1];

        word12[2] = word21[0] = sss_fd[bid_div4][n_id_2][1][0];
        word12[3] = word21[1] = sss_fd[bid_div4][n_id_2][1][1];

        // sss_detect_ml_helper_function(sss_h12_np_est, sss_h12_nrm_est, sss_h12_try, &d_log_lik_nrm[bid * 2 + 0]);
        // sss_detect_ml_helper_function(sss_h12_np_est, sss_h12_nrm_est, sss_h21_try, &d_log_lik_nrm[bid * 2 + 1]);
        // sss_detect_ml_helper_function(sss_h12_np_est, sss_h12_ext_est, sss_h12_try, &d_log_lik_ext[bid * 2 + 0]);
        // sss_detect_ml_helper_function(sss_h12_np_est, sss_h12_ext_est, sss_h21_try, &d_log_lik_ext[bid * 2 + 1]);

        sss_h12_est = (bid_mod4 / 2) ? sss_h12_ext_est : sss_h12_nrm_est;
        sss_try = (bid_mod4 & 1) ? sss_h21_try : sss_h12_try;
        d_log_lik = (bid_mod4 / 2) ? &d_log_lik_ext[(bid_mod4 & 1) * 168 + bid_div4] : &d_log_lik_nrm[(bid_mod4 & 1) * 168 + bid_div4]; 
    }

    __syncthreads();

    sss_h12_np_est[tid] = d_sss_h12_np_est[tid];
    sss_h12_nrm_est[tid].x = d_sss_h12_nrm_est[tid].x;
    sss_h12_nrm_est[tid].y = d_sss_h12_nrm_est[tid].y;
    sss_h12_ext_est[tid].x = d_sss_h12_ext_est[tid].x;
    sss_h12_ext_est[tid].y = d_sss_h12_ext_est[tid].y;

    /* to_cvec(sss_h12_try_orig) and to_cvec(sss_h21_try_orig) */
    /* tid = 0...30, 31...61, 62...92, 93...123 */

    sss_h12_try[tid] = 1 - 2 * ((word12[tid / 31] >> (tid % 31)) & 1);
    sss_h21_try[tid] = 1 - 2 * ((word21[tid / 31] >> (tid % 31)) & 1);

    __syncthreads();

    sss_detect_ml_helper_function(sss_h12_np_est, sss_h12_est, sss_try, d_log_lik);
}



/*
 *
 */
__device__ void log_lik_maximum(double *d_in, double *d_max, int *d_idx)
{
    __shared__ double shared[168 * 4];
    __shared__ int index[168 * 4];

    const unsigned int tid = threadIdx.x;
    const unsigned int tid_mod168 = tid % 168;
    const unsigned int tid_div168 = tid / 168;

    shared[tid] = d_in[tid];
    index[tid] = tid_mod168;

    __syncthreads();

    for (unsigned int s = 256 / 2; s > 0; s >>= 1) {
         if ((tid_mod168 < s) && (tid_mod168 + s < 168)) {
             if (shared[tid] < shared[tid + s]) {
                 double value;
                 value = shared[tid];
                 shared[tid] = shared[tid + s];
                 shared[tid + s] = value;

                 int idx;
                 idx = index[tid];
                 index[tid] = index[tid + s];
                 index[tid + s] = idx;
             }
         }
         __syncthreads();
    }

    if (tid_mod168 == 0) {
        d_max[tid_div168] = shared[tid_div168 * 168];
        d_idx[tid_div168] = index[tid_div168 * 168];
    }
}



/*
 *
 */
__device__ void log_lik_mean_var(double *d_in, double *d_mean, double *d_var)
{
    __shared__ double shared[168 * 4], shared_sqr[168 * 4];
    const unsigned int tid = threadIdx.x;
    double mean;

    shared[tid] = d_in[tid];
    shared_sqr[tid] = d_in[tid] * d_in[tid];

    __syncthreads();

    for (unsigned int s = 1024 / 2; s > 0; s >>= 1) {
        if ((tid < s) && (tid + s < 168 * 4)) {
             shared[tid] += shared[tid + s];
             shared_sqr[tid] += shared_sqr[tid + s];
        }
        __syncthreads();
    }

    mean = shared[0] / (168 * 4);

    if (tid == 0) {
        *d_mean = mean;
        *d_var = (shared_sqr[tid] / (168 * 4)) - mean * mean;
    }
}



/*
 *
 */
__global__ void sss_detect_ml_decision_kernel(double *d_log_lik, int thresh2_n_sigma, int ind,
                                              double fc_requested, double fc_programmed, double fs_programmed, double freq,
                                              // output
                                              int *d_n_id_1_est, double *d_frame_start, int *d_cp_type)
{
    __shared__ double log_lik_max[4], log_lik_mean, log_lik_var;
    __shared__ int log_lik_idx[4];
 
    const unsigned int tid = threadIdx.x;

    log_lik_maximum(&d_log_lik[0], &log_lik_max[0], &log_lik_idx[0]);
    __syncthreads();

    log_lik_mean_var(&d_log_lik[0], &log_lik_mean, &log_lik_var);
    __syncthreads(); 

    if (tid == 0) {
        double max_value = log_lik_max[0];
        int max_idx = 0;

        for (int i = 1; i <= 3; i++) {
            if (log_lik_max[i] > max_value) {
                max_value = log_lik_max[i];
                max_idx = i;
            }
        }

        *d_n_id_1_est = -1;

        const double k_factor = (fc_requested - freq) / fc_programmed;
        double frame_start = ind + (128 + 9 - 960 - 2) * 16 / FS_LTE * fs_programmed * k_factor;
        if (max_idx & 1)
            frame_start = frame_start + 9600 * k_factor * 16 / FS_LTE * fs_programmed * k_factor;

#undef WRAP
#define WRAP(x,sm,lg) (fmod((x)-(sm),(lg)-(sm))+(sm))
        frame_start = WRAP(frame_start, -0.5, (2*9600.0-0.5)*16/FS_LTE*fs_programmed*k_factor);

        if (max_value < log_lik_mean + sqrt(log_lik_var) * thresh2_n_sigma)
            return;
        *d_cp_type = (max_idx / 2) ? 2 /* extend cp */ : 1 /* normal cp */;
        *d_n_id_1_est = log_lik_idx[max_idx];
        *d_frame_start = frame_start;
    }
}



/*
 *
 */
__global__ void pss_sss_foe_multiblocks_step1_kernel(hipfftDoubleComplex *d_capbuf, int n_sss,
                                                     unsigned short n_id_cell, int n_symb_dl, double first_sss_dft_location, unsigned int pss_sss_dist, int sn,
                                                     double fc_requested, double fc_programmed, double fs_programmed, double freq,
                                                     // output
                                                     hipfftDoubleComplex *d_M)
{
    __shared__ hipfftDoubleComplex s_pss_dft[128], s_sss_dft[128];
    __shared__ hipfftDoubleComplex h_raw_fo_pss[62], sss_raw_fo[62];
    __shared__ hipfftDoubleComplex coeff;
    __shared__ float pss_np, M_real, M_imag;

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;

    hipfftDoubleComplex shift, acc, *p_pss_fd;
    unsigned int sss_fd_bit;
    const double k_factor = (fc_requested - freq) / fc_programmed;
    unsigned int *p_sss_fd;     
    const int n_id_1 = n_id_cell / 3, n_id_2 = n_id_cell % 3;
    double real, imag;

    // Determine where we can find both PSS and SSS
    // Q : original code snippet for EXTENDED_CP : pss_sss_dist=round_i((128+32)*k_factor); ???
    const unsigned int sss_dft_location = lround(first_sss_dft_location + bid * (9600 * 16 / FS_LTE * fs_programmed * k_factor));

    // Find the PSS and use it to estimate the channel.
    const unsigned int pss_dft_location = sss_dft_location + pss_sss_dist;

    if (tid == 0) {
        pss_np = M_real = M_imag = 0.0;

        // exp(J*pi*(-cell_in.freq)/(FS_LTE/16/2)*(-pss_sss_dist))

        coeff.x = cos(HIP_PI * freq / (FS_LTE / 16 / 2) * pss_sss_dist);
        coeff.y = sin(HIP_PI * freq / (FS_LTE / 16 / 2) * pss_sss_dist);
    }

    p_pss_fd = (hipfftDoubleComplex *)&pss_fd[n_id_2];
    p_sss_fd = (unsigned int *)&sss_fd[n_id_1][n_id_2][(bid + sn) & 1];

    double k = HIP_PI * (-freq) / ((fs_programmed * k_factor) / 2);

    shift.x = cos(k * tid);
    shift.y = sin(k * tid);

    __syncthreads();

    if (tid < 2) {
        s_pss_dft[tid + 126].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location + tid], shift);
        s_pss_dft[tid + 126].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location + tid], shift);
        s_sss_dft[tid + 126].x = COMPLEX_MUL_REAL(d_capbuf[sss_dft_location + tid], shift);
        s_sss_dft[tid + 126].y = COMPLEX_MUL_IMAG(d_capbuf[sss_dft_location + tid], shift);
    } else if (tid < 128) {
        s_pss_dft[tid - 2].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location + tid], shift);
        s_pss_dft[tid - 2].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location + tid], shift);
        s_sss_dft[tid - 2].x = COMPLEX_MUL_REAL(d_capbuf[sss_dft_location + tid], shift);
        s_sss_dft[tid - 2].y = COMPLEX_MUL_IMAG(d_capbuf[sss_dft_location + tid], shift);
    }

    __syncthreads();

    if (tid == 0) {
        kernel_fft_radix2(s_pss_dft, 128);
    } else if (tid == 64) {
        kernel_fft_radix2(s_sss_dft, 128);
    }
    
    __syncthreads();

    // h_raw_fo_pss : concat(dft_out.right(31), dft_out.mid(1,31)) .* conj(pss_fd)
    // sss_raw_fo   : concat(dft_out.right(31), dft_out.mid(1,31)) .* exp(J*pi*-cell_in.freq/(FS_LTE/16/2)*-pss_sss_dist) .* sss_fd(n_id_1,n_id_2, 0 or 1)

    // index   :  0,  1,       30, 31, 32, 33,    , 61
    // dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    // pss_fd  :  0,  1, ....  30, 31, 32, 33, ..., 61

    if (tid < 31) {
        sss_fd_bit = (p_sss_fd[0] >> tid) & 1;

        h_raw_fo_pss[tid].x = SQRT128_INV * (s_pss_dft[tid + 97].x * p_pss_fd[tid].x + s_pss_dft[tid + 97].y * p_pss_fd[tid].y);
        h_raw_fo_pss[tid].y = SQRT128_INV * (s_pss_dft[tid + 97].y * p_pss_fd[tid].x - s_pss_dft[tid + 97].x * p_pss_fd[tid].y);
        sss_raw_fo[tid].x = SQRT128_INV * COMPLEX_MUL_REAL(s_sss_dft[tid + 97], coeff);
        sss_raw_fo[tid].y = SQRT128_INV * COMPLEX_MUL_IMAG(s_sss_dft[tid + 97], coeff);
    } else if (tid < 62) {
        sss_fd_bit = ((p_sss_fd[1] >> (tid - 31)) & 1);

        h_raw_fo_pss[tid].x = SQRT128_INV * (s_pss_dft[tid - 30].x * p_pss_fd[tid].x + s_pss_dft[tid - 30].y * p_pss_fd[tid].y);
        h_raw_fo_pss[tid].y = SQRT128_INV * (s_pss_dft[tid - 30].y * p_pss_fd[tid].x - s_pss_dft[tid - 30].x * p_pss_fd[tid].y);
        sss_raw_fo[tid].x = SQRT128_INV * COMPLEX_MUL_REAL(s_sss_dft[tid - 30], coeff);
        sss_raw_fo[tid].y = SQRT128_INV * COMPLEX_MUL_IMAG(s_sss_dft[tid - 30], coeff);
    }

    __syncthreads();

    // Smoothing... Basic...

    //                  t  = 0, 1, 2, 3,  4,  5,  6,  7,  8,  9, 10,        , 53, 54, 55, 56, 57, 58, 59, 60, 61
    // lt  = MAX(0, t - 6) = 0, 0, 0, 0,  0,  0,  0,  1,  2,  3,  4, ...... , 47, 48, 49, 50, 51, 52, 53, 54, 55
    // rt  = MIN(61,t + 6) = 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, ...... , 59, 60, 61, 61, 61, 61, 61, 61, 61
    // len = rt-lt + 1     = 7, 8, 9,10, 11, 12, 13, 13, 13, 13, 13, ...... , 13, 13, 13, 12, 11, 10,  9,  8,  7

    acc.x = 0.0; acc.y = 0.0;
    if (tid < 6) {
        for (unsigned int i = 0; i <= tid + 6; i++) {
            acc.x += h_raw_fo_pss[i].x;
            acc.y += h_raw_fo_pss[i].y;
        }
        acc.x /= (tid + 7);
        acc.y /= (tid + 7);
    } else if (tid < 56) {
        for (unsigned int i = tid - 6; i <= tid + 6; i++) {
            acc.x += h_raw_fo_pss[i].x;
            acc.y += h_raw_fo_pss[i].y;
        }
        acc.x /= 13;
        acc.y /= 13;
    } else if (tid < 62) {
        for (unsigned int i = tid - 6; i <= 61; i++) {
            acc.x += h_raw_fo_pss[i].x;
            acc.y += h_raw_fo_pss[i].y;
        }
        acc.x /= (68 - tid);
        acc.y /= (68 - tid);
    }

    // Estimate noise power.
    // pss_np = sigpower(h_sm.get_row(tid) - h_raw_fo_pss.get_row(tid));

    if (tid < 62) {
        double noise_r = acc.x - h_raw_fo_pss[tid].x;
        double noise_i = acc.y - h_raw_fo_pss[tid].y;
        atomicAdd(&pss_np, (float)(noise_r * noise_r + noise_i * noise_i));
    }

    __syncthreads();

    // sss_raw_fo : concat(dft_out.right(31), dft_out.mid(1,31)) .* conj(pss_fd)

    // index   :  0,  1,       30, 31, 32, 33,    , 61
    // dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    // pss_fd  :  0,  1, ....  30, 31, 32, 33, ..., 61

    // Compare PSS to SSS. With no frequency offset, arg(M) is zero.

    if (tid < 62) {
        // sss_fd_bit = 1 : a+bj -> -a-bj -> -a+bj
        // sss_fd_bit = 0 : a+bj ->  a+bj ->  a-bj

        if (sss_fd_bit)
            sss_raw_fo[tid].x = - sss_raw_fo[tid].x;
        else
            sss_raw_fo[tid].y = - sss_raw_fo[tid].y;
 
        // elem_mult(conj(sss_raw_fo), h_raw_fo_pss, to_cvec(elem_mult(sqr(h_sm), 1.0/(2*sqr(h_sm)*pss_np+sqr(pss_np))

        double sqr_sm = (acc.x * acc.x + acc.y * acc.y);
        imag = real = (sqr_sm * 62) / ((2 * sqr_sm * pss_np) + (pss_np * pss_np / 62));

        real *= COMPLEX_MUL_REAL(sss_raw_fo[tid], h_raw_fo_pss[tid]);
        imag *= COMPLEX_MUL_IMAG(sss_raw_fo[tid], h_raw_fo_pss[tid]);

        atomicAdd(&M_real, (float)real);
        atomicAdd(&M_imag, (float)imag);
    }

    __syncthreads();

    if (tid == 0) {
        d_M[bid].x = 1.0 * M_real;
        d_M[bid].y = 1.0 * M_imag;
    }
}



/*
 *
 */
__global__ void pss_sss_foe_multiblocks_step2_kernel(hipfftDoubleComplex *d_M, int n_sss, unsigned int pss_sss_dist,
                                                     double fc_requested, double fc_programmed, double fs_programmed, double freq,
                                                     // output
                                                     double *d_adjust_f)
{
    __shared__ hipfftDoubleComplex s_M[64];

    const unsigned int tid = threadIdx.x;
    const double k_factor = (fc_requested - freq) / fc_programmed;

    if (tid > 64) {
        printf("Caution : number of thread is larger than reserved space !\n");
        return;
    }

    if (tid < n_sss) {
        s_M[tid] = d_M[tid];
    }

    __syncthreads();

    for (unsigned int s = 64; s > 0; s >>= 1) {
        if ((tid < s) && ((tid + s) < n_sss) && ((tid + s) < 50)) {
            s_M[tid].x += s_M[tid + s].x;
            s_M[tid].y += s_M[tid + s].y;
        }
        __syncthreads();
    }
    

    if (tid == 0) {
        *d_adjust_f = angle((float)s_M[0].x, (float)s_M[0].y) / (2 * HIP_PI) / (1 / (fs_programmed * k_factor) * pss_sss_dist);
    }
}



// Perform FOE using only the PSS and SSS.
// The PSS correlation peak gives us the frequency offset within 2.5kHz.
// The PSS/SSS can be used to estimate the frequency offset within a
// much finer resolution.

/*
 *
 */
__global__ void pss_sss_foe_singleblock_kernel(hipfftDoubleComplex *d_capbuf, int n_sss,
                                               unsigned short n_id_cell, int n_symb_dl, double first_sss_dft_location, unsigned int pss_sss_dist, int sn,
                                               double fc_requested, double fc_programmed, double fs_programmed, double freq,
                                               // output
                                               double *d_adjust_f)
{
    __shared__ float M_real, M_imag;
    __shared__ hipfftDoubleComplex shift[128], coeff, *p_pss_fd;
    hipfftDoubleComplex s_capbuf[128];
    hipfftDoubleComplex h_raw_fo_pss[62], h_sm[62];
    hipfftDoubleComplex acc, M;
    unsigned int *p_sss_fd;

    const double k_factor = (fc_requested - freq) / fc_programmed;
    const unsigned int tid = threadIdx.x;
    const int n_id_1 = n_id_cell / 3, n_id_2 = n_id_cell % 3;
    float real, imag;
    double pss_np;

    // Determine where we can find both PSS and SSS
    // Q : original code snippet for EXTENDED_CP : pss_sss_dist=round_i((128+32)*k_factor); ???
    const unsigned int sss_dft_location = lround(first_sss_dft_location + tid * (9600 * 16 / FS_LTE * fs_programmed * k_factor));

    // Find the PSS and use it to estimate the channel.
    const unsigned int pss_dft_location = sss_dft_location + pss_sss_dist;

    if (tid == 0) {
        M_real = 0.0; M_imag = 0.0;

        // exp(J*pi*(-cell_in.freq)/(FS_LTE/16/2)*(-pss_sss_dist))

        coeff.x = cos(HIP_PI * freq / (FS_LTE / 16 / 2) * pss_sss_dist);
        coeff.y = sin(HIP_PI * freq / (FS_LTE / 16 / 2) * pss_sss_dist);

        p_pss_fd = (hipfftDoubleComplex *)&pss_fd[n_id_2];
    }

    p_sss_fd = (unsigned int *)&sss_fd[n_id_1][n_id_2][(tid + sn) & 1];

    double k = HIP_PI * (-freq) / ((fs_programmed * k_factor) / 2);

    for (unsigned int i = tid; i < 128; i += n_sss) {
        shift[i].x = cos(k * i);
        shift[i].y = sin(k * i);
    }

    __syncthreads();

    // implement extract_psss(capbuf.mid(pss_dft_location, 128), -cell_in.freq, k_factor, fs_programmed)
    for (unsigned int t = pss_dft_location + 2, i = 2; i < 128; i++, t++) {
        /* 128 points data shift, so k multiply from 0 to 125 */

        s_capbuf[i - 2].x = COMPLEX_MUL_REAL(d_capbuf[t], shift[i]);
        s_capbuf[i - 2].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift[i]);
    }

    // time shift 2 points
    s_capbuf[126].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location], shift[0]);
    s_capbuf[126].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location], shift[0]);
    s_capbuf[127].x = COMPLEX_MUL_REAL(d_capbuf[pss_dft_location + 1], shift[1]);
    s_capbuf[127].y = COMPLEX_MUL_IMAG(d_capbuf[pss_dft_location + 1], shift[1]);

    kernel_fft_radix2(s_capbuf, 128);

    // h_raw_fo_pss : concat(dft_out.right(31), dft_out.mid(1,31)) .* conj(pss_fd)

    // index   :  0,  1,       30, 31, 32, 33,    , 61
    // dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    // pss_fd  :  0,  1, ....  30, 31, 32, 33, ..., 61

    for (unsigned int i = 1; i <= 31; i++) {
        // concat(dft_out.right(31), dft_out.mid(1,31)) * conj(pss_fd)

        h_raw_fo_pss[i + 30].x = SQRT128_INV * (s_capbuf[i].x * p_pss_fd[i + 30].x + s_capbuf[i].y * p_pss_fd[i + 30].y);
        h_raw_fo_pss[i + 30].y = SQRT128_INV * (s_capbuf[i].y * p_pss_fd[i + 30].x - s_capbuf[i].x * p_pss_fd[i + 30].y);

        h_raw_fo_pss[i - 1].x = SQRT128_INV * (s_capbuf[i + 96].x * p_pss_fd[i - 1].x + s_capbuf[i + 96].y * p_pss_fd[i - 1].y);
        h_raw_fo_pss[i - 1].y = SQRT128_INV * (s_capbuf[i + 96].y * p_pss_fd[i - 1].x - s_capbuf[i + 96].x * p_pss_fd[i - 1].y);
    }

    // Smoothing... Basic...

    //                  t  = 0, 1, 2, 3,  4,  5,  6,  7,  8,  9, 10,        , 53, 54, 55, 56, 57, 58, 59, 60, 61
    // lt  = MAX(0, t - 6) = 0, 0, 0, 0,  0,  0,  0,  1,  2,  3,  4, ...... , 47, 48, 49, 50, 51, 52, 53, 54, 55
    // rt  = MIN(61,t + 6) = 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, ...... , 59, 60, 61, 61, 61, 61, 61, 61, 61
    // len = rt-lt + 1     = 7, 8, 9,10, 11, 12, 13, 13, 13, 13, 13, ...... , 13, 13, 13, 12, 11, 10,  9,  8,  7

    acc.x = 0.0; acc.y = 0.0;
    pss_np = 0.0;
    for (unsigned int i = 0; i < 6; i++) {
        acc.x += h_raw_fo_pss[i].x;
        acc.y += h_raw_fo_pss[i].y;
    }

    for (unsigned int i = 0; i <= 6; i++) {
        acc.x += h_raw_fo_pss[i + 6].x;
        acc.y += h_raw_fo_pss[i + 6].y;

        h_sm[i].x = acc.x / (i + 7);
        h_sm[i].y = acc.y / (i + 7);
    }

    for (unsigned int i = 7; i <= 55; i++) {
        acc.x += (h_raw_fo_pss[i + 6].x - h_raw_fo_pss[i - 7].x);
        acc.y += (h_raw_fo_pss[i + 6].y - h_raw_fo_pss[i - 7].y);

        h_sm[i].x = acc.x / 13;
        h_sm[i].y = acc.y / 13;
    }

    for (unsigned int i = 56; i < 62; i++) {
        acc.x -= h_raw_fo_pss[i - 7].x;
        acc.y -= h_raw_fo_pss[i - 7].y;

        h_sm[i].x = acc.x / (61 + 7 - i);
        h_sm[i].y = acc.y / (61 + 7 - i);
    }

    // Estimate noise power.
    // pss_np = sigpower(h_sm.get_row(tid) - h_raw_fo_pss.get_row(tid));
    pss_np = 0.0;
    for (unsigned int i = 0; i < 62; i++) {
        double noise_r = h_sm[i].x - h_raw_fo_pss[i].x;
        double noise_i = h_sm[i].y - h_raw_fo_pss[i].y;
        pss_np += (noise_r * noise_r + noise_i * noise_i);
    }
    pss_np /= 62;

    // Calculate the SSS in the frequency domain

    // extract_psss(capbuf.mid(sss_dft_location,128),-cell_in.freq,k_factor,fs_programmed)
    for (unsigned int t = sss_dft_location + 2, i = 2; i < 128; i++, t++) {
        /* 128 points data shift, so k multiply from 0 to 125 */

        s_capbuf[i - 2].x = COMPLEX_MUL_REAL(d_capbuf[t], shift[i]);
        s_capbuf[i - 2].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift[i]);
    }

    s_capbuf[126].x = COMPLEX_MUL_REAL(d_capbuf[sss_dft_location], shift[0]);
    s_capbuf[126].y = COMPLEX_MUL_IMAG(d_capbuf[sss_dft_location], shift[0]);
    s_capbuf[127].x = COMPLEX_MUL_REAL(d_capbuf[sss_dft_location + 1], shift[1]);
    s_capbuf[127].y = COMPLEX_MUL_IMAG(d_capbuf[sss_dft_location + 1], shift[1]);

    kernel_fft_radix2(s_capbuf, 128);

    // sss_raw_fo : concat(dft_out.right(31), dft_out.mid(1,31)) .* conj(pss_fd)

    // index   :  0,  1,       30, 31, 32, 33,    , 61
    // dft_out : 97, 98, .... 127,  1,  2,  3, ..., 31
    // pss_fd  :  0,  1, ....  30, 31, 32, 33, ..., 61

    // Compare PSS to SSS. With no frequency offset, arg(M) is zero.

    M.x = 0.0; M.y = 0.0;
    for (unsigned int i = 1, word1 = p_sss_fd[0], word2 = p_sss_fd[1]; i <= 31; i++, word1 >>= 1, word2 >>= 1) {
        hipfftDoubleComplex sss_raw_fo;
        double sm_power;

        // index = 0, 1 ... 31 of sss_raw_fo

        sm_power = (h_sm[i - 1].x * h_sm[i - 1].x + h_sm[i - 1].y * h_sm[i - 1].y);
        sm_power = SQRT128_INV * sm_power / (2 * sm_power * pss_np + pss_np * pss_np);

        if (word1 & 1) sm_power *= -1.0;

        // dft_out .* sss_fd

        // conj( .* exp(J*pi*(-cell_in.freq)/(FS_LTE/16/2)*(-pss_sss_dist)))

        sss_raw_fo.x = COMPLEX_MUL_REAL(s_capbuf[i + 96], coeff);
        sss_raw_fo.y = -COMPLEX_MUL_IMAG(s_capbuf[i + 96], coeff);

        // .* h_raw_fo_pss

        M.x += sm_power * COMPLEX_MUL_REAL(sss_raw_fo, h_raw_fo_pss[i - 1]);
        M.y += sm_power * COMPLEX_MUL_IMAG(sss_raw_fo, h_raw_fo_pss[i - 1]);

        // index = 32, 33 ... 61 of sss_raw_fo

        sm_power = (h_sm[i + 30].x * h_sm[i + 30].x + h_sm[i + 30].y * h_sm[i + 30].y);
        sm_power = SQRT128_INV * sm_power / (2 * sm_power * pss_np + pss_np * pss_np);

        if (word2 & 1) sm_power *= -1.0;

        // dft_out .* sss_fd

        // conj( .* exp(J*pi*(-cell_in.freq)/(FS_LTE/16/2)*(-pss_sss_dist)))

        sss_raw_fo.x = COMPLEX_MUL_REAL(s_capbuf[i], coeff);
        sss_raw_fo.y = -COMPLEX_MUL_IMAG(s_capbuf[i], coeff);

        // .* h_raw_fo_pss

        M.x += sm_power * COMPLEX_MUL_REAL(sss_raw_fo, h_raw_fo_pss[i + 30]);
        M.y += sm_power * COMPLEX_MUL_IMAG(sss_raw_fo, h_raw_fo_pss[i + 30]);
    }

    real = 1.0 * M.x;
    imag = 1.0 * M.y;

    atomicAdd(&M_real, real);
    atomicAdd(&M_imag, imag);

    __syncthreads();

    if (tid == 0) {
        *d_adjust_f = angle(M_real, M_imag) / (2 * HIP_PI) / (1 / (fs_programmed * k_factor) * pss_sss_dist);
    }
}



/**
 * Implement 36.211 7.2.
 * Generate Pseudo-random sequence and store the result into array rather than queue.
 * Bits are stored from LSB to MSB.
 *
 * \param init_in       Initial value of pseudo-random sequence generator
 * \param seqLn         How many consecutive pseudo-random number to be generated.
 * \param initOffset    The starting position of pseudo-random number to be generated
 * \param pSeqOut       Pointer to output array of UNSG32 which holds generated pseudo-random sequence
 */
__device__ void pn_seq_lsb_to_msb(unsigned int d_init_in, unsigned int d_seq_len, unsigned int d_init_offset, unsigned int *d_pseq_out)
{
    unsigned int x1, x2, tmp_val;
    unsigned int i;

    const unsigned int m2_v1600[31] = {
        0x0099110E, 0x004C8887, 0x40264444, 0x20132222, 0x10099111, 0x4804C88F, 0x64026440, 0x32013220,
        0x19009910, 0x0C804C88, 0x06402644, 0x03201322, 0x01900991, 0x40C804CF, 0x60640260, 0x30320130,
        0x18190098, 0x0C0C804C, 0x06064026, 0x03032013, 0x4181900E, 0x20C0C807, 0x50606404, 0x28303202,
        0x14181901, 0x4A0C0C87, 0x65060644, 0x32830322, 0x19418191, 0x4CA0C0CF, 0x66506060
    };

    unsigned int init_in = d_init_in;
    unsigned int seq_len = d_seq_len;
    unsigned int init_offset = d_init_offset;
    unsigned int *pseq_out = d_pseq_out;

    /* x1 is independent of c_init,
       so it can be pre-calculated at N=1600.
       x2 depends on c_init,
       so it need to multiply M^1570 to obtain its value at N=1600 */
    x1 = 0x54D21B24;
    x2 = 0;
    for (i = 0; i < 31; i++) {
        tmp_val = init_in & m2_v1600[i];

        /* determine there are even or odd number of bits set in tmp_val */
        tmp_val ^= (tmp_val >> 16);
        tmp_val ^= (tmp_val >> 8);
        tmp_val ^= (tmp_val >> 4);
        tmp_val &= 0xF;
        tmp_val = ((0x6996 >> tmp_val) & 1);

        x2 |= (tmp_val << (31 - i));
    }

    for (i = 0; i < init_offset; i++) {
        x1 >>= 1;
        x1  |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);   /* bit0 of 0x55AA is x(0)^x(3) of [3:0] */
        x2 >>= 1;
        x2  |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);   /* bit0 of 0x55AA is x(0)^x(3) */
    }

    tmp_val = 0;
    for (i = 0; i < seq_len; i++) {
        /* store from LSB to MSB */
        tmp_val |= ((x1 ^ x2) >> (31 - (i % 32)));
        if (((i + 1) % 32) == 0) {
            *pseq_out++ = tmp_val;
            tmp_val    = 0;
        }

        x1 >>= 1;
        x1 |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);
        x2 >>= 1;
        x2 |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);
    }

    if (seq_len && (seq_len % 32))
        *pseq_out = tmp_val;
}

/**
 * Implement 36.211 7.2.
 * Generate Pseudo-random sequence and store the result into array rather than queue.
 * Bits are stored from MSB to LSB.
 *
 * \param init_in       Initial value of pseudo-random sequence generator
 * \param seqLn         How many consecutive pseudo-random number to be generated.
 * \param initOffset    The starting position of pseudo-random number to be generated
 * \param pSeqOut       Pointer to output array of unsigned int which holds generated pseudo-random sequence
 */
__device__ void pn_seq_msb_to_lsb(unsigned int d_init_in, unsigned int d_seq_len, unsigned int d_init_offset, unsigned int *d_pseq_out)
{
    unsigned int x1, x2, tmp_val;
    unsigned int i;

    const unsigned int m2_v1600[31] = {
        0x0099110E, 0x004C8887, 0x40264444, 0x20132222, 0x10099111, 0x4804C88F, 0x64026440, 0x32013220,
        0x19009910, 0x0C804C88, 0x06402644, 0x03201322, 0x01900991, 0x40C804CF, 0x60640260, 0x30320130,
        0x18190098, 0x0C0C804C, 0x06064026, 0x03032013, 0x4181900E, 0x20C0C807, 0x50606404, 0x28303202,
        0x14181901, 0x4A0C0C87, 0x65060644, 0x32830322, 0x19418191, 0x4CA0C0CF, 0x66506060
    };

    unsigned int init_in = d_init_in;
    unsigned int seq_len = d_seq_len;
    unsigned int init_offset = d_init_offset;
    unsigned int *pseq_out = d_pseq_out;

    /* x1 is independent of c_init,
       so it can be pre-calculated at N=1600.
       x2 depends on c_init,
       so it need to multiply M^1570 to obtain its value at N=1600 */
    x1 = 0x54D21B24;
    x2 = 0;
    for (i = 0; i < 31; i++) {
        tmp_val = init_in & m2_v1600[i];

        /* determine there are even or odd number of bits set in tmp_val */
        tmp_val ^= (tmp_val >> 16);
        tmp_val ^= (tmp_val >> 8);
        tmp_val ^= (tmp_val >> 4);
        tmp_val &= 0xF;
        tmp_val = ((0x6996 >> tmp_val) & 1);

        x2 |= (tmp_val << (31 - i));
    }

    for (i = 0; i < init_offset; i++) {
        x1 >>= 1;
        x1 |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);
        x2 >>= 1;
        x2 |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);
    }

    tmp_val = 0;
    for (i = 0; i < seq_len; i++) {
        /* store from MSB to LSB */
        tmp_val |= (((x1 ^ x2) & 0x80000000) >> (i % 32));
        if (((i + 1) % 32) == 0) {
            *pseq_out++ = tmp_val;
            tmp_val    = 0;
        }

        x1 >>= 1;
        x1 |= (((0x55AA >> (x1 & 0xF)) & 1) << 31);
        x2 >>= 1;
        x2 |= (((0x6996 >> (x2 & 0xF)) & 1) << 31);
    }

    if (seq_len && (seq_len % 32))
        *pseq_out = tmp_val;
}

__device__ void kernel_fft_radix2(hipfftDoubleComplex *c_io, int N)
{
    int n, s, l, i;
    hipfftDoubleComplex *d_tw = &d_tw128[0];

    for (n = N >> 1, l = 1; n >= 1; n >>= 1, l <<= 1) {
        for (i = 0; i < l; i++) {
            for (s = 0; s < n; s++) {
                hipfftDoubleComplex a, b, aa, bb, tw;

                a = c_io[s + n*0 + i*n*2];
                b = c_io[s + n*1 + i*n*2];
                tw = d_tw[s * l];

                aa.x = a.x + b.x;
                aa.y = a.y + b.y;

                bb.x = (a.x - b.x) * tw.x - (a.y - b.y) * tw.y;
                bb.y = (a.y - b.y) * tw.x + (a.x - b.x) * tw.y;

                c_io[s + n*0 + i*n*2] = aa;
                c_io[s + n*1 + i*n*2] = bb;
            }
        }
    }

    // bit reverse
    for (n = 0; n < N; n++) {
        hipfftDoubleComplex c;
        int idx = d_radix2_bitreverse[n];

        if (idx <= n)
            continue;

        c = c_io[idx];
        c_io[idx] = c_io[n];
        c_io[n] = c;
    }
}



/*
 *
 */
__global__ void extract_tfg_multiblocks_kernel(hipfftDoubleComplex *d_capbuf, hipfftDoubleComplex *d_tfg, double *d_tfg_timestamp, double *d_adjust_f,
                                               unsigned short n_id_cell, int n_symb_dl, double frame_start,
                                               double fc_requested, double fc_programmed, double fs_programmed, double freq)
{
    __shared__ hipfftDoubleComplex s_capbuf[128];

    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;

    int dft_location_i;
    double freq_fine = freq + *d_adjust_f;
    const double k_factor = (fc_requested - freq_fine) / fc_programmed;
    double dft_location = frame_start + ((n_symb_dl == 6) ? 32 : 10) * 16 / FS_LTE * fs_programmed * k_factor;
    hipfftDoubleComplex shift, coeff;

    if (dft_location - .01 * fs_programmed * k_factor > -0.5) {
        dft_location = dft_location - .01 * fs_programmed * k_factor;
    }

    dft_location += ((bid / n_symb_dl) * 960 + (bid % n_symb_dl) * (n_symb_dl == 6 ? 160 : 137))  * 16 / FS_LTE * fs_programmed * k_factor;
    dft_location_i = lround(dft_location);

    // cvec capbuf = fshift(capbuf_raw, -freq_fine, fs_programmed * k_factor);
    double k = HIP_PI * (-freq_fine) / (fs_programmed * k_factor / 2);
    double theta = k * (dft_location_i + tid);
    shift.x = cos(theta);
    shift.y = sin(theta);

    s_capbuf[tid].x = COMPLEX_MUL_REAL(d_capbuf[dft_location_i + tid], shift);
    s_capbuf[tid].y = COMPLEX_MUL_IMAG(d_capbuf[dft_location_i + tid], shift);

    __syncthreads();

    // DFT of 128 points
    // cvec dft_out = dft(capbuf.mid(round_i(dft_location), 128));

    if (tid == 0) {
        d_tfg_timestamp[bid] = dft_location;

        kernel_fft_radix2(s_capbuf, 128);
    }

    __syncthreads();

    //  92,  93,  94, ... , 127,  1,  2,  3, ..., 36 -> concat(dft_out.right(36), dft_out.mid(1, 36))
    //   0,   1,   2,     ,  35, 36, 37, 38,    , 71
    // -36, -35, -34, ... ,  -1,  1,  2,  3, .... 36

    // concat(dft_out.right(36), dft_out.mid(1,36));
    // exp((-J * 2 * pi * late / 128) * cn)

    double late = dft_location_i - dft_location;

    if (tid < 36) {
        coeff.x =  cos(2 * HIP_PI * late * (36 - tid) / 128);
        coeff.y =  sin(2 * HIP_PI * late * (36 - tid) / 128);

        d_tfg[bid * 72 + tid].x = SQRT128_INV * COMPLEX_MUL_REAL(s_capbuf[tid + 92], coeff);
        d_tfg[bid * 72 + tid].y = SQRT128_INV * COMPLEX_MUL_IMAG(s_capbuf[tid + 92], coeff);
    } else if (tid < 72) {
        coeff.x =  cos(2 * HIP_PI * late * (tid - 35) / 128);
        coeff.y = -sin(2 * HIP_PI * late * (tid - 35) / 128);

        d_tfg[bid * 72 + tid].x = SQRT128_INV * COMPLEX_MUL_REAL(s_capbuf[tid - 35], coeff);
        d_tfg[bid * 72 + tid].y = SQRT128_INV * COMPLEX_MUL_IMAG(s_capbuf[tid - 35], coeff);
    }
}



/*
 *
 */
__global__ void extract_tfg_singleblock_kernel(hipfftDoubleComplex *d_capbuf, hipfftDoubleComplex *d_tfg, double *d_tfg_timestamp, double *d_adjust_f,
                                               unsigned short n_id_cell, int n_symb_dl, double frame_start,
                                               double fc_requested, double fc_programmed, double fs_programmed, double freq)
{
    const unsigned int tid = threadIdx.x;

    hipfftDoubleComplex s_capbuf[128];
    int dft_location_i;
    double freq_fine = freq + *d_adjust_f;
    const double k_factor = (fc_requested - freq_fine) / fc_programmed;
    double dft_location = frame_start + ((n_symb_dl == 6) ? 32 : 10) * 16 / FS_LTE * fs_programmed * k_factor;

    if (dft_location - .01 * fs_programmed * k_factor > -0.5) {
        dft_location = dft_location - .01 * fs_programmed * k_factor;
    }

    dft_location += ((tid / n_symb_dl) * 960 + (tid % n_symb_dl) * (n_symb_dl == 6 ? 160 : 137))  * 16 / FS_LTE * fs_programmed * k_factor;
    dft_location_i = lround(dft_location);
    d_tfg_timestamp[tid] = dft_location;

    // cvec capbuf = fshift(capbuf_raw, -freq_fine, fs_programmed * k_factor);
    double k = HIP_PI * (-freq_fine) / (fs_programmed * k_factor / 2);

    for (unsigned int t = dft_location_i, i = 0; i < 128; i++, t++) {
        hipfftDoubleComplex shift;

        shift.x = cos(k * t);
        shift.y = sin(k * t);

        s_capbuf[i].x = COMPLEX_MUL_REAL(d_capbuf[t], shift);
        s_capbuf[i].y = COMPLEX_MUL_IMAG(d_capbuf[t], shift);
    }

    // DFT of 128 points
    // cvec dft_out = dft(capbuf.mid(round_i(dft_location), 128));

    kernel_fft_radix2(s_capbuf, 128);


    //  92,  93,  94, ... , 127,  1,  2,  3, ..., 36 -> concat(dft_out.right(36), dft_out.mid(1, 36))
    //   0,   1,   2,     ,  35, 36, 37, 38,    , 71
    // -36, -35, -34, ... ,  -1,  1,  2,  3, .... 36

    // concat(dft_out.right(36), dft_out.mid(1,36));
    // exp((-J * 2 * pi * late / 128) * cn)

    double late = dft_location_i - dft_location;

    for (unsigned int i = 1; i <= 36; i++) {
        hipfftDoubleComplex coeff;
        coeff.x =  cos(2 * HIP_PI * late * i / 128);
        coeff.y = -sin(2 * HIP_PI * late * i / 128);

        d_tfg[tid * 72 + 35 + i].x = SQRT128_INV * COMPLEX_MUL_REAL(s_capbuf[i], coeff);
        d_tfg[tid * 72 + 35 + i].y = SQRT128_INV * COMPLEX_MUL_IMAG(s_capbuf[i], coeff);

        coeff.y = -coeff.y;

        d_tfg[tid * 72 + 36 - i].x = SQRT128_INV * COMPLEX_MUL_REAL(s_capbuf[128 - i], coeff);
        d_tfg[tid * 72 + 36 - i].y = SQRT128_INV * COMPLEX_MUL_IMAG(s_capbuf[128 - i], coeff);
    }
}



/*
 *
 */
__global__ void tfoec_kernel(hipfftDoubleComplex *d_tfg, hipfftDoubleComplex *d_rs_extracted, double *d_tfg_timestamp,
                             unsigned short n_id_cell, int n_symb_dl,
                             double fc_requested, double fc_programmed, double fs_programmed,
                             // output
                             double *d_residual_f)
{
    __shared__ unsigned int rs_dl[20 * 2];
    __shared__ float foe_real, foe_imag;
    __shared__ float toe_real, toe_imag;


    const unsigned int tid = threadIdx.x;
    double dft_location;
    double late;

    // generate random sequences for symbol 0, 1, 3/4 of 20 slots

    if (tid < 20 * 2) {
        int slot = tid / 2;
        int l = (tid & 1) * (n_symb_dl - 3);
        int cinit = ((7 * (slot + 1) + l + 1) * (2 * n_id_cell + 1) << 10) + 2 * n_id_cell + (n_symb_dl == 7 ? 1 : 0);

        pn_seq_lsb_to_msb(cinit, 6 * 2 * 2, (55 - 3) * 2 * 2, &rs_dl[tid]);
    }

    if (tid == 0) {
        foe_real = 0.0; foe_imag = 0.0;
        toe_real = 0.0; toe_imag = 0.0;
    }

    dft_location = d_tfg_timestamp[tid];

    __syncthreads();

    if (tid < 122 * 2) {
        int slot = tid / 2;
        int second_sym = tid & 1;
        int l = second_sym * (n_symb_dl - 3);
        int rs_bits = rs_dl[(slot % 20) * 2 + second_sym];
        int v_offset = ((n_id_cell % 6) + second_sym * 3) % 6;

        // elem_mult(rs_extracted.get_row(t), conj(rs_dl.get_rs(mod(t, 20), sym_num)))
        for (unsigned int i = 0; i < 12; i++, rs_bits >>= 2, v_offset += 6) {
            hipfftDoubleComplex std_rs, rcvd_rs;

            // rs_symb = 1 / sqrt(2) ((1 - 2 * c(0)) + j (1 - 2 * c(1))

            std_rs.x = SQRT2_INV * (1.0 - ((rs_bits & 1) * 2));
            std_rs.y = SQRT2_INV * (1.0 - (rs_bits & 2));

            rcvd_rs = d_tfg[(slot * n_symb_dl + l) * 72 + v_offset];

            std_rs.y = -std_rs.y;

            d_rs_extracted[((tid & 1) * 122 + (tid / 2)) * 12 + i].x = COMPLEX_MUL_REAL(rcvd_rs, std_rs);
            d_rs_extracted[((tid & 1) * 122 + (tid / 2)) * 12 + i].y = COMPLEX_MUL_IMAG(rcvd_rs, std_rs);
        }
    }

    __syncthreads();

    if (tid < 121) {
        // CUDA 2.0+ capability support atomic addition of 32 bit floating point numbers
        // CUDA 6.0+ capability support atomic addition of 64 bit floating point numbers

        // sum(elem_mult(conj(col(0,n_slot-2)), col(1,-1)));

        float real = 0.0, imag = 0.0;
        for (unsigned int i = 0; i < 12; i++) {
            hipfftDoubleComplex rs_1, rs_2;

            rs_1 = d_rs_extracted[(0 + tid + 0) * 12 + i];
            rs_2 = d_rs_extracted[(0 + tid + 1) * 12 + i];

            rs_1.y = -rs_1.y;
            real += COMPLEX_MUL_REAL(rs_1, rs_2);
            imag += COMPLEX_MUL_IMAG(rs_1, rs_2);

            rs_1 = d_rs_extracted[(122 + tid + 0) * 12 + i];
            rs_2 = d_rs_extracted[(122 + tid + 1) * 12 + i];

            rs_1.y = -rs_1.y;
            real += COMPLEX_MUL_REAL(rs_1, rs_2);
            imag += COMPLEX_MUL_IMAG(rs_1, rs_2);
        }

        atomicAdd(&foe_real, real);
        atomicAdd(&foe_imag, imag);
    }

    __syncthreads();

    double residual_f = angle(foe_real, foe_imag) / (2*HIP_PI) / 0.0005;
    double k_factor_residual = (fc_requested - residual_f) / fc_programmed;
    late = dft_location - k_factor_residual * dft_location;

    if (tid == 0) *d_residual_f = residual_f;

    // -36, -35, -34, ... ,  -1,  1,  2,  3, .... 36
    // exp((-J * 2 * pi * late / 128) * cn)

    // tfg.get_row(t)*exp(J*2*pi* -residual_f*tfg_comp_timestamp(t)/(FS_LTE/16))
    // elem_mult(tfg_comp.get_row(t), exp((-J*2*pi*late/128)*cn))
    for (unsigned int i = 1; i <= 36; i++) {

        hipfftDoubleComplex coeff;
        double real, imag;

        coeff.x = cos(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) - (late * i / 128)));
        coeff.y = sin(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) - (late * i / 128)));

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 35 + i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 35 + i], coeff);

        d_tfg[tid * 72 + 35 + i].x = real;
        d_tfg[tid * 72 + 35 + i].y = imag;

        coeff.x = cos(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) + (late * i / 128)));
        coeff.y = sin(2 * HIP_PI * ((-residual_f) * dft_location / (FS_LTE / 16) + (late * i / 128)));

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 36 - i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 36 - i], coeff);

        d_tfg[tid * 72 + 36 - i].x = real;
        d_tfg[tid * 72 + 36 - i].y = imag;
    }

    __syncthreads();

    // Perform TOE.
    // Implemented by comparing subcarrier k of one OFDM symbol with subcarrier
    // k+3 of another OFDM symbol. This is why FOE must be performed first.
    // Slightly less performance but faster execution time could be obtained
    // by comparing subcarrier k with subcarrier k+6 of the same OFDM symbol.

    if (tid < 2 * 122 - 1) {

        int slot1 = tid / 2;
        int second_sym1 = tid & 1;
        int l1 = second_sym1 * (n_symb_dl - 3);
        int rs_bits1 = rs_dl[(slot1 % 20) * 2 + second_sym1];
        int v_offset1 = ((n_id_cell % 6) + second_sym1 * 3) % 6;

        int slot2 = (tid + 1) / 2;
        int second_sym2 = (tid + 1) & 1;
        int l2 = second_sym2 * (n_symb_dl - 3);
        int rs_bits2 = rs_dl[(slot2 % 20) * 2 + second_sym2];
        int v_offset2 = ((n_id_cell % 6) + second_sym2 * 3) % 6;

        float real, imag;

        hipfftDoubleComplex toe1, toe2;
        hipfftDoubleComplex std_rs, rcvd_rs;
        hipfftDoubleComplex r1v, r2v, r2v_prev;

        toe1.x = 0.0; toe1.y = 0.0;
        toe2.x = 0.0; toe2.y = 0.0;
        r2v_prev.x = 0.0; r2v_prev.y = 0.0;

        if (v_offset2 < v_offset1) {
            SWAP(slot1, slot2);
            SWAP(l1, l2);
            SWAP(rs_bits1, rs_bits2);
            SWAP(v_offset1, v_offset2);
        }

        for (unsigned int i = 0; i < 12; i++, rs_bits1 >>= 2, v_offset1 += 6, rs_bits2 >>= 2, v_offset2 += 6) {

            // rs_symb = 1 / sqrt(2) ((1 - 2 * c(0)) + j (1 - 2 * c(1))

            std_rs.x = SQRT2_INV * (1.0 - ((rs_bits1 & 1) * 2));
            std_rs.y = SQRT2_INV * (1.0 - ((rs_bits1 & 2)));

            rcvd_rs = d_tfg[(slot1 * n_symb_dl + l1) * 72 + v_offset1];

            std_rs.y = -std_rs.y;

            r1v.x = COMPLEX_MUL_REAL(rcvd_rs, std_rs);
            r1v.y = -COMPLEX_MUL_IMAG(rcvd_rs, std_rs); // this r1v is actually conj(r1v)

            std_rs.x = SQRT2_INV * (1.0 - ((rs_bits2 & 1) * 2));
            std_rs.y = SQRT2_INV * (1.0 - ((rs_bits2 & 2)));

            std_rs.y = -std_rs.y;

            rcvd_rs = d_tfg[(slot2 * n_symb_dl + l2) * 72 + v_offset2];

            r2v.x = COMPLEX_MUL_REAL(rcvd_rs, std_rs);
            r2v.y = COMPLEX_MUL_IMAG(rcvd_rs, std_rs);

            // elem_mult(conj(r1v), r2v)

            toe1.x += COMPLEX_MUL_REAL(r1v, r2v);
            toe1.y += COMPLEX_MUL_IMAG(r1v, r2v);

            r1v.y = -r1v.y;
            r2v.y = -r2v.y;   // this r2v is actually conj(r2v)

            // elem_mult(conj(r2v(i-1)), r1v(i))

            toe2.x += COMPLEX_MUL_REAL(r1v, r2v_prev);
            toe2.y += COMPLEX_MUL_IMAG(r1v, r2v_prev);

            r2v_prev = r2v;
        }

        real = 1.0 * (toe1.x + toe2.x);
        imag = 1.0 * (toe1.y + toe2.y);
        atomicAdd(&toe_real, real);
        atomicAdd(&toe_imag, imag);
    }

    __syncthreads();

    // double delay = -arg(toe)/3/(2*pi/128);
    double delay = -angle(toe_real, toe_imag) / 3 / (2 * HIP_PI / 128);

    // Perform TOC
    for (unsigned int i = 1; i <= 36; i++) {

        hipfftDoubleComplex coeff;
        double real, imag;

        coeff.x = cos(2 * HIP_PI * delay * i / 128);
        coeff.y = sin(2 * HIP_PI * delay * i / 128);

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 35 + i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 35 + i], coeff);

        d_tfg[tid * 72 + 35 + i].x = real;
        d_tfg[tid * 72 + 35 + i].y = imag;

        coeff.y = -coeff.y;

        real = COMPLEX_MUL_REAL(d_tfg[tid * 72 + 36 - i], coeff);
        imag = COMPLEX_MUL_IMAG(d_tfg[tid * 72 + 36 - i], coeff);

        d_tfg[tid * 72 + 36 - i].x = real;
        d_tfg[tid * 72 + 36 - i].y = imag;
    }
}



/*
 *
 */
__global__ void chan_est_kernel(hipfftDoubleComplex *d_tfg, int num_slot, int ant_port,
                                unsigned short n_id_cell, int n_symb_dl,
                                // output
                                hipfftDoubleComplex *d_ce_filt, double *d_err_pwr_acc)
{
    __shared__ unsigned int rs_dl[3];
    __shared__ hipfftDoubleComplex rcvd_rs[12 * 3], tfg_rs, conj_std_rs;
    __shared__ float ce_err_pwr;

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;

    /*  */
    int rs_no = bid;
    int port = ant_port;
    int rs_out = bid;

    /*  */

    const int port01 = 1 - (port / 2), port23 = port / 2;
    const int rs_per_slot = 1 << port01;
    const int total_rs = num_slot * rs_per_slot;
    const int rs_slot = (rs_no / rs_per_slot) % 20;
    // const int l_rs = port01 * ((rs_no & 1) * (n_symb_dl - 3)) + port23;
    const int v = port01 * ((port & 1) ^ (rs_no & 1)) * 3 + port23 * ((port & 1) + (rs_slot & 1)) * 3;
    const int k_offset = (v + (n_id_cell % 6)) % 6;

    int pos, rs_count;
    hipfftDoubleComplex filt;


    if (tid < 3) {
        const int cinit_rs_no = rs_no - 1 + tid;
        const int cinit_rs_slot = (cinit_rs_no / rs_per_slot) % 20;
        const int cinit_l_rs = port01 * ((cinit_rs_no & 1) * (n_symb_dl - 3)) + port23;
        const int cinit = ((7 * (cinit_rs_slot + 1) + cinit_l_rs + 1) * (2 * n_id_cell + 1) << 10) + 2 * n_id_cell + (n_symb_dl == 7 ? 1 : 0);

        pn_seq_lsb_to_msb(cinit, 6 * 2 * 2, (55 - 3) * 2 * 2, &rs_dl[tid]);

        ce_err_pwr = 0.0;
    }

    __syncthreads();

    if (tid < 36) {
        const int copy_rs_no = rs_no - 1 + (tid / 12);
        const int copy_rs_slot = copy_rs_no / rs_per_slot;
        const int copy_l_rs = port01 * ((copy_rs_no & 1) * (n_symb_dl - 3)) + port23;
        const int copy_v = port01 * ((port & 1) ^ (copy_rs_no & 1)) * 3 + port23 * ((port & 1) + (copy_rs_slot & 1)) * 3;
        const int copy_k_offset = (copy_v + (n_id_cell % 6)) % 6;

        const unsigned int rs_bits = (rs_dl[tid / 12] >> (2 * (tid % 12))) & 0x3;
        const int copy_pos = (tid / 12) * 12 + (tid % 12);


        if ((0 <= copy_rs_no) && (copy_rs_no < total_rs)) {
            tfg_rs.x = d_tfg[(copy_rs_slot * n_symb_dl + copy_l_rs) * 72 + copy_k_offset + 6 * (tid % 12)].x;
            tfg_rs.y = d_tfg[(copy_rs_slot * n_symb_dl + copy_l_rs) * 72 + copy_k_offset + 6 * (tid % 12)].y;

            conj_std_rs.x = SQRT2_INV * (1.0 - ((rs_bits & 0x1) * 2));
            conj_std_rs.y = - SQRT2_INV * (1.0 - (rs_bits & 0x2));

            rcvd_rs[copy_pos].x = COMPLEX_MUL_REAL(tfg_rs, conj_std_rs);
            rcvd_rs[copy_pos].y = COMPLEX_MUL_IMAG(tfg_rs, conj_std_rs);
        } else {
            rcvd_rs[copy_pos].x = 0.0;
            rcvd_rs[copy_pos].y = 0.0;
        }
    }

    __syncthreads();

    //  0   1   2   3                   9   10    11
    //    0   1   2   3                   9    10    11
    //  0   1   2   3                   9   10    11
    //    0   1   2   3                   9    10    11
    //  0   1   2   3                   9   10    11
    //
    //  k_offset < 3
    //
    //  (i,j=0)    ->                           |(i-1,j)(i,j)(i+1,j)|(i,j+1)
    //  (i,j=1-10) -> (i-1,j-1)(i+1,j-1)|(i,j-1)|(i-1,j)(i,j)(i+1,j)|(i,j+1)
    //  (i,j=11)   -> (i-1,j-1)(i+1,j-1)|(i,j-1)|(i-1,j)(i,j)(i+1,j)|
    //
    //  k_offset >= 3`
    //
    //  (i,j=0)    ->        |(i-1,j)(i,j)(i+1,j)|(i,j+1)|(i-1,j+1)(i+1,j+1)
    //  (i,j=1-10) -> (i,j-1)|(i-1,j)(i,j)(i+1,j)|(i,j+1)|(i-1,j+1)(i+1,j+1)
    //  (i,j=11)   -> (i,j-1)|(i-1,j)(i,j)(i+1,j)|
    //

    if (tid < 12) {
        pos = 12 + tid;

        filt.x = rcvd_rs[pos].x + rcvd_rs[pos - 12].x + rcvd_rs[pos + 12].x;
        filt.y = rcvd_rs[pos].y + rcvd_rs[pos - 12].y + rcvd_rs[pos + 12].y;
        rs_count = 3;
        if (0 < tid) {
            filt.x += rcvd_rs[pos - 1].x;
            filt.y += rcvd_rs[pos - 1].y;
            rs_count += 1;
            if (k_offset < 3) {
                filt.x += (rcvd_rs[pos - 12 - 1].x + rcvd_rs[pos + 12 - 1].x);
                filt.y += (rcvd_rs[pos - 12 - 1].y + rcvd_rs[pos + 12 - 1].y);
                rs_count += 2;
            }
        }
        if (tid < 11) {
            filt.x += rcvd_rs[pos + 1].x;
            filt.y += rcvd_rs[pos + 1].y;
            rs_count += 1;
            if (3 <= k_offset) {
                filt.x += (rcvd_rs[pos - 12 + 1].x + rcvd_rs[pos + 12 + 1].x);
                filt.y += (rcvd_rs[pos - 12 + 1].y + rcvd_rs[pos + 12 + 1].y);
                rs_count += 2;
            }
        }
        if ((rs_no == 0) || (rs_no == total_rs - 1)) {
            if (((k_offset < 3) && (tid == 0)) || ((k_offset >= 3) && (tid == 11))) {
                rs_count -= 1;
            } else {
                rs_count -= 2;
            }
        }
        filt.x /= rs_count;
        filt.y /= rs_count;

        d_ce_filt[rs_out * 12 + tid].x = filt.x;
        d_ce_filt[rs_out * 12 + tid].y = filt.y;

        double error_r = rcvd_rs[pos].x - filt.x;
        double error_i = rcvd_rs[pos].y - filt.y;

        atomicAdd(&ce_err_pwr, (float)(error_r * error_r + error_i * error_i));
    }

    __syncthreads();

    if (tid == 0) {
        d_err_pwr_acc[bid] = ce_err_pwr;
    }
}




/*
 *
 */
__global__ void chan_est_four_port_step1_kernel(hipfftDoubleComplex *d_tfg, int num_slot,
                                                unsigned short n_id_cell, int n_symb_dl,
                                                // output
                                                hipfftDoubleComplex *d_ce_filt, double *d_err_pwr_acc)
{
    __shared__ unsigned int rs_dl[3];
    __shared__ hipfftDoubleComplex rcvd_rs[12 * 3], tfg_rs, conj_std_rs;
    __shared__ float ce_err_pwr;

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;

    /* port 0 and port 1 has (2 * num_slot) symbols, port 3 and port 4 has (num_slot) symbols */

    int rs_no = (bid < (4 * num_slot)) ? (bid % (2 * num_slot)) : (bid % num_slot);
    int port = (bid < (4 * num_slot)) ? (bid / (num_slot * 2)) : ((bid / num_slot) - 2);
    int rs_out = bid;

    /*  */

    const int port01 = 1 - (port / 2), port23 = port / 2;
    const int rs_per_slot = 1 << port01;
    const int total_rs = num_slot * rs_per_slot;
    const int rs_slot = (rs_no / rs_per_slot) % 20;
    const int l_rs = port01 * ((rs_no & 1) * (n_symb_dl - 3)) + port23;
    const int v = port01 * ((port & 1) ^ (rs_no & 1)) * 3 + port23 * ((port & 1) + (rs_slot & 1)) * 3;
    const int k_offset = (v + (n_id_cell % 6)) % 6;

    int pos, rs_count;
    hipfftDoubleComplex filt;


    if (tid < 3) {
        const int cinit_rs_no = rs_no - 1 + tid;
        const int cinit_rs_slot = (cinit_rs_no / rs_per_slot) % 20;
        const int cinit_l_rs = port01 * ((cinit_rs_no & 1) * (n_symb_dl - 3)) + port23;
        const int cinit = ((7 * (cinit_rs_slot + 1) + cinit_l_rs + 1) * (2 * n_id_cell + 1) << 10) + 2 * n_id_cell + (n_symb_dl == 7 ? 1 : 0);

        pn_seq_lsb_to_msb(cinit, 6 * 2 * 2, (55 - 3) * 2 * 2, &rs_dl[tid]);

        ce_err_pwr = 0.0;
    }

    __syncthreads();

    if (tid < 36) {
        const int copy_rs_no = rs_no - 1 + (tid / 12);
        const int copy_rs_slot = copy_rs_no / rs_per_slot;
        const int copy_l_rs = port01 * ((copy_rs_no & 1) * (n_symb_dl - 3)) + port23;
        const int copy_v = port01 * ((port & 1) ^ (copy_rs_no & 1)) * 3 + port23 * ((port & 1) + (copy_rs_slot & 1)) * 3;
        const int copy_k_offset = (copy_v + (n_id_cell % 6)) % 6;

        const unsigned int rs_bits = (rs_dl[tid / 12] >> (2 * (tid % 12))) & 0x3;
        const int copy_pos = (tid / 12) * 12 + (tid % 12);


        if ((0 <= copy_rs_no) && (copy_rs_no < total_rs)) {
            tfg_rs.x = d_tfg[(copy_rs_slot * n_symb_dl + copy_l_rs) * 72 + copy_k_offset + 6 * (tid % 12)].x;
            tfg_rs.y = d_tfg[(copy_rs_slot * n_symb_dl + copy_l_rs) * 72 + copy_k_offset + 6 * (tid % 12)].y;

            conj_std_rs.x = SQRT2_INV * (1.0 - ((rs_bits & 0x1) * 2));
            conj_std_rs.y = - SQRT2_INV * (1.0 - (rs_bits & 0x2));

            rcvd_rs[copy_pos].x = COMPLEX_MUL_REAL(tfg_rs, conj_std_rs);
            rcvd_rs[copy_pos].y = COMPLEX_MUL_IMAG(tfg_rs, conj_std_rs);
        } else {
            rcvd_rs[copy_pos].x = 0.0;
            rcvd_rs[copy_pos].y = 0.0;
        }
    }

    __syncthreads();

    //  0   1   2   3                   9   10    11
    //    0   1   2   3                   9    10    11
    //  0   1   2   3                   9   10    11
    //    0   1   2   3                   9    10    11
    //  0   1   2   3                   9   10    11
    //
    //  k_offset < 3
    //
    //  (i,j=0)    ->                           |(i-1,j)(i,j)(i+1,j)|(i,j+1)
    //  (i,j=1-10) -> (i-1,j-1)(i+1,j-1)|(i,j-1)|(i-1,j)(i,j)(i+1,j)|(i,j+1)
    //  (i,j=11)   -> (i-1,j-1)(i+1,j-1)|(i,j-1)|(i-1,j)(i,j)(i+1,j)|
    //
    //  k_offset >= 3`
    //
    //  (i,j=0)    ->        |(i-1,j)(i,j)(i+1,j)|(i,j+1)|(i-1,j+1)(i+1,j+1)
    //  (i,j=1-10) -> (i,j-1)|(i-1,j)(i,j)(i+1,j)|(i,j+1)|(i-1,j+1)(i+1,j+1)
    //  (i,j=11)   -> (i,j-1)|(i-1,j)(i,j)(i+1,j)|
    //

    if (tid < 12) {
        pos = 12 + tid;

        filt.x = rcvd_rs[pos].x + rcvd_rs[pos - 12].x + rcvd_rs[pos + 12].x;
        filt.y = rcvd_rs[pos].y + rcvd_rs[pos - 12].y + rcvd_rs[pos + 12].y;
        rs_count = 3;
        if (0 < tid) {
            filt.x += rcvd_rs[pos - 1].x;
            filt.y += rcvd_rs[pos - 1].y;
            rs_count += 1;
            if (k_offset < 3) {
                filt.x += (rcvd_rs[pos - 12 - 1].x + rcvd_rs[pos + 12 - 1].x);
                filt.y += (rcvd_rs[pos - 12 - 1].y + rcvd_rs[pos + 12 - 1].y);
                rs_count += 2;
            }
        }
        if (tid < 11) {
            filt.x += rcvd_rs[pos + 1].x;
            filt.y += rcvd_rs[pos + 1].y;
            rs_count += 1;
            if (3 <= k_offset) {
                filt.x += (rcvd_rs[pos - 12 + 1].x + rcvd_rs[pos + 12 + 1].x);
                filt.y += (rcvd_rs[pos - 12 + 1].y + rcvd_rs[pos + 12 + 1].y);
                rs_count += 2;
            }
        }
        if ((rs_no == 0) || (rs_no == total_rs - 1)) {
            if (((k_offset < 3) && (tid == 0)) || ((k_offset >= 3) && (tid == 11))) {
                rs_count -= 1;
            } else {
                rs_count -= 2;
            }
        }

        filt.x /= rs_count;
        filt.y /= rs_count;

        d_ce_filt[rs_out * 12 + tid].x = filt.x;
        d_ce_filt[rs_out * 12 + tid].y = filt.y;

        double error_r = rcvd_rs[pos].x - filt.x;
        double error_i = rcvd_rs[pos].y - filt.y;

        atomicAdd(&ce_err_pwr, (float)(error_r * error_r + error_i * error_i));
    }

    __syncthreads();

    if (tid == 0) {
        d_err_pwr_acc[bid] = ce_err_pwr;
    }
}




/*
 *
 */
__global__ void chan_est_four_port_step2_kernel(double *d_err_pwr_acc, int num_slot,
                                                // output
                                                double *d_np)
{
    extern __shared__ double err_pwr_acc[];

    const unsigned int bid = blockIdx.x;
    const unsigned int tid = threadIdx.x;

    const int port = bid;
    const int no_sym = port < 2 ? 2 * num_slot : num_slot;
    const int rs_no = port < 2 ? port * 2 : 4 + (port - 2);
    double *d_err_pwr_acc_start = &d_err_pwr_acc[rs_no * num_slot];

    if (tid < no_sym) {
        err_pwr_acc[tid] = d_err_pwr_acc_start[tid];
    }

    __syncthreads();

    for (unsigned int s = 128; s > 0; s >>= 1) {
        if ((tid < s) && (tid + s < no_sym)) {
            err_pwr_acc[tid] += err_pwr_acc[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_np[bid] = err_pwr_acc[0] / (no_sym * 12);
    }
}



/*
 *  for debug
 */
__global__ void print_kernel_float(float *d_data, int len)
{
    printf("\n\nprint_kernel_float\n");
    for (int i = 0; i < len;i++)
        printf("(%f)", d_data[i]);
    printf("\n\n");
}


__global__ void print_kernel_double(double *d_data, int len)
{
    printf("\n\nprint_kernel_double\n");
    for (int i = 0; i < len;i++)
        printf("(%f)",d_data[i]);
    printf("\n\n");
}

__global__ void print_kernel_complex(hipfftDoubleComplex *d_data, int len)
{
    printf("\n\nprint_kernel_complex\n");
    for (int i = 0; i < len;i++)
        printf("(%lf,%lf)", d_data[i].x, d_data[i].y);
    printf("\n\n");
}


/*
 *
 */
extern "C" Cell extract_tfg_and_tfoec(
    const Cell & cell,
    const cvec & capbuf_raw,
    const double & fc_requested,
    const double & fc_programmed,
    const double & fs_programmed,
    // Output
    cmat & my_tfg_comp)
{
    const unsigned int n_cap = capbuf_raw.length();

    Cell cell_out(cell);

    hipfftDoubleComplex *h_capbuf = (hipfftDoubleComplex *)NULL, *d_capbuf = (hipfftDoubleComplex *)NULL;
    hipfftDoubleComplex *h_tfg = (hipfftDoubleComplex *)NULL, *d_tfg = (hipfftDoubleComplex *)NULL;
    hipfftDoubleComplex *d_rs_extracted = (hipfftDoubleComplex *)NULL;
    double h_frame_start;
    int h_n_id_1_est, h_cp_type;
    double h_residual_f, *d_residual_f = (double *)NULL;
    double h_adjust_f, *d_adjust_f = (double *)NULL;
    double *d_tfg_timestamp = (double *)NULL;
    float *d_sss_h12_np_est = (float *)NULL;
    hipfftDoubleComplex *d_sss_h12_est = (hipfftDoubleComplex *)NULL;
    double *d_log_lik = (double *)NULL, *d_frame_start = (double *)NULL;
    int *d_n_id_1_est = (int *)NULL, *d_cp_type = (int *)NULL;

    hipfftDoubleComplex *d_h_sm = (hipfftDoubleComplex *)NULL;
    hipfftDoubleComplex *d_sss_raw = (hipfftDoubleComplex *)NULL;
    double *d_pss_np_inv = (double *)NULL;

    hipfftDoubleComplex *d_M = (hipfftDoubleComplex *)NULL;
    double *d_err_pwr_acc = (double *)NULL;

    checkCudaErrors(hipMalloc(&d_rs_extracted, (2 + 2 + 1 + 1) * 122 * 12 * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_adjust_f, sizeof(double)));
    checkCudaErrors(hipMalloc(&d_residual_f, sizeof(double)));

    checkCudaErrors(hipMalloc(&d_n_id_1_est, sizeof(int)));
    checkCudaErrors(hipMalloc(&d_cp_type, sizeof(int)));
    checkCudaErrors(hipMalloc(&d_frame_start, sizeof(double)));

    checkCudaErrors(hipMalloc(&d_err_pwr_acc, (2 + 2 + 1 + 1) * 122  * sizeof(double)));

    h_capbuf = (hipfftDoubleComplex *)malloc(n_cap * sizeof(hipfftDoubleComplex));

    for (unsigned int i = 0; i < n_cap; i++) {
        h_capbuf[i].x = capbuf_raw[i].real();
        h_capbuf[i].y = capbuf_raw[i].imag();
    }
    checkCudaErrors(hipMalloc(&d_capbuf, n_cap * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMemcpy(d_capbuf, h_capbuf, n_cap * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

    /* prologue of function sss_detect() of searcher.cpp */

    const double peak_freq = cell_out.freq;
    double k_factor = (fc_requested - peak_freq) / fc_programmed; 
    double peak_loc = (cell.ind + 9 < 162 ? cell_out.ind + 9600 * k_factor : cell_out.ind);
    const unsigned int n_id_2_est = cell_out.n_id_2;
    const int n_pss = ceil((n_cap - 125 - 9 - peak_loc) / (k_factor * 9600));

    checkCudaErrors(hipMalloc(&d_h_sm, 62 * n_pss * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_sss_raw, n_pss * 62 * 2 * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_pss_np_inv, n_pss * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_sss_h12_np_est, 62 * 2 * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_sss_h12_est, 62 * 2 * 2 * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_log_lik, 168 * 2 * 2 * sizeof(double)));

    sss_detect_getce_sss_multiblocks_step1_kernel<<<n_pss, 128>>>(d_capbuf, n_pss,
                                                                  n_id_2_est, peak_loc,
                                                                  fc_requested, fc_programmed, fs_programmed, peak_freq,
                                                                  // output
                                                                  d_h_sm, d_pss_np_inv, &d_sss_raw[0], &d_sss_raw[n_pss * 62]);
    checkCudaErrors(hipDeviceSynchronize());

    sss_detect_getce_sss_multiblocks_step2_kernel<<<62, n_pss>>>(n_pss,
                                                                 d_h_sm, d_pss_np_inv, &d_sss_raw[0], &d_sss_raw[n_pss * 62],
                                                                 // output
                                                                 d_sss_h12_np_est, &d_sss_h12_est[0], &d_sss_h12_est[62 * 2]);
    checkCudaErrors(hipDeviceSynchronize());

    sss_detect_ml_kernel<<<168*4, 124>>>(d_sss_h12_np_est, &d_sss_h12_est[0], &d_sss_h12_est[62 * 2], n_id_2_est,
                                         &d_log_lik[0], &d_log_lik[168 * 2]);
    checkCudaErrors(hipDeviceSynchronize());

#define THRESH2_N_SIGMA 3
    sss_detect_ml_decision_kernel<<<1, 168*4>>>(&d_log_lik[0], THRESH2_N_SIGMA, cell.ind,
                                                fc_requested, fc_programmed, fs_programmed, peak_freq,
                                                // output
                                                d_n_id_1_est, d_frame_start, d_cp_type);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(&h_cp_type, d_cp_type, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&h_n_id_1_est, d_n_id_1_est, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&h_frame_start, d_frame_start, sizeof(double), hipMemcpyDeviceToHost));

    cell_out.frame_start = h_frame_start;
    cell_out.n_id_1 = h_n_id_1_est;
    cell_out.cp_type = (cp_type_t::cp_type_t)h_cp_type;

    const int n_symb_dl = (h_cp_type == (int)cp_type_t::NORMAL ? 7 : 6);
    const int n_ofdm_sym = (6*10*2+2)*n_symb_dl;
    const double frame_start = h_frame_start;

    h_tfg = (hipfftDoubleComplex *)malloc(n_ofdm_sym * 12 * 6 * sizeof(hipfftDoubleComplex));

    checkCudaErrors(hipMalloc(&d_tfg, n_ofdm_sym * 12 * 6 * sizeof(hipfftDoubleComplex)));
    checkCudaErrors(hipMalloc(&d_tfg_timestamp, n_ofdm_sym * sizeof(double)));

    /* prologue of function pss_sss_foe() of searcher.cpp */

    const unsigned int pss_sss_dist = lround((128 + (n_symb_dl == 7 ? 9 : 32)) * 16 / FS_LTE * fs_programmed * k_factor);
    double first_sss_dft_location = frame_start + (960 - 128 - (n_symb_dl == 7 ? 9 : 32)-128) * 16 / FS_LTE * fs_programmed * k_factor;
    const int n_sss = ceil((n_cap - 127 - pss_sss_dist - 100) / (9600 * 16 / FS_LTE * fs_programmed * k_factor));
    int sn = 0;
    first_sss_dft_location = fmod(first_sss_dft_location + 0.5, 19200.0) - 0.5;
    if (first_sss_dft_location - 9600 * k_factor > -0.5) {
        first_sss_dft_location -= 9600 * k_factor;
        sn = 1;
    }

    checkCudaErrors(hipMalloc(&d_M, n_sss * sizeof(hipfftDoubleComplex)));

    const unsigned int n_id_cell = cell_out.n_id_cell();

    pss_sss_foe_multiblocks_step1_kernel<<<n_sss, 128>>>(d_capbuf, n_sss,
                                                         n_id_cell, n_symb_dl, first_sss_dft_location, pss_sss_dist, sn,
                                                         fc_requested, fc_programmed, fs_programmed, cell.freq,
                                                         // output
                                                         d_M);
    checkCudaErrors(hipDeviceSynchronize());

    pss_sss_foe_multiblocks_step2_kernel<<<1, n_sss>>>(d_M, n_sss, pss_sss_dist,
                                                       fc_requested, fc_programmed, fs_programmed, cell.freq,
                                                       // output
                                                       d_adjust_f);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(&h_adjust_f, d_adjust_f, sizeof(double), hipMemcpyDeviceToHost));

    cell_out.freq_fine = cell_out.freq + h_adjust_f;

    extract_tfg_multiblocks_kernel<<<n_ofdm_sym, 128>>>(d_capbuf, d_tfg, d_tfg_timestamp, d_adjust_f,
                                                        n_id_cell, n_symb_dl, frame_start,
                                                        fc_requested, fc_programmed, fs_programmed, cell_out.freq);
    checkCudaErrors(hipDeviceSynchronize());

    tfoec_kernel<<<1, n_ofdm_sym>>>(d_tfg, d_rs_extracted, d_tfg_timestamp,
                                    n_id_cell, n_symb_dl,
                                    fc_requested, fc_programmed, fs_programmed,
                                    // output
                                    d_residual_f);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(&h_residual_f, d_residual_f, sizeof(double), hipMemcpyDeviceToHost));

    cell_out.freq_superfine = cell_out.freq_fine + h_residual_f;

    hipfftDoubleComplex *d_ce_filt = d_rs_extracted; // re-use memory allocated for intermediate variables
    double *d_np = d_log_lik; // re-use memory allocated for intermediate variables

    chan_est_four_port_step1_kernel<<<122 * 6, 36>>>(d_tfg, 122, n_id_cell, n_symb_dl, &d_ce_filt[122 * 0 * 12], d_err_pwr_acc);

    chan_est_four_port_step2_kernel<<<4, 122 * 2, 2 * 122 * sizeof(double)>>>(d_err_pwr_acc, 122, &d_np[0]);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_tfg, d_tfg, n_ofdm_sym * 12 * 6 * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    my_tfg_comp = cmat(n_ofdm_sym, 72);
    for (int i = 0; i < n_ofdm_sym; i++) {
        for (unsigned int j = 0; j < 72; j++) {
            my_tfg_comp(i,j).real() = h_tfg[i * 72 + j].x;
            my_tfg_comp(i,j).imag() = h_tfg[i * 72 + j].y;
        }
    }

    free(h_capbuf);
    free(h_tfg);

    checkCudaErrors(hipFree(d_capbuf));
    checkCudaErrors(hipFree(d_tfg));
    checkCudaErrors(hipFree(d_rs_extracted));
    checkCudaErrors(hipFree(d_adjust_f));
    checkCudaErrors(hipFree(d_residual_f));
    checkCudaErrors(hipFree(d_tfg_timestamp));
    checkCudaErrors(hipFree(d_sss_h12_np_est));
    checkCudaErrors(hipFree(d_sss_h12_est));
    checkCudaErrors(hipFree(d_frame_start));
    checkCudaErrors(hipFree(d_n_id_1_est));
    checkCudaErrors(hipFree(d_cp_type));

    checkCudaErrors(hipFree(d_h_sm));
    checkCudaErrors(hipFree(d_sss_raw));
    checkCudaErrors(hipFree(d_pss_np_inv));

    checkCudaErrors(hipFree(d_M));

    checkCudaErrors(hipFree(d_err_pwr_acc));
    return cell_out;
}

